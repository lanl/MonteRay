#include "genericGPU_test_helper.hh"

#include <iostream>

#include "MonteRayDefinitions.hh"

GenericGPUTestHelper::GenericGPUTestHelper(){}

GenericGPUTestHelper::~GenericGPUTestHelper(){}

void GenericGPUTestHelper::setupTimers(){
#ifdef __HIPCC__
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
#endif
}

void GenericGPUTestHelper::stopTimers(){
	float elapsedTime;

#ifdef __HIPCC__
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop );
#endif

	std::cout << "Elapsed time in CUDA kernel=" << elapsedTime << " msec" << std::endl;
}


