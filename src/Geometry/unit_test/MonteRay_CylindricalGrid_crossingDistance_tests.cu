#include "hip/hip_runtime.h"
#include <UnitTest++.h>

#include "MonteRay_CylindricalGrid.hh"
#include "MonteRay_SpatialGrid.hh"
#include "MonteRayVector3D.hh"
#include "MonteRay_GridBins.hh"
#include "MonteRayDefinitions.hh"
#include "MonteRayCopyMemory.t.hh"

namespace MonteRay_CylindricalGrid_crossingDistance_tests{

using namespace MonteRay;

SUITE( MonteRay_CylindricalGrid_crossingDistance_Tests) {
    using Grid_t = MonteRay_CylindricalGrid;
    using GridBins_t = MonteRay_GridBins;
    using GridBins_t = Grid_t::GridBins_t;
    using pGridInfo_t = GridBins_t*;
    using pArrayOfpGridInfo_t = Grid_t::pArrayOfpGridInfo_t;
    using Position_t = MonteRay::Vector3D<gpuRayFloat_t>;

    const gpuFloatType_t s2 = std::sqrt(2.0);
    const unsigned OUTSIDE_GRID = MonteRay_GridSystemInterface::OUTSIDE_GRID;

    enum coord {R=0,Z=1,Theta=2,DIM=3};

    class gridTestData {
    public:

        gridTestData(){
            std::vector<gpuRayFloat_t> Rverts = { 1.0, 2.0, 3.0, 5.0 };
            std::vector<gpuRayFloat_t> Zverts = { 0.0, 1.0, 2.0, 3.0, 5.0 };

            pGridInfo[R] = new GridBins_t();
            pGridInfo[Z] = new GridBins_t();

            pGridInfo[R]->initialize( Rverts );
            pGridInfo[Z]->initialize( Zverts );

        }
        ~gridTestData(){
            delete pGridInfo[R];
            delete pGridInfo[Z];
        }

        MonteRay_SpatialGrid::pArrayOfpGridInfo_t pGridInfo;
    };

    typedef singleDimRayTraceMap_t distances_t;
    typedef singleDimRayTraceMap_t rayTraceMap_t;
    typedef rayTraceList_t rayTrace_t;
    typedef MonteRay_CylindricalGrid CylindricalGrid;

    inline void checkDistances( const char *file, int line,
            const std::vector<unsigned>& expectedIndex,
            const std::vector<gpuFloatType_t>& expectedDistance, const singleDimRayTraceMap_t& distances )
    {
        char const* const errorFormat = "%s(%d): error: Failure \n";
        if( expectedIndex.size() != expectedDistance.size() ) {
            printf(errorFormat, file, line);
        }
        CHECK_EQUAL( expectedIndex.size(), expectedDistance.size() );

        if( expectedIndex.size() != distances.size() ) {
            printf(errorFormat, file, line);
        }
        CHECK_EQUAL( expectedIndex.size(), distances.size() );

        for( auto i=0; i<distances.size(); ++i ) {
            if( expectedIndex[i] != distances.id(i) ) {
                printf("%s(%d): error: Failure in cell id #%d \n", file, line, i);
            }
            CHECK_EQUAL( expectedIndex   [i], distances.id(i) );

            if( std::abs( expectedDistance[i] - distances.dist(i) ) > 1.0e-5  ) {
                printf("%s(%d): error: Failure in distance #%d \n", file, line, i);
            }
            CHECK_CLOSE( expectedDistance[i], distances.dist(i), 1e-5 );
        }
    }

#define checkDistances(expectedIndex, expectedDistance, distances) { checkDistances(__FILE__, __LINE__, expectedIndex, expectedDistance, distances); }

    // ************************ rayTrace Testing ****************************

    TEST( CrossingDistance_in_1D_R_inward_from_outside_to_outside ) {

        gridTestData data;
        CylindricalGrid grid(2, data.pGridInfo);

        Position_t position (  -6.5, 0.0,  0.5 );
        Position_t direction(    1,   0,    0 );
        gpuFloatType_t distance = 100.0;

        singleDimRayTraceMap_t distances;
        grid.radialCrossingDistancesSingleDirection( distances, position, direction, distance, false);

        CHECK_EQUAL( 8,  distances.size() );
        checkDistances( std::vector<unsigned>({4,3,2,1,0,1,2,3}),
                std::vector<gpuFloatType_t>({1.5,3.5,4.5,5.5,7.5,8.5,9.5,11.5}),
                distances );
    }

    TEST( CrossingDistance_in_1D_R_inward_from_outside_to_inside_stop_inward ) {

        gridTestData data;
        CylindricalGrid grid(2, data.pGridInfo);

        Position_t position (  -6.5, 0.0,  0.5 );
        Position_t direction(    1,   0,    0 );
        gpuFloatType_t distance = 6.0;

        singleDimRayTraceMap_t distances;
        grid.radialCrossingDistancesSingleDirection( distances, position, direction, distance, false);

        CHECK_EQUAL( 5,  distances.size() );
        checkDistances( std::vector<unsigned>({4,3,2,1,0}),
                std::vector<gpuFloatType_t>({1.5,3.5,4.5,5.5,6.0}),
                distances );
    }

    TEST( CrossingDistance_in_1D_R_inward_from_outside_to_inside_stop_outward ) {

        gridTestData data;
        CylindricalGrid grid(2, data.pGridInfo);

        Position_t position (  -6.5, 0.0,  0.5 );
        Position_t direction(    1,   0,    0 );
        gpuFloatType_t distance = 9.0;

        singleDimRayTraceMap_t distances;
        grid.radialCrossingDistancesSingleDirection( distances, position, direction, distance, false);

        CHECK_EQUAL( 7,  distances.size() );
        checkDistances( std::vector<unsigned>({4,3,2,1,0,1,2}),
                std::vector<gpuFloatType_t>({1.5,3.5,4.5,5.5,7.5,8.5,9.0}),
                distances );
    }

    TEST( CrossingDistance_through_a_single_cylinder_in_2D_R_inward_from_inside_to_outside ) {

        gridTestData data;
        CylindricalGrid grid(2, data.pGridInfo);

        gpuFloatType_t y = 3.0f / std::sqrt(2.0f );
        Position_t position (  -4.0, y,  0.5 );
        Position_t direction(    1,   0,    0 );
        gpuFloatType_t distance = 9.0;

        singleDimRayTraceMap_t distances;
        grid.radialCrossingDistancesSingleDirection( distances, position, direction, distance, false);

        CHECK_EQUAL( 2,  distances.size() );
        checkDistances( std::vector<unsigned>({3,2}),
                std::vector<gpuFloatType_t>({4.0f-y,4.0f+y}),
                distances );
    }

    TEST( CrossingDistance_tanget_to_first_inner_cylinder_posY ) {
        gridTestData data;
        CylindricalGrid grid(2, data.pGridInfo );

        gpuFloatType_t x = -3.5;
        gpuFloatType_t y = 3.0;

        Position_t position (  x, y, 0.5 );
        Position_t direction(  1, 0,   0 );
        gpuFloatType_t distance = 9.0;

        singleDimRayTraceMap_t distances;
        grid.radialCrossingDistancesSingleDirection( distances, position, direction, distance, false);

        CHECK_EQUAL( 0, distances.size() );
    }
    TEST( CrossingDistance_tanget_to_first_inner_cylinder_negY ) {
        gridTestData data;
        CylindricalGrid grid(2, data.pGridInfo );

        gpuFloatType_t x = -3.5;
        gpuFloatType_t y = -3.0;

        Position_t position (  x, y, 0.5 );
        Position_t direction(    1,   0,    0 );
        gpuFloatType_t distance = 9.0;

        singleDimRayTraceMap_t distances;
        grid.radialCrossingDistancesSingleDirection( distances, position, direction, distance, false);

        CHECK_EQUAL( 0, distances.size() );
    }
    TEST( CrossingDistance_tanget_to_first_second_cylinder_posY ) {
        gridTestData data;
        CylindricalGrid grid(2, data.pGridInfo );

        gpuFloatType_t y = 2.0;
        Position_t position (  -4.0, y,  0.5 );
        Position_t direction(    1,   0,    0 );
        gpuFloatType_t distance = 9.0;

        singleDimRayTraceMap_t distances;
        grid.radialCrossingDistancesSingleDirection( distances, position, direction, distance, false);

        CHECK_EQUAL( 2, distances.size() );
        CHECK_EQUAL( 3, distances.id(0) );
        CHECK_CLOSE( 4.0 - std::sqrt(9.0-4.0), distances.dist(0), 1e-5 );
        CHECK_EQUAL( 2, distances.id(1) );
        CHECK_CLOSE( 4.0 + std::sqrt(9.0-4.0), distances.dist(1), 1e-5 );
    }

    TEST( CrossingDistance_outward_from_Origin_posX_to_outside ) {
        //        std::cout << "Debug: ---------------------------------------------------------" << std::endl;
        gridTestData data;
        CylindricalGrid grid(2, data.pGridInfo );

        Position_t position (  0.0, 0.0,  0.5 );
        Position_t direction(    1,   0,    0 );
        gpuFloatType_t distance = 9.0;

        singleDimRayTraceMap_t distances;
        grid.radialCrossingDistancesSingleDirection( distances, position, direction, distance, true);

        CHECK_EQUAL( 5, distances.size() );
        CHECK_EQUAL( 0, distances.id(0) );
        CHECK_CLOSE( 1.0, distances.dist(0), 1e-5 );
        CHECK_EQUAL( 1, distances.id(1) );
        CHECK_CLOSE( 2.0, distances.dist(1), 1e-5 );
        CHECK_EQUAL( 2, distances.id(2) );
        CHECK_CLOSE( 3.0, distances.dist(2), 1e-5 );
        CHECK_EQUAL( 3, distances.id(3) );
        CHECK_CLOSE( 5.0, distances.dist(3), 1e-5 );
        CHECK_EQUAL( 4, distances.id(4) );
        CHECK_CLOSE( 9.0, distances.dist(4), 1e-5 );
    }
    TEST( CrossingDistance_outward_from_Origin_posX_to_inside ) {
        //        std::cout << "Debug: ---------------------------------------------------------" << std::endl;
        gridTestData data;
        CylindricalGrid grid(2, data.pGridInfo );

        Position_t position (  0.0, 0.0,  0.5 );
        Position_t direction(    1,   0,    0 );
        gpuFloatType_t distance = 4.5;

        singleDimRayTraceMap_t distances;
        grid.radialCrossingDistancesSingleDirection( distances, position, direction, distance, true);

        CHECK_EQUAL( 4, distances.size() );
        CHECK_EQUAL( 0, distances.id(0) );
        CHECK_CLOSE( 1.0, distances.dist(0), 1e-5 );
        CHECK_EQUAL( 1, distances.id(1) );
        CHECK_CLOSE( 2.0, distances.dist(1), 1e-5 );
        CHECK_EQUAL( 2, distances.id(2) );
        CHECK_CLOSE( 3.0, distances.dist(2), 1e-5 );
        CHECK_EQUAL( 3, distances.id(3) );
        CHECK_CLOSE( 4.5, distances.dist(3), 1e-5 );
    }


    TEST( CrossingDistance_outward_from_posX_Postion_negX_Direction ) {
        //        std::cout << "Debug: ---------------------------------------------------------" << std::endl;
        gridTestData data;
        CylindricalGrid grid(2, data.pGridInfo );

        Position_t position (  3.5, 0.0,  0.5 );
        Position_t direction(   -1,   0,    0 );
        gpuFloatType_t distance = 9.0;

        singleDimRayTraceMap_t distances;
        grid.radialCrossingDistancesSingleDirection( distances, position, direction, distance, true);

        CHECK_EQUAL( 2, distances.size() );
        CHECK_EQUAL( 3, distances.id(0) );
        CHECK_CLOSE( 8.5, distances.dist(0), 1e-5 );
        CHECK_EQUAL( 4, distances.id(1) );
        CHECK_CLOSE( 9.0, distances.dist(1), 1e-5 );
    }

    TEST( CrossingDistance_outward_from_posX_Postion_negX_Direction_not_outside ) {
        // std::cout << "Debug: ---------------------------------------------------------" << std::endl;
        gridTestData data;
        CylindricalGrid grid(2, data.pGridInfo );

        Position_t position (  3.5, 0.0,  0.5 );
        Position_t direction(   -1,   0,    0 );
        gpuFloatType_t distance = 7.5;

        singleDimRayTraceMap_t distances;
        grid.radialCrossingDistancesSingleDirection( distances, position, direction, distance, true);

        CHECK_EQUAL( 1, distances.size() );
        CHECK_EQUAL( 3, distances.id(0) );
        CHECK_CLOSE( 7.5, distances.dist(0), 1e-5 );
    }

    TEST( radialCrossingDistances_inside_thru_to_outside ) {
        gridTestData data;
        CylindricalGrid grid(2, data.pGridInfo );

        Position_t position (  -4.5, 0.0,  0.5 );
        Position_t direction(    1,   0,    0 );
        gpuFloatType_t distance = 100.0;

        singleDimRayTraceMap_t distances;
        grid.radialCrossingDistances( distances, position, direction, distance);

        CHECK_EQUAL( 8, distances.size() );
        CHECK_EQUAL( 3, distances.id(0) );
        CHECK_CLOSE( 1.5, distances.dist(0), 1e-5 );
        CHECK_EQUAL( 2, distances.id(1) );
        CHECK_CLOSE( 2.5, distances.dist(1), 1e-5 );
        CHECK_EQUAL( 1, distances.id(2) );
        CHECK_CLOSE( 3.5, distances.dist(2), 1e-5 );
        CHECK_EQUAL( 0, distances.id(3) );
        CHECK_CLOSE( 5.5, distances.dist(3), 1e-5 );
        CHECK_EQUAL( 1, distances.id(4) );
        CHECK_CLOSE( 6.5, distances.dist(4), 1e-5 );
        CHECK_EQUAL( 2, distances.id(5) );
        CHECK_CLOSE( 7.5, distances.dist(5), 1e-5 );
        CHECK_EQUAL( 3, distances.id(6) );
        CHECK_CLOSE( 9.5, distances.dist(6), 1e-5 );
        CHECK_EQUAL( 4, distances.id(7) );
        CHECK_CLOSE( 100.0, distances.dist(7), 1e-5 );
    }

    TEST( radialCrossingDistances_inside_misses_inner_cells ) {
        gridTestData data;
        CylindricalGrid grid(2, data.pGridInfo );

        Position_t position (  -3.5, 3.1,  0.5 );
        Position_t direction(    1,   0,    0 );
        gpuFloatType_t distance = 100.0;

        singleDimRayTraceMap_t distances;
        grid.radialCrossingDistances( distances, position, direction, distance);

        CHECK_EQUAL( 2, distances.size() );
        CHECK_EQUAL( 3, distances.id(0) );
        CHECK_CLOSE( 3.5+std::sqrt(5.0*5.0-3.1*3.1), distances.dist(0), 1e-5 );
        CHECK_EQUAL( 4, distances.id(1) );
        CHECK_CLOSE( 100.0, distances.dist(1), 1e-5 );
    }

    TEST( radialCrossingDistances_twice_through_a_single_cylinder_going_inward_single_crossing_outward  ) {
        gridTestData data;
        CylindricalGrid grid(2, data.pGridInfo );

        gpuFloatType_t y = 3.0 / std::sqrt(2.0 );
        Position_t position (  -4.0, y,  0.5 );
        Position_t direction(    1,   0,    0 );
        gpuFloatType_t distance = 9.0;

        singleDimRayTraceMap_t distances;
        grid.radialCrossingDistances( distances, position, direction, distance);

        CHECK_EQUAL( 4, distances.size() );
        CHECK_EQUAL( 3, distances.id(0) );
        CHECK_CLOSE( 4.0 - y, distances.dist(0), 1e-5 );
        CHECK_EQUAL( 2, distances.id(1) );
        CHECK_CLOSE( 4.0 + y, distances.dist(1), 1e-5 );
        CHECK_EQUAL( 3, distances.id(2) );
        CHECK_CLOSE( 4.0 + std::sqrt(5.0*5.0-y*y) , distances.dist(2), 1e-5 );
        CHECK_EQUAL( 4, distances.id(3) );
        CHECK_CLOSE( 9.0, distances.dist(3), 1e-5 );
    }


}

}

