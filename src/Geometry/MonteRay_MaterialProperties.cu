#include "hip/hip_runtime.h"
#include "MonteRay_MaterialProperties.hh"

#include <iostream>
#include <fstream>
#include <ostream>

#include "GPUErrorCheck.hh"
#include "MonteRay_binaryIO.hh"

namespace MonteRay{
typedef MonteRay_MaterialProperties_Data::offset_t offset_t;
typedef MonteRay_MaterialProperties_Data::MatID_t MatID_t;
typedef MonteRay_MaterialProperties_Data::Density_t Density_t;


void copy(MonteRay_MaterialProperties_Data& theCopy, const MonteRay_MaterialProperties_Data& theOrig) {
	theCopy = theOrig;
}

void copy(MonteRay_MaterialProperties_Data* pCopy, const MonteRay_MaterialProperties_Data* pOrig) {
	copy( *pCopy, *pOrig);
}

void ctor(MonteRay_MaterialProperties_Data* ptr, unsigned numCells, unsigned numComponents ) {
    if( numCells <=0 ) { numCells = 1; }
    if( numComponents <=0 ) { numComponents = 1; }

    ptr->numCells = numCells;
    ptr->numMaterialComponents = numComponents;

    unsigned long long allocSize = sizeof(offset_t)*(numCells+1);
    ptr->offset = (offset_t*) malloc( allocSize);
    if(ptr->offset == 0) abort ();

    allocSize = sizeof(MatID_t)*numComponents;
    ptr->ID = (MatID_t*) malloc( allocSize);
    if(ptr->ID == 0) abort ();

    allocSize = sizeof(Density_t)*numComponents;
    ptr->density = (Density_t*) malloc( allocSize);
    if(ptr->density == 0) abort ();

    for( unsigned i=0; i<numCells; ++i ){
    	ptr->offset[i] = 0;
    }

    for( unsigned i=0; i<numComponents; ++i ){
    	ptr->ID[i] = 0;
    	ptr->density[i] = 0.0;
    }
}

void cudaCtor(MonteRay_MaterialProperties_Data* ptr, unsigned numCells, unsigned numComponents ) {
    if( numCells <=0 ) { numCells = 1; }
    if( numComponents <=0 ) { numComponents = 1; }

    ptr->numCells = numCells;
    ptr->numMaterialComponents = numComponents;

    unsigned long long allocSize = sizeof(offset_t)*(numCells+1);
    CUDA_CHECK_RETURN( hipMalloc(&ptr->offset, allocSize ));

    allocSize = sizeof(MatID_t)*numComponents;
    CUDA_CHECK_RETURN( hipMalloc(&ptr->ID, allocSize ));

    allocSize = sizeof(Density_t)*numComponents;
    CUDA_CHECK_RETURN( hipMalloc(&ptr->density, allocSize ));
}

void cudaCtor(struct MonteRay_MaterialProperties_Data* pCopy, struct MonteRay_MaterialProperties_Data* pOrig){
	unsigned numCells = pOrig->numCells;
	unsigned numComponents = pOrig->numMaterialComponents;
	cudaCtor( pCopy, numCells, numComponents);
}


void dtor(struct MonteRay_MaterialProperties_Data* ptr){
    if( ptr->offset != 0 ) {
        free(ptr->offset);
        ptr->offset = 0;
    }
    if( ptr->ID != 0 ) {
    	free(ptr->ID);
    	ptr->ID = 0;
    }
    if( ptr->density != 0 ) {
    	free(ptr->density);
    	ptr->density = 0;
    }
}

#ifdef CUDA
void cudaDtor(MonteRay_MaterialProperties_Data* ptr) {
	hipFree( ptr->offset );
	hipFree( ptr->ID );
	hipFree( ptr->density );
}
#endif

void MonteRay_MaterialProperties::cudaDtor(void) {
#ifdef CUDA
    if( cudaCopyMade ) {
        hipFree( ptrData_device );
        MonteRay::cudaDtor( tempData );
        delete tempData;
    }
#endif
}

void MonteRay_MaterialProperties::copyToGPU(void) {
	cudaCopyMade = true;
	tempData = new MonteRay_MaterialProperties_Data;

	// allocate target dynamic memory
	MonteRay::cudaCtor( tempData, size(), numMatSpecs() );

	// allocate target struct
	CUDA_CHECK_RETURN( hipMalloc(&ptrData_device, sizeof( MonteRay_MaterialProperties_Data) ));

	// copy allocated data arrays
	unsigned long long allocSize = sizeof(offset_t)*(tempData->numCells+1);
	CUDA_CHECK_RETURN( hipMemcpy(tempData->offset, getOffsetData(), allocSize, hipMemcpyHostToDevice));

	allocSize = sizeof(MatID_t)*tempData->numMaterialComponents;
	CUDA_CHECK_RETURN( hipMemcpy(tempData->ID, getMaterialIDData(), allocSize, hipMemcpyHostToDevice));

	allocSize = sizeof(Density_t)*tempData->numMaterialComponents;
	CUDA_CHECK_RETURN( hipMemcpy(tempData->density, getMaterialDensityData(), allocSize, hipMemcpyHostToDevice));

	// copy struct
	CUDA_CHECK_RETURN( hipMemcpy(ptrData_device, tempData, sizeof( MonteRay_MaterialProperties_Data ), hipMemcpyHostToDevice));
}

#ifdef CUDA
__device__ __host__
#endif
size_t getNumCells(struct MonteRay_MaterialProperties_Data* ptr ) {
    return ptr->numCells;
}

#ifdef CUDA
__device__ __host__
#endif
offset_t getNumMats(struct MonteRay_MaterialProperties_Data* ptr, unsigned i ){
    return ptr->offset[i+1] - ptr->offset[i];
}

#ifdef CUDA
__device__ __host__
#endif
Density_t getDensity(struct MonteRay_MaterialProperties_Data* ptr, unsigned cellNum, unsigned matNum ){
    return ptr->density[ ptr->offset[cellNum] + matNum];
}

#ifdef CUDA
__device__ __host__
#endif
MatID_t getMatID(struct MonteRay_MaterialProperties_Data* ptr, unsigned cellNum, unsigned matNum ){
	return ptr->ID[ ptr->offset[cellNum] + matNum];
}

#ifdef CUDA
__global__ void kernelGetNumCells(MonteRay_MaterialProperties_Data* mp, unsigned* results ) {
     results[0] = getNumCells(mp);
}
#endif


#ifdef CUDA
__global__ void kernelGetNumMaterials(MonteRay_MaterialProperties_Data* mp, unsigned cellNum, MonteRay_MaterialProperties_Data::Material_Index_t* results ) {
	results[0] = getNumMats(mp, cellNum);
}
#endif

#ifdef CUDA
__global__ void kernelGetMaterialID(MonteRay_MaterialProperties_Data* mp, unsigned cellNum, unsigned i, MonteRay_MaterialProperties_Data::MatID_t* results ) {
	results[0] = getMatID(mp, cellNum, i);
}
#endif

#ifdef CUDA
__global__ void kernelGetMaterialDensity(MonteRay_MaterialProperties_Data* mp, unsigned cellNum, unsigned i, MonteRay_MaterialProperties_Data::Density_t* results ) {
	results[0] = getDensity(mp, cellNum, i);
}
#endif


#ifdef CUDA
__global__ void kernelSumMatDensity(MonteRay_MaterialProperties_Data* mp, MonteRay_MaterialProperties_Data::MatID_t matIndex, MonteRay_MaterialProperties_Data::Density_t* results ) {
    gpuFloatType_t sum = 0.0f;
    for( unsigned cell=0; cell < getNumCells(mp); ++cell) {
         for( unsigned matNum=0; matNum < getNumMats(mp, cell); ++matNum ) {

        	 Density_t density = getDensity(mp, cell, matNum);
        	 MatID_t matID = getMatID(mp, cell, matNum);

             if( matID == matIndex ) {
                 sum += density;
             }
         }
     }
     results[0] = sum;
}
#endif


size_t MonteRay_MaterialProperties::launchGetNumCells(void) const{
	typedef unsigned type_t;

	type_t* result_device;
	type_t result[1];
	CUDA_CHECK_RETURN( hipMalloc( &result_device, sizeof( type_t) * 1 ));

	hipEvent_t sync;
	hipEventCreate(&sync);
	kernelGetNumCells<<<1,1>>>(ptrData_device, result_device);
    gpuErrchk( hipPeekAtLastError() );
	hipEventRecord(sync, 0);
	hipEventSynchronize(sync);

	CUDA_CHECK_RETURN(hipMemcpy(result, result_device, sizeof(type_t)*1, hipMemcpyDeviceToHost));

	hipFree( result_device );
	return result[0];
}


MonteRay_MaterialProperties::Material_Index_t MonteRay_MaterialProperties::launchGetNumMaterials( Cell_Index_t cellID ) const {
	typedef Material_Index_t type_t;

	type_t* result_device;
	type_t result[1];
	CUDA_CHECK_RETURN( hipMalloc( &result_device, sizeof( type_t) * 1 ));

	hipEvent_t sync;
	hipEventCreate(&sync);
	kernelGetNumMaterials<<<1,1>>>(ptrData_device, cellID, result_device);
    gpuErrchk( hipPeekAtLastError() );
	hipEventRecord(sync, 0);
	hipEventSynchronize(sync);

	CUDA_CHECK_RETURN(hipMemcpy(result, result_device, sizeof(type_t)*1, hipMemcpyDeviceToHost));

	hipFree( result_device );
	return result[0];
}


MonteRay_MaterialProperties::MatID_t MonteRay_MaterialProperties::launchGetMaterialID( Cell_Index_t cellID, Material_Index_t i ) const {
	typedef MatID_t type_t;

	type_t* result_device;
	type_t result[1];
	CUDA_CHECK_RETURN( hipMalloc( &result_device, sizeof( type_t) * 1 ));

	hipEvent_t sync;
	hipEventCreate(&sync);
	kernelGetMaterialID<<<1,1>>>(ptrData_device, cellID, i, result_device);
    gpuErrchk( hipPeekAtLastError() );
	hipEventRecord(sync, 0);
	hipEventSynchronize(sync);

	CUDA_CHECK_RETURN(hipMemcpy(result, result_device, sizeof(type_t)*1, hipMemcpyDeviceToHost));

	hipFree( result_device );
	return result[0];
}

MonteRay_MaterialProperties::Density_t MonteRay_MaterialProperties::launchGetMaterialDensity( Cell_Index_t cellID, Material_Index_t i ) const {
	typedef Density_t type_t;

	type_t* result_device;
	type_t result[1];
	CUDA_CHECK_RETURN( hipMalloc( &result_device, sizeof( type_t) * 1 ));

	hipEvent_t sync;
	hipEventCreate(&sync);
	kernelGetMaterialDensity<<<1,1>>>(ptrData_device, cellID, i, result_device);
    gpuErrchk( hipPeekAtLastError() );
	hipEventRecord(sync, 0);
	hipEventSynchronize(sync);

	CUDA_CHECK_RETURN(hipMemcpy(result, result_device, sizeof(type_t)*1, hipMemcpyDeviceToHost));

	hipFree( result_device );
	return result[0];
}


Density_t MonteRay_MaterialProperties::launchSumMatDensity(MatID_t matID) const{
	typedef Density_t type_t;

	type_t* result_device;
	type_t result[1];
	CUDA_CHECK_RETURN( hipMalloc( &result_device, sizeof( type_t) * 1 ));

	hipEvent_t sync;
	hipEventCreate(&sync);
	kernelSumMatDensity<<<1,1>>>(ptrData_device, matID, result_device);
    gpuErrchk( hipPeekAtLastError() );
	hipEventRecord(sync, 0);
	hipEventSynchronize(sync);

	CUDA_CHECK_RETURN(hipMemcpy(result, result_device, sizeof(type_t)*1, hipMemcpyDeviceToHost));

	hipFree( result_device );
	return result[0];
}


Density_t MonteRay_MaterialProperties::sumMatDensity( MatID_t matIndex) const {
	Density_t sum = 0.0f;
    for( unsigned cell=0; cell < size(); ++cell) {
         for( unsigned matNum=0; matNum < getNumMaterials(cell); ++matNum ) {

        	 Density_t density = getMaterialDensity(cell, matNum);
             MatID_t matID = getMaterialID(cell, matNum);

             if( matID == matIndex ) {
                 sum += density;
             }
         }
     }
     return sum;
}


}
