#include "hip/hip_runtime.h"
/*
 * MonteRaySphericalGrid.cc
 *
 *  Created on: Feb 2, 2018
 *      Author: jsweezy
 */

#include "MonteRayDefinitions.hh"
#include "MonteRay_SphericalGrid.t.hh"
#include "MonteRayConstants.hh"
#include "MonteRay_SingleValueCopyMemory.t.hh"
#include "MonteRayCopyMemory.t.hh"
#include "GPUErrorCheck.hh"

#include <float.h>

namespace MonteRay {

using ptrSphericalGrid_result_t = MonteRay_SingleValueCopyMemory<MonteRay_SphericalGrid*>;

CUDA_CALLABLE_KERNEL
void createDeviceInstance(MonteRay_SphericalGrid** pPtrInstance, ptrSphericalGrid_result_t* pResult, MonteRay_GridBins* pGridR ) {
    *pPtrInstance = new MonteRay_SphericalGrid( 1, pGridR );
    pResult->v = *pPtrInstance;
    //if( debug ) printf( "Debug: createDeviceInstance -- pPtrInstance = %d\n", pPtrInstance );
}

CUDA_CALLABLE_KERNEL
void deleteDeviceInstance(MonteRay_SphericalGrid** pPtrInstance) {
    delete *pPtrInstance;
}

CUDAHOST_CALLABLE_MEMBER
MonteRay_SphericalGrid*
MonteRay_SphericalGrid::getDeviceInstancePtr() {
    return devicePtr;
}

CUDA_CALLABLE_MEMBER
MonteRay_SphericalGrid::MonteRay_SphericalGrid(unsigned dim, pArrayOfpGridInfo_t pBins) :
MonteRay_GridSystemInterface(dim)
{
    MONTERAY_VERIFY( dim == DimMax, "MonteRay_SphericalGrid::ctor -- only 1-D is allowed" ); // No greater than 1-D.

    DIM = 1;
    pRVertices = pBins[0];
    validate();
}

CUDA_CALLABLE_MEMBER
MonteRay_SphericalGrid::MonteRay_SphericalGrid(unsigned dim, GridBins_t* pGridR ) :
MonteRay_GridSystemInterface(dim)
{
    MONTERAY_VERIFY( dim == DimMax, "MonteRay_SphericalGrid::ctor -- only 1-D is allowed" ); // No greater than 1-D.

    DIM = 1;
    pRVertices = pGridR;
    validate();
}

CUDA_CALLABLE_MEMBER
MonteRay_SphericalGrid::~MonteRay_SphericalGrid(void){
#ifdef __HIPCC__
#ifndef __CUDA_ARCH__
    if( ptrDevicePtr ) {
        deleteDeviceInstance<<<1,1>>>( ptrDevicePtr );
        hipDeviceSynchronize();
    }
    MonteRayDeviceFree( ptrDevicePtr );
#endif
#endif
}

CUDAHOST_CALLABLE_MEMBER
void
MonteRay_SphericalGrid::copyToGPU(void) {
    if( debug ) std::cout << "Debug: MonteRay_SphericalGrid::copyToGPU \n";
#ifdef __HIPCC__
    ptrDevicePtr = (MonteRay_SphericalGrid**) MONTERAYDEVICEALLOC(sizeof(MonteRay_SphericalGrid*), std::string("device - MonteRay_SphericalGrid::ptrDevicePtr") );

    pRVertices->copyToGPU();

    std::unique_ptr<ptrSphericalGrid_result_t> ptrResult = std::unique_ptr<ptrSphericalGrid_result_t>( new ptrSphericalGrid_result_t() );
    ptrResult->copyToGPU();

    createDeviceInstance<<<1,1>>>( ptrDevicePtr, ptrResult->devicePtr, pRVertices->devicePtr );
    hipDeviceSynchronize();
    ptrResult->copyToCPU();
    devicePtr = ptrResult->v;

#endif
}


CUDA_CALLABLE_MEMBER
void
MonteRay_SphericalGrid::validate(void) {
    validateR();
    numRBins = pRVertices->getNumBins();
}

CUDA_CALLABLE_MEMBER
void
MonteRay_SphericalGrid::validateR(void) {
    // Test for negative R
    for( int i=0; i<pRVertices->nVertices; ++i ){
        MONTERAY_VERIFY( pRVertices->vertices[i] >= 0.0, "MonteRay_SphericalGrid::validateR -- Can't have negative values for radius!!!" );
    }

    pRVertices->modifyForRadial();
}

CUDA_CALLABLE_MEMBER
MonteRay_SphericalGrid::Position_t
MonteRay_SphericalGrid::convertFromCartesian( const Position_t& pos) const {
    Position_t particleMeshPosition = {0.0, 0.0, 0.0};

    gpuRayFloat_t r = sqrt(pos[0]*pos[0] + pos[1]*pos[1] + pos[2]*pos[2]);
    particleMeshPosition[R] = r;

    return particleMeshPosition;
}


CUDA_CALLABLE_MEMBER
unsigned
MonteRay_SphericalGrid::getIndex( const Position_t& particle_pos) const{
    if( debug ) printf("Debug: MonteRay_SphericalGrid::getIndex -- starting\n");

    int index = 0;
    Position_t pos = convertFromCartesian( particle_pos );

    if( debug )  printf("%i\n", pRVertices->isRadial() );
    index = pRVertices->getRadialIndexFromR( pos[R] );
    if( isIndexOutside(R, index ) ) { return UINT_MAX; }

    return index;
}

CUDA_CALLABLE_MEMBER
bool
MonteRay_SphericalGrid::isIndexOutside( unsigned d,  int i) const {
    MONTERAY_VERIFY( d == 0, "MonteRay_SphericalGrid::isIndexOutside -- Index i must not be negative." );
    MONTERAY_VERIFY( d == 0, "MonteRay_SphericalGrid::isIndexOutside -- Dimension d must be 0 because spherical geometry is 1-D." );
    return pRVertices->isIndexOutside(i);
}

CUDA_CALLABLE_MEMBER
bool
MonteRay_SphericalGrid::isOutside( const int i[] ) const {
    if( isIndexOutside(R, i[R]) ) { return true; }
    return false;
}

CUDA_CALLABLE_MEMBER
unsigned
MonteRay_SphericalGrid::calcIndex( const int indices[] ) const{
    return indices[R];
}


CUDA_CALLABLE_MEMBER
gpuRayFloat_t
MonteRay_SphericalGrid::getVolume( unsigned index ) const {
    gpuRayFloat_t innerRadius = 0.0;
    if( index > 0 ){
        innerRadius = pRVertices->vertices[index-1];
    }
    gpuRayFloat_t outerRadius = pRVertices->vertices[index];

    gpuRayFloat_t volume = 4.0 * MonteRay::pi * ( std::pow(outerRadius,3) - std::pow(innerRadius,3) ) / 3.0 ;

    return volume;
}

CUDA_CALLABLE_MEMBER
void
MonteRay_SphericalGrid::rayTrace( rayTraceList_t& rayTraceList, const GridBins_t::Position_t& pos, const GridBins_t::Position_t& dir, gpuRayFloat_t distance,  bool outsideDistances/*=false*/) const {
    if( debug ) printf( "Debug: MonteRay_SphericalGrid::rayTrace -- \n");
    rayTraceList.reset();
    int indices[3] = {0, 0, 0}; // current position indices in the grid, must be int because can be outside

    multiDimRayTraceMap_t<1> distances;

    // Crossing distance in R direction
    {
        distances[R].reset();
        gpuRayFloat_t particleRSq = calcParticleRSq( pos );
        indices[R] = pRVertices->getRadialIndexFromRSq(particleRSq);

        if( debug ) printf( "Debug: MonteRay_SphericalGrid::rayTrace -- dimension=%d, index=%d\n", R, indices[R]);

        radialCrossingDistances( distances[R], pos, dir, indices[R], distance );

        if( debug ) printf( "Debug: MonteRay_SphericalGrid::rayTrace -- dimension=%d, number of radial crossings = %d\n", R, distances[R].size() );

        // if outside and ray doesn't move inside then ray never enters the grid
        if( isIndexOutside(R,indices[R]) && distances[R].size() == 0   ) {
            return;
        }
    }

    orderCrossings( rayTraceList, distances, indices, distance, outsideDistances );

    if( debug ) printf( "Debug: MonteRay_SphericalGrid::rayTrace -- number of total crossings = %d\n", rayTraceList.size() );
    return;
}

CUDA_CALLABLE_MEMBER
void
MonteRay_SphericalGrid::crossingDistance(singleDimRayTraceMap_t& rayTraceMap, const GridBins_t::Position_t& pos, const GridBins_t::Direction_t& dir, gpuRayFloat_t distance ) const {
    int index = pRVertices->getRadialIndexFromRSq(calcParticleRSq(pos));
    radialCrossingDistances( rayTraceMap, pos, dir, index, distance );
    return;
}

CUDA_CALLABLE_MEMBER
void
MonteRay_SphericalGrid::radialCrossingDistances(singleDimRayTraceMap_t& rayTraceMap, const Position_t& pos, const Direction_t& dir, unsigned rIndex, gpuRayFloat_t distance ) const {
    //------ Distance to Sphere's Radial-boundary
    if( debug ) {
        printf("Debug: MonteRay_SphericalGrid::radialCrossingDistances -- \n");
    }

    gpuRayFloat_t particleRSq = calcParticleRSq( pos );

    gpuRayFloat_t           A = calcQuadraticA( dir );
    gpuRayFloat_t           B = calcQuadraticB( pos, dir);

    // trace inward
    bool rayTerminated = radialCrossingDistanceSingleDirection<false>(rayTraceMap, *pRVertices, particleRSq, A, B, distance, rIndex);

    if( debug ) {
        printf("Debug: Inward ray trace size=%d\n",rayTraceMap.size());
        if( rayTerminated ) {
            printf("Debug: - ray terminated!\n");
        } else {
            printf("Debug: - ray not terminated!\n");
        }
    }

    // trace outward
    if( ! rayTerminated ) {
        if( !isIndexOutside(R, rIndex) ) {
            radialCrossingDistanceSingleDirection<true>(rayTraceMap, *pRVertices, particleRSq, A, B, distance, rIndex);
        } else {
            rayTraceMap.add(rIndex, distance);
        }
    }
}

CUDA_CALLABLE_MEMBER
void
MonteRay_SphericalGrid::radialCrossingDistances( singleDimRayTraceMap_t& rayTraceMap, const Position_t& pos, const Direction_t& dir, gpuRayFloat_t distance ) const {
    gpuRayFloat_t particleRSq = calcParticleRSq( pos );
    unsigned rIndex = pRVertices->getRadialIndexFromRSq(particleRSq);
    radialCrossingDistances( rayTraceMap, pos, dir, rIndex, distance );
}

template
CUDA_CALLABLE_MEMBER
void
MonteRay_SphericalGrid::radialCrossingDistancesSingleDirection<true>( singleDimRayTraceMap_t& rayTraceMap, const Position_t& pos, const Direction_t& dir, gpuRayFloat_t distance) const;

template
CUDA_CALLABLE_MEMBER
void
MonteRay_SphericalGrid::radialCrossingDistancesSingleDirection<false>( singleDimRayTraceMap_t& rayTraceMap, const Position_t& pos, const Direction_t& dir, gpuRayFloat_t distance) const;

} /* namespace MonteRay */
