#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <iostream>

#include "MonteRayDefinitions.hh"
#include "GPUErrorCheck.hh"

#include "ManagedMemory_test_helper.hh"


#ifdef __HIPCC__
__global__ void kernelSumVectors(testClass* A, testClass* B, testClass* C) {
    for( unsigned i=0; i<A->N; ++i) {
    	gpuFloatType_t elementA = A->elements[i] * A->multiple;
    	gpuFloatType_t elementB = B->elements[i] * B->multiple;
    	gpuFloatType_t elementC = elementA + elementB;
    	C->elements[i] = elementC;
    }
    C->N = A->N;
    C->multiple = 1.0;
    return;
}
#endif

void
ManagedMemoryTestHelper::launchSumVectors( testClass* A, testClass* B, testClass* C){
#ifdef __HIPCC__
	hipEvent_t sync;
	hipEventCreate(&sync);
	setupTimers();
	kernelSumVectors<<<1,1>>>( A, B, C);
	stopTimers();
	gpuErrchk( hipPeekAtLastError() );
	hipEventRecord(sync, 0);
	hipEventSynchronize(sync);
#endif
}

ManagedMemoryTestHelper::ManagedMemoryTestHelper(){
}

ManagedMemoryTestHelper::~ManagedMemoryTestHelper(){
}

void ManagedMemoryTestHelper::setupTimers(){
#ifdef __HIPCC__
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
#endif
}

void ManagedMemoryTestHelper::stopTimers(){
#ifdef __HIPCC__
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	float elapsedTime;

	hipEventElapsedTime(&elapsedTime, start, stop );

	std::cout << "Elapsed time in CUDA kernel=" << elapsedTime << " msec" << std::endl;
#endif
}
