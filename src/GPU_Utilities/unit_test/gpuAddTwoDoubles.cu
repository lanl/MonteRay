#include "hip/hip_runtime.h"
#include "gpuAddTwoDoubles.hh"
#include "GPUAtomicAdd.hh"
#include "GPUSync.hh"

namespace MonteRay {

#ifdef CUDA
__global__ void add_single(unsigned N, float *a, float *b, float *c ) {
	int bid = blockIdx.x;
	int tid = threadIdx.x;
	if( bid < N ) {
		if( tid == 0 ) {
			gpu_atomicAdd_single( &c[bid], a[bid] );
		} else if ( tid == 1 ) {
			gpu_atomicAdd_single( &c[bid], b[bid] );
		}
	}
}

__global__ void add_double(unsigned N, double *a, double *b, double *c ) {
	int bid = blockIdx.x;
	int tid = threadIdx.x;
	if( bid < N ) {
		if( tid == 0 ) {
			gpu_atomicAdd_double( &c[bid], a[bid] );
		} else if ( tid == 1 ) {
			gpu_atomicAdd_double( &c[bid], b[bid] );
		}
	}
}
#endif

/// Adds two doubles
double gpuAddTwoDoubles( double A, double B) {
	typedef double value_t;

	// Adds two doubles, but uses array notation
	value_t C;

	unsigned N=1;

	value_t a_host[N];
	value_t b_host[N];
	value_t c_host[N];

	unsigned allocSize = N*sizeof(value_t);

	a_host[0] = A;
	b_host[0] = B;
	c_host[0] = 0.0;

#ifdef CUDA
	value_t* pA_device;
	value_t* pB_device;
	value_t* pC_device;

	GPUSync sync;

	hipMalloc( &pA_device, allocSize );
	hipMalloc( &pB_device, allocSize );
	hipMalloc( &pC_device, allocSize );

	hipMemcpy( pA_device, a_host, allocSize, hipMemcpyHostToDevice);
	hipMemcpy( pB_device, b_host, allocSize, hipMemcpyHostToDevice);
	hipMemcpy( pC_device, c_host, allocSize, hipMemcpyHostToDevice);

	add_double<<<N,2>>>(N, pA_device, pB_device, pC_device );
	sync.sync();

	hipMemcpy( c_host, pC_device, allocSize, hipMemcpyDeviceToHost);
	C = c_host[0];

	hipFree( pA_device );
	hipFree( pB_device );
	hipFree( pC_device );
#else
	C = 0.0;
#endif

	return C;
}

/// Adds two floats
float gpuAddTwoFloats( float A, float B) {
	typedef float value_t;

	// Adds two doubles, but uses array notation
	value_t C;

	unsigned N=1;

	value_t a_host[N];
	value_t b_host[N];
	value_t c_host[N];

	unsigned allocSize = N*sizeof(value_t);

	a_host[0] = A;
	b_host[0] = B;
	c_host[0] = 0.0;

#ifdef CUDA
	value_t* pA_device;
	value_t* pB_device;
	value_t* pC_device;

	hipMalloc( &pA_device, allocSize );
	hipMalloc( &pB_device, allocSize );
	hipMalloc( &pC_device, allocSize );

	GPUSync sync;

	hipMemcpy( pA_device, a_host, allocSize, hipMemcpyHostToDevice);
	hipMemcpy( pB_device, b_host, allocSize, hipMemcpyHostToDevice);
	hipMemcpy( pC_device, c_host, allocSize, hipMemcpyHostToDevice);

	add_single<<<N,2>>>(N, pA_device, pB_device, pC_device );
	sync.sync();

	hipMemcpy( c_host, pC_device, allocSize, hipMemcpyDeviceToHost);
	C = c_host[0];

	hipFree( pA_device );
	hipFree( pB_device );
	hipFree( pC_device );
#else
	C = 0.0;
#endif

	return C;
}

}
