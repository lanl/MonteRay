#include <cstdio>
#include <unistd.h>
#include <iostream>

#include "GPUErrorCheck.hh"
#include "GPUUtilityFunctions.hh"
#include "MonteRayDefinitions.hh"

namespace MonteRay{

void cudaReset(const bool verbose) {
#ifdef __HIPCC__
    char hostname[1024];
    gethostname(hostname,1024);

    int deviceID = getCudaDevice(verbose);

    if(verbose) std::cout << "MonteRay::cudaReset -- " << hostname << ", device " << deviceID <<  " -- Reseting the GPU.\n";
    hipError_t error = hipDeviceReset();
    if( error != hipSuccess ) {
        std::cout << "MonteRay::cudaReset -- " << hostname << ", device=" << deviceID << " -- hipDeviceReset() call failed.\n";
        throw std::runtime_error ("MonteRay::cudaReset -- call to hipDeviceReset() failed.");
    }
#endif
}

void gpuReset(const bool verbose) {
    cudaReset(verbose);
}

void gpuCheck(const bool verbose) {
#ifdef __HIPCC__
    int deviceCount;
    char hostname[1024];
    gethostname(hostname,1024);

    if(verbose) std::cout << "MonteRay::gpuCheck -- " << hostname << " -- Initializing and checking GPU Status.\n";

    hipError_t result_error = hipInit(0);
    if( result_error != hipSuccess ) {
        std::cout << "MonteRay::gpuCheck -- " << hostname << " -- hipInit(0) call failed.\n";
        throw std::runtime_error ("MonteRay::gpuCheck -- call to hipInit(0) failed.");
    }

    result_error = hipGetDeviceCount(&deviceCount);
    if( result_error != hipSuccess ) {
        std::cout << "MonteRay::gpuCheck -- " << hostname << " -- hipGetDeviceCount() call failed.\n";
        throw std::runtime_error ("MonteRay::gpuCheck -- call to hipGetDeviceCount() failed.");
    } else {
        if(verbose) std::cout << "MonteRay::gpuCheck -- " << hostname << " -- hipGetDeviceCount() reported " << deviceCount << "GPU(s) on the host. \n";
    }
#endif
}

void gpuCheck() { gpuCheck(false); }

void gpuInfo() {
#ifdef __HIPCC__
    int deviceCount;

    hipError_t result_error = hipGetDeviceCount(&deviceCount);
    if( result_error != hipSuccess ) {
        printf("CUDA call: hipGetDeviceCount failed!\n");
        exit(1);
    }

    printf("Number of CUDA devices=%d\n",deviceCount);

    for (int i = 0; i < deviceCount; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        printf("Device Number: %d\n", i);
        printf("  Device name: %s\n", prop.name);
        printf("  Compute capability %d,%d\n", prop.major, prop.minor);
        printf("  Memory Clock Rate (KHz): %d\n",
                prop.memoryClockRate);
        printf("  Memory Bus Width (bits): %d\n",
                prop.memoryBusWidth);
        printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
                2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
    }
#endif
}

int getNumberOfGPUS(const bool verbose) {
    int count = 0;
    char hostname[1024];
    gethostname(hostname,1024);

    if(verbose) std::cout << "MonteRay::getNumberOfGPUS -- " << hostname << " -- Getting number of GPUs on the host.\n";

#ifdef __HIPCC__
    hipError_t error = hipGetDeviceCount( &count ) ;
    if(verbose) std::cout << "MonteRay::getNumberOfGPUS -- " << hostname << " -- Number of GPUs = " << count << ".\n";
    if( error != hipSuccess ) {
        std::cout << "MonteRay::getNumberOfGPUS -- " << hostname << " -- getNumberOfGPUS() call failed.\n";
        throw std::runtime_error ("MonteRay::getNumberOfGPUS -- call to hipGetDeviceCount() failed.");
    }

#endif
    return count;
}

void setCudaDevice(int id, const bool verbose ) {
    char hostname[1024];
    gethostname(hostname,1024);

    if(verbose) std::cout << "MonteRay::setCudaDevice -- " << hostname << " -- setting the cuda device, requested device id = " << id << "\n";

#ifdef __HIPCC__
    hipError_t error = hipSetDevice( id ) ;
    if( error != hipSuccess ) {
        std::cout << "MonteRay::setCudaDevice -- " << hostname << " -- hipSetDevice() call failed or device id = " << id << ".\n";
        throw std::runtime_error ("MonteRay::setCudaDevice -- call to hipSetDevice() failed.");
    }
#endif
}

void setCudaPrintBufferSize( size_t size, const bool verbose ) {
    char hostname[1024];
    gethostname(hostname,1024);

#ifdef __HIPCC__
    int device = getCudaDevice(verbose);
    hipError_t error = hipDeviceSetLimit(hipLimitPrintfFifoSize, size );
    if( error != hipSuccess ) {
        std::cout << "MonteRay::setCudaPrintBufferSize -- " << hostname << ", device=" << device << " -- hipDeviceSetLimit() call failed.\n";
        throw std::runtime_error ("MonteRay::setCudaPrintBufferSize -- call to hipDeviceSetLimit() failed.");
    }
#endif
}

int getCudaDevice( const bool verbose ) {
    int deviceID = 0;
    char hostname[1024];
    gethostname(hostname,1024);

    if(verbose) std::cout << "MonteRay::getCudaDevice -- " << hostname << " -- getting the current cuda device id...\n";
#ifdef __HIPCC__
    hipError_t error = hipGetDevice(&deviceID);
    if( error != hipSuccess ) {
        std::cout << "MonteRay::getCudaDevice -- " << hostname << " -- hipGetDevice() call failed.\n";
        throw std::runtime_error ("MonteRay::getCudaDevice -- call to hipGetDevice() failed.");
    }
#endif
    return deviceID;
}

void setCudaStackSize( size_t size, const bool verbose) {
    char hostname[1024];
    gethostname(hostname,1024);
    int device;

    if(verbose) std::cout << "MonteRay::setCudaDevice -- " << hostname << " -- setting the cuda stack size...\n";

#ifdef __HIPCC__
    device = getCudaDevice(verbose);

    if(verbose) std::cout << "MonteRay::setCudaDevice -- " << hostname << " -- setting the cuda stack size for device =" << device << " ...\n";
    hipError_t error = hipDeviceSetLimit( hipLimitStackSize, size );
    if( error != hipSuccess ) {
        std::cout << "MonteRay::setCudaStackSize -- " << hostname << ", device=" << device << " -- hipSetDevice() call failed.\n";
        throw std::runtime_error ("MonteRay::setCudaStackSize -- call to hipSetDevice() failed.");
    }
#endif
}

} /* namespace MonteRay */
