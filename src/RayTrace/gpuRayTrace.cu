#include "hip/hip_runtime.h"
#include "gpuRayTrace.hh"

#include <iostream>
#include <math.h>

#include "MonteRayDefinitions.hh"
#include "MonteRayConstants.hh"
#include "GridBins.hh"

#ifdef __HIPCC__
#include "hip/hip_math_constants.h"
#endif

namespace MonteRay{

//CUDA_CALLABLE_MEMBER unsigned
//cudaCalcCrossings(
//		const float_t* const vertices,
//		unsigned nVertices,
//		int* cells,
//		gpuRayFloat_t* distances,
//		gpuRayFloat_t pos,
//		gpuRayFloat_t dir,
//		gpuRayFloat_t distance,
//		int index )
//{
//	const bool debug = false;
//
//#ifdef __HIPCC__
//	constexpr gpuRayFloat_t minFloat = HIP_TWO_TO_M126_F;
//#else
//	constexpr gpuRayFloat_t minFloat = 1.175494351e-38f;
//#endif
//
//	unsigned nDistances = 0;
//
//	if( debug ) {
//		printf("cudaCalcCrossings:: Starting cudaCalcCrossings ******************\n");
//		printf("cudaCalcCrossings:: pos=%f\n", pos);
//		printf("cudaCalcCrossings:: dir=%f\n", dir);
//		printf("cudaCalcCrossings:: index=%d\n", index);
//	}
//
//	//if( abs(1/dir) >= HIP_NORM_HUGE_F )
//    if( abs(dir) <= minFloat )  // HIP_TWO_TO_M126_F    1.175494351e-38f
//    {
//    	return nDistances;
//    }
//
//    int start_index = index;
//    int cell_index = start_index;
//
//    if( start_index < 0 ) {
//        if( dir < 0.0 ) {
//            return nDistances;
//        }
//    }
//
//    int nBins = nVertices - 1;
//
//	if( debug ) {
//		printf("cudaCalcCrossings:: nBins=%d\n", nBins);
//	}
//
//    if( start_index >= nBins ) {
//        if( dir > 0.0 ) {
//        	return nDistances;
//        }
//    }
//
//    unsigned offset = 0;
//    if( dir > 0.0 ) {
//    	offset = 1;
//    }
//
//    int end_index = offset*(nBins-1);;
//
//    int dirIncrement = copysign( 1.0f, dir );
//
//    unsigned num_indices = abs(end_index - start_index ) + 1;
//
//    int current_index = start_index;
//
//    // Calculate boundary crossing distances
//    gpuRayFloat_t invDir = 1/dir;
//    bool rayTerminated = false;
//    for( int i = 0; i < num_indices ; ++i ) {
//
//    	gpuRayFloat_t minDistance = ( gpuRayFloat_t(vertices[current_index + offset]) - pos) * invDir;
//
//        if( minDistance >= distance ) {
//        	cells[nDistances] = cell_index;
//        	distances[nDistances] = distance;
//
//        	if( debug ) {
//        		printf("cudaCalcCrossings:: crossing num=%d, index=%d, distance=%f\n",
//        				nDistances,
//        				cells[nDistances],
//        				distances[nDistances]);
//        	}
//
//        	++nDistances;
//            rayTerminated = true;
//            break;
//        }
//
//        cells[nDistances] = cell_index;
//        distances[nDistances] = minDistance;
//
//    	if( debug ) {
//    		printf("cudaCalcCrossings:: crossing num=%d, index=%d, distance=%f\n",
//    				nDistances,
//    				cells[nDistances],
//    				distances[nDistances]);
//    	}
//
//        ++nDistances;
//
//        current_index += dirIncrement;
//        cell_index = current_index;
//    }
//
//    if( !rayTerminated ) {
//        // finish with distance into area outside
//    	cells[nDistances] = cell_index;
//    	distances[nDistances] = distance;
//
//    	if( debug ) {
//    		printf("cudaCalcCrossings:: crossing num=%d, index=%d, distance=%f\n",
//    				nDistances,
//    				cells[nDistances],
//    				distances[nDistances]);
//    	}
//
//    	++nDistances;
//        rayTerminated = true;
//    }
//
//    return nDistances;
//}
//
//CUDA_CALLABLE_MEMBER unsigned
//cudaOrderCrossings(
//		const GridBins* const grid,
//		int* global_indices,
//		gpuRayFloat_t* distances,
//		unsigned num,
//		const int* const cells,
//		const gpuRayFloat_t* const crossingDistances,
//		const uint3& numCrossings,
//		const int3& cudaindices,
//		gpuRayFloat_t distance,
//		bool outsideDistances )
//{
//    // Order the distance crossings to provide a rayTrace
//
//	const bool debug = false;
//
//#ifdef __HIPCC__
//	constexpr gpuRayFloat_t maxFloat = HIP_NORM_HUGE_F;
//#else
//	constexpr gpuRayFloat_t maxFloat = 3.402823466e38f;
//#endif
//
//
//	if( debug ) {
//		printf("cudaRayTrace:: Starting cudaOrderCrossings %%%%%%%%%%%%%%%%%%%%%%\n");
//		printf("cudaRayTrace:: cudaindices.x = %d, cudaindices.y = %d, cudaindices.z = %d,\n", cudaindices.x, cudaindices.y, cudaindices.z);
//	}
//
//
//	unsigned end[3] = { numCrossings.x, numCrossings.y, numCrossings.z}; //    last location in the distance[i] vector
//    int indices[3];
//    indices[0] = cudaindices.x; indices[1] = cudaindices.y; indices[2] = cudaindices.z;
//
//    int maxNumCrossings = numCrossings.x + numCrossings.y + numCrossings.z;
//    if( debug ) printf("cudaRayTrace::cudaOrderCrossings  maxNumCrossings = %d\n", maxNumCrossings);
//    gpuRayFloat_t minDistances[3];
//
//    bool outside;
//    gpuRayFloat_t priorDistance = 0.0;
//    unsigned start[3] = {0, 0, 0}; // current location in the distance[i] vector
//
//    unsigned numRayCrossings = 0;
//    for( unsigned i=0; i<maxNumCrossings; ++i){
//
//    	unsigned minDim;
//    	gpuRayFloat_t minimumDistance = maxFloat;
//        for( unsigned j = 0; j<3; ++j) {
//            if( start[j] < end[j] ) {
//            	minDistances[j] = *((crossingDistances+j*num)+start[j]);
//            	if( minDistances[j] < minimumDistance ) {
//            		minimumDistance = minDistances[j];
//            		minDim = j;
//            	}
//            } else {
//                minDistances[j] = maxFloat;
//            }
//        }
//        if( debug ) printf("cudaRayTrace::cudaOrderCrossings  crossing # %d, min dimension = %d, distance = %f\n", i, minDim, minimumDistance);
//
//        indices[minDim] =  *((cells+minDim*num) + start[minDim]);
//        if( debug ) printf("cudaRayTrace::cudaOrderCrossings  current indices: i = %d, j = %d, k = %d\n", indices[0], indices[1], indices[2]);
//
//        // test for outside of the grid
//        outside = cudaIsOutside(grid, indices );
//
//        if( debug ) {
//        	if( outside ) {
//        		printf("cudaRayTrace::cudaOrderCrossings  -- ray is outside the mesh\n");
//        	}
//        }
//
//        gpuRayFloat_t currentDistance = minimumDistance;
//
//        if( !outside || outsideDistances ) {
//        	gpuRayFloat_t deltaDistance = currentDistance - priorDistance;
//
//            if( deltaDistance > 0.0  ) {
//                unsigned global_index;
//                if( !outside ) {
//                    global_index = cudaCalcIndex(grid, indices );
//                } else {
//                    global_index = UINT_MAX;
//                }
//                global_indices[numRayCrossings] = global_index;
//                distances[numRayCrossings] = deltaDistance;
//
//            	if( debug ) {
//            		printf("cudaRayTrace:: crossing num=%d, index=%d, distance=%f\n", numRayCrossings,
//            				                                                          global_indices[numRayCrossings],
//            				                                                          distances[numRayCrossings]);
//            	}
//
//                ++numRayCrossings;
//            }
//        }
//
//        if( currentDistance >= distance ) {
//            break;
//        }
//
//        indices[minDim] = *((cells+minDim*num) + start[minDim]+1);
//
//        if( ! outside ) {
//            if( cudaIsIndexOutside(grid, minDim, indices[minDim] ) ) {
//                // ray has moved outside of grid
//                break;
//            }
//        }
//
//        ++start[minDim];
//        priorDistance = currentDistance;
//    }
//
//    return numRayCrossings;
//}
//
//CUDA_CALLABLE_MEMBER unsigned cudaRayTrace(const GridBins* const grid,
//		                         int* global_indices,
//		                         gpuRayFloat_t* distances,
//		                         const float3_t& pos,
//		                         const float3_t& dir,
//		                         gpuRayFloat_t distance,
//		                         bool outsideDistances)
//{
//	const bool debug = false;
//
//	if( debug ) {
//		printf("cudaRayTrace:: Starting cudaRayTrace ******************\n");
//	}
//
//	int3 current_indices;
//
//    int cells[3][MAXNUMVERTICES];
//    gpuRayFloat_t crossingDistances[3][MAXNUMVERTICES];
//    uint3 numCrossings;
//
//    current_indices.x = cudaGetDimIndex(grid, 0, pos.x );
//	numCrossings.x = cudaCalcCrossings( grid->vertices + grid->offset[0], grid->num[0]+1, cells[0], crossingDistances[0], pos.x, dir.x, distance, current_indices.x);
//
//	if( debug ) {
//		printf("cudaRayTrace:: current_indices.x =%d\n", current_indices.x );
//		printf("cudaRayTrace:: numCrossings.x =%d\n", numCrossings.x );
//	}
//
//	if( cudaIsIndexOutside(grid, 0, current_indices.x ) && numCrossings.x == 0  ) {return 0U;}
//
//    current_indices.y = cudaGetDimIndex(grid, 1, pos.y );
//	numCrossings.y = cudaCalcCrossings( grid->vertices + grid->offset[1], grid->num[1]+1, cells[1], crossingDistances[1], pos.y, dir.y, distance, current_indices.y);
//
//	if( debug ) {
//		printf("cudaRayTrace:: current_indices.y =%d\n", current_indices.y );
//		printf("cudaRayTrace:: numCrossings.y =%d\n", numCrossings.y );
//	}
//
//	if( cudaIsIndexOutside(grid, 1, current_indices.y ) && numCrossings.y == 0  ) {return 0U;}
//
//	current_indices.z = cudaGetDimIndex(grid, 2, pos.z );
//	numCrossings.z = cudaCalcCrossings( grid->vertices + grid->offset[2], grid->num[2]+1, cells[2], crossingDistances[2], pos.z, dir.z, distance, current_indices.z);
//
//	if( debug ) {
//		printf("cudaRayTrace:: current_indices.z =%d\n", current_indices.z );
//		printf("cudaRayTrace:: numCrossings.z =%d\n", numCrossings.z );
//	}
//
//	if( cudaIsIndexOutside(grid, 2, current_indices.z ) && numCrossings.z == 0  ) {return 0U;}
//
//    unsigned numRayCrossings = cudaOrderCrossings(grid, global_indices, distances, MAXNUMVERTICES, cells[0], crossingDistances[0], numCrossings, current_indices, distance, outsideDistances);
//
//	if( debug ) {
//		printf("cudaRayTrace:: numRayCrossings=%d\n", numRayCrossings );
//	}
//
//    return numRayCrossings;
//}
//
//CUDA_CALLABLE_MEMBER unsigned cudaRayTrace(const GridBins* const grid,
//		                         int* global_indices,
//		                         gpuRayFloat_t* distances,
//		                         const MonteRay::Vector3D<gpuRayFloat_t>& pos,
//		                         const MonteRay::Vector3D<gpuRayFloat_t>& dir,
//		                         gpuRayFloat_t distance,
//		                         bool outsideDistances)
//{
//	const bool debug = false;
//
//	if( debug ) {
//		printf("cudaRayTrace:: Starting cudaRayTrace ******************\n");
//	}
//
//	int3 current_indices;
//
//    int cells[3][MAXNUMVERTICES];
//    gpuRayFloat_t crossingDistances[3][MAXNUMVERTICES];
//    uint3 numCrossings;
//
//    current_indices.x = cudaGetDimIndex(grid, 0, pos[0] );
//	numCrossings.x = cudaCalcCrossings( grid->vertices + grid->offset[0], grid->num[0]+1, cells[0], crossingDistances[0], pos[0], dir[0], distance, current_indices.x);
//
//	if( debug ) {
//		printf("cudaRayTrace:: current_indices.x =%d\n", current_indices.x );
//		printf("cudaRayTrace:: numCrossings.x =%d\n", numCrossings.x );
//	}
//
//	if( cudaIsIndexOutside(grid, 0, current_indices.x ) && numCrossings.x == 0  ) {return 0U;}
//
//    current_indices.y = cudaGetDimIndex(grid, 1, pos[1] );
//	numCrossings.y = cudaCalcCrossings( grid->vertices + grid->offset[1], grid->num[1]+1, cells[1], crossingDistances[1], pos[1], dir[1], distance, current_indices.y);
//
//	if( debug ) {
//		printf("cudaRayTrace:: current_indices.y =%d\n", current_indices.y );
//		printf("cudaRayTrace:: numCrossings.y =%d\n", numCrossings.y );
//	}
//
//	if( cudaIsIndexOutside(grid, 1, current_indices.y ) && numCrossings.y == 0  ) {return 0U;}
//
//	current_indices.z = cudaGetDimIndex(grid, 2, pos[2] );
//	numCrossings.z = cudaCalcCrossings( grid->vertices + grid->offset[2], grid->num[2]+1, cells[2], crossingDistances[2], pos[2], dir[2], distance, current_indices.z);
//
//	if( debug ) {
//		printf("cudaRayTrace:: current_indices.z =%d\n", current_indices.z );
//		printf("cudaRayTrace:: numCrossings.z =%d\n", numCrossings.z );
//	}
//
//	if( cudaIsIndexOutside(grid, 2, current_indices.z ) && numCrossings.z == 0  ) {return 0U;}
//
//    unsigned numRayCrossings = cudaOrderCrossings(grid, global_indices, distances, MAXNUMVERTICES, cells[0], crossingDistances[0], numCrossings, current_indices, distance, outsideDistances);
//
//	if( debug ) {
//		printf("cudaRayTrace:: numRayCrossings=%d\n", numRayCrossings );
//	}
//
//    return numRayCrossings;
//}

CUDA_CALLABLE_KERNEL
void
kernelCudaRayTrace(void* ptrNumCrossings,
		GridBins* ptrGrid,
		int* ptrCells,
		gpuRayFloat_t* ptrDistances,
		gpuFloatType_t x, gpuFloatType_t y, gpuFloatType_t z,
		gpuFloatType_t u, gpuFloatType_t v, gpuFloatType_t w,
		gpuFloatType_t distance,
		bool outsideDistances) {

	const bool debug = false;

	if( debug ) {
		printf("kernelCudaRayTrace:: Starting kernelCudaRayTrace ******************\n");
	}

	unsigned* numCrossings = (unsigned*) ptrNumCrossings;

//	float3_t pos = make_float3( x, y, z);
//	float3_t dir = make_float3( u, v, w);
	Position_t pos( x, y, z );
	Direction_t dir( u, v, w );

	numCrossings[0] = ptrGrid->rayTrace( ptrCells, ptrDistances, pos, dir, distance, outsideDistances);

	if( debug ) {
		printf("kernelCudaRayTrace:: numCrossings=%d\n",numCrossings[0]);
	}
}


//CUDA_CALLABLE_KERNEL
//void
//kernelCudaRayTraceToAllCenters(
//		void* ptrGrid,
//		void* ptrDistances,
//		float_t x, float_t y, float_t z)
//{
//	const bool debug = false;
//
//	if( debug ) {
//		printf("kernelCudaRayTraceToAllCenters:: Starting kernelCudaRayTraceToAllCenters ******************\n");
//	}
//
//	GridBins* grid = (GridBins*) ptrGrid;
//	gpuRayFloat_t* distances = (gpuRayFloat_t*) ptrDistances;
//
//#ifdef __HIPCC__
//	int tid = threadIdx.x + blockIdx.x*blockDim.x;
//#else
//	int tid = 0;
//#endif
//
//	if( debug ) {
//		printf("kernelCudaRayTraceToAllCenters:: tid=%d\n", tid );
//#ifdef __HIPCC__
//		printf("kernelCudaRayTraceToAllCenters:: threadIdx.x=%d\n", threadIdx.x );
//		printf("kernelCudaRayTraceToAllCenters::  blockIdx.x=%d\n", blockIdx.x );
//		printf("kernelCudaRayTraceToAllCenters::  blockDim.x=%d\n", blockDim.x );
//#endif
//	}
//
//	int N = grid->numXY*grid->num[2];
//
//	if( debug ) {
//		printf("kernelCudaRayTraceToAllCenters:: Num Cells =%d\n", N );
//	}
//
//	uint3 indices;
//
//	//float3_t pos1 = make_float3( x, y, z);
//	MonteRay::Vector3D<gpuRayFloat_t> pos1( x, y, z );
//
//	//float3_t pos2;
//	MonteRay::Vector3D<gpuRayFloat_t> pos2;
//
//	//float3 dir;
//	MonteRay::Vector3D<gpuRayFloat_t> dir;
//
//	int cells[2*MAXNUMVERTICES];
//	gpuRayFloat_t crossingDistances[2*MAXNUMVERTICES];
//
//	while( tid < N ) {
//
//		if( debug ) {
//			printf("kernelCudaRayTraceToAllCenters:: tid=%d\n", tid );
//			printf("------------------------------------------" );
//		}
//
//		cudaCalcIJK(grid, tid, indices );
//		cudaGetCenterPointByIndices(grid, indices, pos2);
//
//	    dir = pos2 - pos1;
//	    gpuRayFloat_t length = dir.magnitude();
//
//		if( debug ) {
//			printf("kernelCudaRayTraceToAllCenters:: length=%f\n", length );
//		}
//
//		dir[0] = length / dir[0];
//		dir[1] = length / dir[1];
//		dir[2] = length / dir[2];
//
//
//		if( length > 0.0 ) {
////			dir.x /= length;
////			dir.y /= length;
////			dir.z /= length;
//
//			if( debug ) {
//				printf("kernelCudaRayTraceToAllCenters:: u=%f v=%f w=%f\n",dir[0],dir[1],dir[2]);
//			}
//
//			unsigned numCrossings;
//			numCrossings = cudaRayTrace( grid, cells, crossingDistances, pos1, dir, length, false);
//
//			if( debug ) {
//				printf("kernelCudaRayTraceToAllCenters:: numCrossings=%d\n",numCrossings );
//			}
////
////			if( debug ) {
////				printf("kernelCudaRayTraceToAllCenters:: Exiting\n" );
////			}
////			return;
//
//			gpuRayFloat_t length2 = 0.0f;
//			for( unsigned i=0; i < numCrossings; ++i){
//				length2 += crossingDistances[i];
//				//length2 += cells[i]*crossingDistances[i];
//			}
//			distances[tid] = length2;
//
//		} else {
//			distances[tid] = 0.0f;
//		}
//
//		if( debug ) {
//			printf("kernelCudaRayTraceToAllCenters:: distance=%f\n", distances[tid] );
//			printf("------------------------------------------" );
//		}
//
//#ifdef __HIPCC__
//		tid += blockDim.x*gridDim.x;
//#else
//		++tid;
//#endif
//		//if( tid >= 10 ) return;
//
//	}
//}

}
