#include "hip/hip_runtime.h"
#include "ExpectedPathLength.h"

#include <math.h>

#include "GridBins.h"
#include "GPUTiming.hh"
#include "MonteRayDefinitions.hh"
#include "GPUAtomicAdd.hh"

namespace MonteRay{

 template<unsigned N>
 __device__
 gpuTallyType_t
 tallyAttenuation(GridBins* pGrid, SimpleMaterialList* pMatList, MonteRay_MaterialProperties_Data* pMatProps, HashLookup* pHash, Ray_t<N>* p){


	 gpuTallyType_t enteringFraction = p->weight[0];
	 gpuFloatType_t energy = p->energy[0];
	 unsigned HashBin = getHashBin(pHash, energy);

	 if( energy < 1e-20 ) {
		 return enteringFraction;
	 }

	 int cells[2*MAXNUMVERTICES];
	 gpuFloatType_t crossingDistances[2*MAXNUMVERTICES];

	 unsigned numberOfCells;

	 float3_t pos = make_float3( p->pos[0], p->pos[1], p->pos[2]);
	 float3_t dir = make_float3( p->dir[0], p->dir[1], p->dir[2]);

	 numberOfCells = cudaRayTrace( pGrid, cells, crossingDistances, pos, dir, 1.0e6f, false);

	 for( unsigned i=0; i < numberOfCells; ++i ){
		 int cell = cells[i];
		 gpuFloatType_t distance = crossingDistances[i];
		 if( cell == UINT_MAX ) continue;

		 enteringFraction = attenuateRayTraceOnly(pMatList, pMatProps, pHash, HashBin, cell, distance, energy, enteringFraction );

		 if( enteringFraction < 1e-11 ) {
			 // cut off at 25 mean free paths
			 return enteringFraction;
		 }
	 }
	 return enteringFraction;
 }

 template __device__ gpuTallyType_t
 tallyAttenuation<1>(GridBins* pGrid, SimpleMaterialList* pMatList, MonteRay_MaterialProperties_Data* pMatProps, HashLookup* pHash, Ray_t<1>* p);

 template __device__ gpuTallyType_t
 tallyAttenuation<3>(GridBins* pGrid, SimpleMaterialList* pMatList, MonteRay_MaterialProperties_Data* pMatProps, HashLookup* pHash, Ray_t<3>* p);


 __device__
 gpuTallyType_t
 attenuateRayTraceOnly(SimpleMaterialList* pMatList, MonteRay_MaterialProperties_Data* pMatProps, HashLookup* pHash, unsigned HashBin, unsigned cell, gpuFloatType_t distance, gpuFloatType_t energy, gpuTallyType_t enteringFraction ) {

	 gpuTallyType_t totalXS = 0.0;
	 unsigned numMaterials = getNumMats( pMatProps, cell);
	 for( unsigned i=0; i<numMaterials; ++i ) {

		 unsigned matID = getMatID(pMatProps, cell, i);
		 gpuFloatType_t density = getDensity(pMatProps, cell, i );
		 if( density > 1e-5 ) {
			 //unsigned materialIndex = materialIDtoIndex(pMatList, matID);
			 totalXS +=  getTotalXS( pMatList, matID, pHash, HashBin, energy, density);
		 }
	 }

	 gpuTallyType_t attenuation = 1.0;

	 if( totalXS > 1e-5 ) {
		 attenuation = exp( - totalXS*distance );
	 }
	 return enteringFraction * attenuation;

 }

 template<unsigned N> __device__ void
 tallyCollision(GridBins* pGrid, SimpleMaterialList* pMatList, MonteRay_MaterialProperties_Data* pMatProps, HashLookup* pHash, Ray_t<N>* p, gpuTallyType_t* tally){

	 gpuTallyType_t opticalPathLength = 0.0;
	 gpuFloatType_t energy = p->energy[0];
	 unsigned HashBin = getHashBin(pHash, energy);

	 if( energy < 1e-20 ) {
		 return;
	 }

	int cells[2*MAXNUMVERTICES];
	gpuFloatType_t crossingDistances[2*MAXNUMVERTICES];

	unsigned numberOfCells;

	float3_t pos = make_float3( p->pos[0], p->pos[1], p->pos[2]);
	float3_t dir = make_float3( p->dir[0], p->dir[1], p->dir[2]);

	numberOfCells = cudaRayTrace( pGrid, cells, crossingDistances, pos, dir, 1.0e6f, false);

	gpuFloatType_t materialXS[MAXNUMMATERIALS];
	for( unsigned i=0; i < pMatList->numMaterials; ++i ){
		materialXS[i] = getTotalXS( pMatList, i, pHash, HashBin, energy, 1.0);
	}

	for( unsigned i=0; i < numberOfCells; ++i ){
		int cell = cells[i];
		gpuFloatType_t distance = crossingDistances[i];
		if( cell == UINT_MAX ) continue;

		opticalPathLength += tallyCellSegment(pMatList, pMatProps, materialXS, tally, cell, distance, energy, p->weight[0], opticalPathLength );

		if( opticalPathLength > 5.0 ) {
			// cut off at 5 mean free paths
			return;
		}
	}
}

 template __device__ void
 tallyCollision<1>(GridBins* pGrid, SimpleMaterialList* pMatList, MonteRay_MaterialProperties_Data* pMatProps, HashLookup* pHash, Ray_t<1>* p, gpuTallyType_t* tally);

 template __device__ void
 tallyCollision<3>(GridBins* pGrid, SimpleMaterialList* pMatList, MonteRay_MaterialProperties_Data* pMatProps, HashLookup* pHash, Ray_t<3>* p, gpuTallyType_t* tally);

__device__
gpuTallyType_t
tallyCellSegment(SimpleMaterialList* pMatList, MonteRay_MaterialProperties_Data* pMatProps, gpuFloatType_t* materialXS, gpuTallyType_t* tally, unsigned cell, gpuFloatType_t distance, gpuFloatType_t energy, gpuFloatType_t weight, gpuTallyType_t opticalPathLength ) {

	gpuTallyType_t totalXS = 0.0;
	unsigned numMaterials = getNumMats( pMatProps, cell);
	for( unsigned i=0; i<numMaterials; ++i ) {

		unsigned matID = getMatID(pMatProps, cell, i);
		gpuFloatType_t density = getDensity(pMatProps, cell, i );
		if( density > 1e-5 ) {
	       	//unsigned materialIndex = materialIDtoIndex(pMatList, matID);
			//totalXS +=   getTotalXS( pMatList, matID, energy, density);
			totalXS +=   materialXS[matID]*density;
		}
	}

	gpuTallyType_t attenuation = 1.0;
	gpuTallyType_t score = distance;
	gpuTallyType_t cellOpticalPathLength = totalXS*distance;

	if( totalXS >  1e-5 ) {
		attenuation =  exp( - cellOpticalPathLength );
		score = ( 1.0 / totalXS ) * ( 1.0 - attenuation );
	}
	score *= exp( -opticalPathLength ) * weight;

	gpu_atomicAdd( &tally[cell], score);

	return cellOpticalPathLength;
}

template<unsigned N> __global__ void
rayTraceTally(GridBins* pGrid, RayList_t<N>* pCP, SimpleMaterialList* pMatList,
		      MonteRay_MaterialProperties_Data* pMatProps, HashLookup* pHash,
		      gpuTallyType_t* tally){

	const bool debug = false;

	unsigned tid = threadIdx.x + blockIdx.x*blockDim.x;
	unsigned num = pCP->size();

	if( debug ) printf("GPU::rayTraceTally:: starting tid=%d  N=%d\n", tid, N );

	while( tid < num ) {
		Ray_t<N> p = pCP->getParticle(tid);
		tallyCollision(pGrid, pMatList, pMatProps, pHash, &p, tally);

		tid += blockDim.x*gridDim.x;
	}
	return;
}

template __global__ void
rayTraceTally<1>(GridBins* pGrid, RayList_t<1>* pCP, SimpleMaterialList* pMatList,
		         MonteRay_MaterialProperties_Data* pMatProps, HashLookup* pHash,
		         gpuTallyType_t* tally);

template __global__ void
rayTraceTally<3>(GridBins* pGrid, RayList_t<3>* pCP, SimpleMaterialList* pMatList,
		         MonteRay_MaterialProperties_Data* pMatProps, HashLookup* pHash,
		         gpuTallyType_t* tally);

template<unsigned N> __device__ void
tallyCollision(GridBins* pGrid, SimpleMaterialList* pMatList,
		            MonteRay_MaterialProperties_Data* pMatProps, HashLookup* pHash, Ray_t<N>* p,
		            gpuTally* pTally, unsigned tid)
{
	const bool debug = false;

	if( debug ) {
		printf("--------------------------------------------------------------------------------------------------------\n");
		printf("GPU::tallyCollision:: nCollisions=%d, x=%f, y=%f, z=%f, u=%f, v=%f, e=%f, w=%f, weight=%f, index=%d \n",
				tid+1,
				p->pos[0],
				p->pos[1],
				p->pos[2],
				p->dir[0],
				p->dir[1],
				p->dir[2],
				p->energy[0],
				p->weight[0],
				p->index
		);
	}

	typedef gpuTallyType_t enteringFraction_t;

	gpuTallyType_t opticalPathLength = 0.0;

	gpuFloatType_t energy = p->energy[0];
	unsigned HashBin = getHashBin(pHash, energy);

	if( energy < 1e-20 ) {
		return;
	}

	int cells[2*MAXNUMVERTICES];
	gpuFloatType_t crossingDistances[2*MAXNUMVERTICES];

	unsigned numberOfCells;

	float3_t pos = make_float3( p->pos[0], p->pos[1], p->pos[2]);
	float3_t dir = make_float3( p->dir[0], p->dir[1], p->dir[2]);

	numberOfCells = cudaRayTrace( pGrid, cells, crossingDistances, pos, dir, 1.0e6f, false);

	gpuFloatType_t materialXS[MAXNUMMATERIALS];
	for( unsigned i=0; i < pMatList->numMaterials; ++i ){
		materialXS[i] = getTotalXS( pMatList, i, pHash, HashBin, energy, 1.0);
	}

	for( unsigned i=0; i < numberOfCells; ++i ){
		int cell = cells[i];
		gpuFloatType_t distance = crossingDistances[i];
		if( cell == UINT_MAX ) continue;

		opticalPathLength += tallyCellSegment(pMatList, pMatProps, materialXS, pTally,
				                              cell, distance, energy, p->weight[0], opticalPathLength);

		if( opticalPathLength > 5.0 ) {
			// cut off at 5 mean free paths
			return;
		}
	}
}

template __device__ void
tallyCollision<1>(GridBins* pGrid, SimpleMaterialList* pMatList,
		            MonteRay_MaterialProperties_Data* pMatProps, HashLookup* pHash, Ray_t<1>* p,
		            gpuTally* pTally, unsigned tid);

template __device__ void
tallyCollision<3>(GridBins* pGrid, SimpleMaterialList* pMatList,
		            MonteRay_MaterialProperties_Data* pMatProps, HashLookup* pHash, Ray_t<3>* p,
		            gpuTally* pTally, unsigned tid);

__device__ gpuTallyType_t
tallyCellSegment(SimpleMaterialList* pMatList, MonteRay_MaterialProperties_Data* pMatProps,
		         gpuFloatType_t* materialXS , struct gpuTally* pTally, unsigned cell,
		         gpuFloatType_t distance, gpuFloatType_t energy, gpuFloatType_t weight,
		         gpuTallyType_t opticalPathLength ) {
	const bool debug = false;

	typedef gpuTallyType_t xs_t;
	typedef gpuTallyType_t attenuation_t;
	typedef gpuTallyType_t score_t;

	xs_t totalXS = 0.0;
	unsigned numMaterials = getNumMats( pMatProps, cell);
	if( debug ) {
		printf("GPU::tallyCellSegment:: cell=%d, numMaterials=%d\n", cell, numMaterials);
	}
	for( unsigned i=0; i<numMaterials; ++i ) {
		unsigned matID = getMatID(pMatProps, cell, i);
		gpuFloatType_t density = getDensity(pMatProps, cell, i );
        if( density > 1e-5 ) {
//        	totalXS +=   getTotalXS( pMatList, matID, energy, density);
//             totalXS +=   getTotalXS( pMatList, matID, pHash, HashBin, energy, density);
        	//unsigned materialIndex = materialIDtoIndex(pMatList, matID);
            totalXS +=   materialXS[matID]*density;
        }
//		if( debug ) {
//			printf("GPU::tallyCellSegment::       material=%d, density=%f, xs=%f, totalxs=%f\n", i, density, xs, totalXS);
//		}
	}

	attenuation_t attenuation = 1.0;
	score_t score = distance;
	gpuTallyType_t cellOpticalPathLength = totalXS*distance;

	if( totalXS >  1e-5 ) {
		attenuation =  exp( - cellOpticalPathLength );
		score = ( 1.0 / totalXS ) * ( 1.0 - attenuation );
	}
	score *= exp( -opticalPathLength ) * weight;

	if( debug ) {
		printf("GPU::tallyCellSegment:: cell=%d, distance=%f, totalXS=%f, score=%f\n", cell, distance, totalXS, score);
	}

	//atomicAdd( &(tally->tally[cell]), score);
	MonteRay::score( pTally, cell, score );

	if( debug ) {
		printf("GPU::tallyCellSegment:: total score=%f\n", pTally->tally[cell] );
	}

	return cellOpticalPathLength;
}

template<unsigned N> __global__ void
rayTraceTally(GridBins* pGrid, RayList_t<N>* pCP, SimpleMaterialList* pMatList,
		      MonteRay_MaterialProperties_Data* pMatProps, HashLookup* pHash, gpuTally* pTally ){

	const bool debug = false;

	unsigned tid = threadIdx.x + blockIdx.x*blockDim.x;

	unsigned num = pCP->size();

	if( debug ) printf("GPU::rayTraceTally:: starting tid=%d  N=%d\n", tid, N );

	while( tid < num ) {
		Ray_t<N> p = pCP->getParticle(tid);

		if( debug ) {
		    printf("--------------------------------------------------------------------------------------------------------\n");
            printf("GPU::rayTraceTally:: tid=%d\n", tid );
            printf("GPU::rayTraceTally:: x=%f\n", p.pos[0] );
            printf("GPU::rayTraceTally:: y=%f\n", p.pos[1] );
            printf("GPU::rayTraceTally:: z=%f\n", p.pos[2] );
            printf("GPU::rayTraceTally:: u=%f\n", p.dir[0] );
            printf("GPU::rayTraceTally:: v=%f\n", p.dir[1] );
            printf("GPU::rayTraceTally:: w=%f\n", p.dir[2] );
            printf("GPU::rayTraceTally:: energy=%f\n", p.energy[0] );
            printf("GPU::rayTraceTally:: weight=%f\n", p.weight[0] );
            printf("GPU::rayTraceTally:: index=%d\n", p.index );
		}

		tallyCollision(pGrid, pMatList, pMatProps, pHash, &p, pTally, tid);

		tid += blockDim.x*gridDim.x;
	}
	return;
}

template __global__ void
rayTraceTally<1>(GridBins* pGrid, RayList_t<1>* pCP, SimpleMaterialList* pMatList,
		         MonteRay_MaterialProperties_Data* pMatProps, HashLookup* pHash, gpuTally* pTally );

template __global__ void
rayTraceTally<3>(GridBins* pGrid, RayList_t<3>* pCP, SimpleMaterialList* pMatList,
		         MonteRay_MaterialProperties_Data* pMatProps, HashLookup* pHash, gpuTally* pTally );

template<unsigned N>
MonteRay::tripleTime launchRayTraceTally(
		                 std::function<void (void)> cpuWork,
		                 unsigned nBlocks,
		                 unsigned nThreads,
		                 GridBinsHost* pGrid,
		                 RayListInterface<N>* pCP,
		                 SimpleMaterialListHost* pMatList,
		                 MonteRay_MaterialProperties* pMatProps,
		                 gpuTallyHost* pTally
		                )
{
	MonteRay::tripleTime time;

	hipEvent_t startGPU, stopGPU, start, stop;

	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventCreate(&startGPU);
	hipEventCreate(&stopGPU);

	hipStream_t stream;
	hipStreamCreate( &stream );

	hipEventRecord(start,0);
	hipEventRecord(startGPU,stream);

	rayTraceTally<<<nBlocks,nThreads,0,stream>>>(pGrid->ptr_device, pCP->getPtrPoints()->devicePtr, pMatList->ptr_device, pMatProps->ptrData_device, pMatList->getHashPtr()->getPtrDevice(), pTally->ptr_device);
	hipEventRecord(stopGPU,stream);
	hipStreamWaitEvent(stream, stopGPU, 0);

	{
		MonteRay::cpuTimer timer;
		timer.start();
		cpuWork();
		timer.stop();
		time.cpuTime = timer.getTime();
	}

	hipStreamSynchronize( stream );
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipStreamDestroy(stream);

	float_t gpuTime;
	hipEventElapsedTime(&gpuTime, startGPU, stopGPU );
	time.gpuTime = gpuTime / 1000.0;

	float_t totalTime;
	hipEventElapsedTime(&totalTime, start, stop );
	time.totalTime = totalTime/1000.0;

	return time;
}

template MonteRay::tripleTime
launchRayTraceTally<1>( std::function<void (void)> cpuWork, unsigned nBlocks, unsigned nThreads,
		                GridBinsHost* pGrid, RayListInterface<1>* pCP, SimpleMaterialListHost* pMatList,
		                MonteRay_MaterialProperties* pMatProps, gpuTallyHost* pTally );

template MonteRay::tripleTime
launchRayTraceTally<3>( std::function<void (void)> cpuWork, unsigned nBlocks, unsigned nThreads,
		                GridBinsHost* pGrid, RayListInterface<3>* pCP, SimpleMaterialListHost* pMatList,
		                MonteRay_MaterialProperties* pMatProps, gpuTallyHost* pTally );

}


