#include "hip/hip_runtime.h"
#include "RayListController.hh"

#include <algorithm>

#include "GridBins.hh"
#include "MonteRayMaterialList.hh"
#include "MonteRay_MaterialProperties.hh"
#include "gpuTally.hh"
#include "RayListInterface.hh"
#include "ExpectedPathLength.hh"
#include "GPUErrorCheck.hh"
#include "GPUSync.hh"
#include "MonteRayNextEventEstimator.hh"

namespace MonteRay {

template<unsigned N >
RayListController<N>::RayListController(
		unsigned blocks,
        unsigned threads,
        GridBinsHost* pGB,
        MonteRayMaterialListHost* pML,
        MonteRay_MaterialProperties* pMP,
        gpuTallyHost* pT
	) :
        nBlocks(blocks),
        nThreads(threads),
        pGrid( pGB ),
        pMatList( pML ),
        pMatProps( pMP ),
        pTally(pT)
{
	pNextEventEstimator.reset();
	initialize();
	kernel = [&] ( void ) {
#ifdef __HIPCC__
		rayTraceTally<<<nBlocks,nThreads,0,stream1>>>(pGrid->ptr_device,
				currentBank->getPtrPoints()->devicePtr, pMatList->ptr_device,
				pMatProps->ptrData_device, pMatList->getHashPtr()->getPtrDevice(),
				pTally->temp->tally);
#else
		rayTraceTally(pGrid->getPtr(),
						currentBank->getPtrPoints(), pMatList->getPtr(),
						pMatProps->getPtr(), pMatList->getHashPtr()->getPtr(),
						pTally->getPtr()->tally );
#endif
	};

}

template<unsigned N >
RayListController<N>::RayListController(
		unsigned blocks,
        unsigned threads,
        GridBinsHost* pGB,
        MonteRayMaterialListHost* pML,
        MonteRay_MaterialProperties* pMP,
        unsigned numPointDets
	) :
        nBlocks(blocks),
        nThreads(threads),
        pGrid( pGB ),
        pMatList( pML ),
        pMatProps( pMP ),
        pTally(NULL)
{
	pNextEventEstimator = std::make_shared<MonteRayNextEventEstimator>( numPointDets );
	usingNextEventEstimator = true;
	initialize();
	kernel = [&] ( void ) {
		const bool debug = false;
		if( currentBank->size() > 0 ) {
			if( debug ) std::cout << "Debug: RayListController::kernel() -- Next Event Estimator kernel. Calling pNextEventEstimator->launch_ScoreRayList.\n";
#ifdef __HIPCC__
			pNextEventEstimator->launch_ScoreRayList(nBlocks,nThreads,stream1, currentBank->getPtrPoints());
#else
			pNextEventEstimator->launch_ScoreRayList(nBlocks,nThreads,currentBank->getPtrPoints());
#endif
		}
	};
}

template<unsigned N >
RayListController<N>::RayListController( unsigned numPointDets, std::string filename ) :
        nBlocks(0),
        nThreads(0),
        pGrid( NULL ),
        pMatList( NULL ),
        pMatProps( NULL ),
        pTally(NULL)
{
	initialize();
	pNextEventEstimator = std::make_shared<MonteRayNextEventEstimator>( numPointDets );
	setOutputFileName( filename );
	usingNextEventEstimator = true;
	kernel = [&] ( void ) {
		// do nothing
		return;
	};
}

template<unsigned N >
void
RayListController<N>::initialize(){
	nFlushs = 0;
	cpuTime = 0.0;
	gpuTime = 0.0;
	wallTime = 0.0;
	toFile = false;
	fileIsOpen = false;
	bank1 = new RayListInterface<N>(1000000); // default 1 millions
	bank2 = new RayListInterface<N>(1000000); // default 1 millions

#ifdef __HIPCC__
	hipStreamCreate( &stream1 );
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventCreate(&startGPU);
	hipEventCreate(&stopGPU);
	hipEventCreate(&copySync1);
	hipEventCreate(&copySync2);
#endif

	currentBank = bank1;

#ifdef __HIPCC__
	currentCopySync = &copySync1;
#endif
}

template<unsigned N >
RayListController<N>::~RayListController(){
	delete bank1;
	delete bank2;

#ifdef __CUDACC
	hipStreamDestroy(stream1);
#endif
}

template<unsigned N >
unsigned
RayListController<N>::capacity(void) const {
	return currentBank->capacity();
}

template<unsigned N >
unsigned
RayListController<N>::size(void) const {
	return currentBank->size();
}

template<unsigned N >
void
RayListController<N>::setCapacity(unsigned n) {
	delete bank1;
	delete bank2;
	bank1 = new RayListInterface<N>(n);
	bank2 = new RayListInterface<N>(n);
	currentBank = bank1;
}

template<unsigned N >
void
RayListController<N>::add( const Ray_t<N>& ray){
	currentBank->add( ray );
	if( size() == capacity() ) {
		std::cout << "Debug: bank full, flushing.\n";
		flush();
	}
}

template<unsigned N >
void
RayListController<N>::add( const Ray_t<N>* rayArray, unsigned num){
	int NSpaces = capacity() - size();

	int NAdding = std::min(NSpaces, int(num));
	int NRemaining = num - NAdding;
	currentBank->add( rayArray, NAdding );
	if( size() == capacity() ) {
		std::cout << "Debug: bank full, flushing.\n";
		flush();
	}
	if( NRemaining > 0 ) {
		add( rayArray + NAdding, NRemaining );
	}
}

template<unsigned N >
void
RayListController<N>::flush(bool final){
	const bool debug = false;
	if( debug ) std::cout << "Debug: RayListController<N>::flush\n";

	if( isSendingToFile() ) { flushToFile(final); }

	if( currentBank->size() == 0 ) {
		if( final ) {
			printTotalTime();
			currentBank->clear();
		}
		return;
	}

	if( nFlushs > 0 ) {
		std::cout << "Debug: flush nFlushs =" <<nFlushs-1 << " -- stopping timers\n";
		stopTimers();
	}
	std::cout << "Debug: flush nFlushs =" <<nFlushs << " -- starting timers\n";

	startTimers();

	++nFlushs;

#ifdef __HIPCC__
	currentBank->copyToGPU();
	gpuErrchk( hipEventRecord(*currentCopySync, 0) );
	gpuErrchk( hipEventSynchronize(*currentCopySync) );
#endif

	// launch kernel
	kernel();

	// only uncomment for testing, forces the cpu and gpu to sync
	//gpuErrchk( hipPeekAtLastError() );

#ifdef __HIPCC__
	gpuErrchk( hipEventRecord(stopGPU,stream1) );
	gpuErrchk( hipStreamWaitEvent(stream1, stopGPU, 0) );
#endif

	if( final ) {
		std::cout << "Debug: final flush nFlushs =" <<nFlushs-1 << " -- stopping timers\n";
		stopTimers();
		printTotalTime();
		currentBank->clear();
		return;
	}

	swapBanks();
}

template<unsigned N >
void
RayListController<N>::flushToFile(bool final){
	const bool debug = false;

	if( debug ) {
		if( final ) {
			std::cout << "Debug: RayListController::flushToFile - starting -- final = true \n";
		} else {
			std::cout << "Debug: RayListController::flushToFile - starting -- final = false \n";
		}
	}

	if( ! fileIsOpen ) {
		try {
			if( debug ) std::cout << "Debug: RayListController::flushToFile - opening file, filename=" << outputFileName << "\n";
			currentBank->openOutput( outputFileName );
		} catch ( ... ) {
	        std::stringstream msg;
	        msg << "Failure opening file for collision writing!\n";
	        msg << "Called from : " << __FILE__ << "[" << __LINE__ << "] : " << "RayListController::flushToFile" << "\n\n";
	        std::cout << "MonteRay Error: " << msg.str();
	        throw std::runtime_error( msg.str() );
		}

		fileIsOpen = true;
	}

	try {
		if( debug )  std::cout << "Debug: RayListController::flushToFile - writing bank -- bank size = "<< currentBank->size() << "\n";
		currentBank->writeBank();
	} catch ( ... ) {
        std::stringstream msg;
        msg << "Failure writing collisions to file!\n";
        msg << "Called from : " << __FILE__ << "[" << __LINE__ << "] : " << "RayListController::flushToFile" << "\n\n";
        std::cout << "MonteRay Error: " << msg.str();
        throw std::runtime_error( msg.str() );
	}

	currentBank->clear();

	if( final ) {
		try {
			if( debug ) std::cout << "Debug: RayListController::flushToFile - file flush, closing collision file\n";
			currentBank->closeOutput();
		} catch ( ... ) {
	        std::stringstream msg;
	        msg << "Failure closing collision file!\n";
	        msg << "Called from : " << __FILE__ << "[" << __LINE__ << "] : " <<"RayListController::flushToFile" << "\n\n";
	        std::cout << "MonteRay Error: " << msg.str();
	        throw std::runtime_error( msg.str() );
		}

		fileIsOpen = false;
	}
}

template<unsigned N >
size_t
RayListController<N>::readCollisionsFromFile(std::string name) {

	bool end = false;
	unsigned numParticles = 0;
	do  {
		end = currentBank->readToBank(name, numParticles);
		numParticles += currentBank->size();
		flush(end);
	} while ( ! end );
	return numParticles;
}

template<unsigned N >
void
RayListController<N>::startTimers(){
	// start timers
	timer.start();
#ifdef __HIPCC__
	gpuErrchk( hipEventRecord(start,0) );
	gpuErrchk( hipEventRecord(startGPU,stream1) );
#endif
}

template<unsigned N >
void
RayListController<N>::stopTimers(){
	// stop timers and sync

	timer.stop();
	float_t cpuCycleTime = timer.getTime();
	cpuTime += cpuCycleTime;

#ifdef __HIPCC__
	gpuErrchk( hipStreamSynchronize( stream1 ) );
	gpuErrchk( hipEventRecord(stop, 0) );
	gpuErrchk( hipEventSynchronize(stop) );

	float_t gpuCycleTime;
	gpuErrchk( hipEventElapsedTime(&gpuCycleTime, startGPU, stopGPU ) );
	gpuCycleTime /= 1000.0;
	if( gpuCycleTime < 0.0 ) {
		gpuCycleTime = 0.0;
	}
	gpuTime += gpuCycleTime;

	float totalCycleTime;
	gpuErrchk( hipEventElapsedTime(&totalCycleTime, start, stop ) );
	totalCycleTime /= 1000.0;
	wallTime += totalCycleTime;
	printCycleTime(cpuCycleTime, gpuCycleTime , totalCycleTime);
#else
	printCycleTime(cpuCycleTime, cpuCycleTime , cpuCycleTime);
#endif

}

template<unsigned N >
void
RayListController<N>::swapBanks(){
	// Swap banks
	if( currentBank == bank1 ) {
		currentBank = bank2;
#ifdef __HIPCC__
		currentCopySync = &copySync2;
#endif
	} else {
		currentBank = bank1;
#ifdef __HIPCC__
		currentCopySync = &copySync1;
#endif
	}

#ifdef __HIPCC__
	hipEventSynchronize(*currentCopySync);
#endif
	currentBank->clear();
}

template<unsigned N >
void
RayListController<N>::sync(void){
	GPUSync sync;
	sync.sync();
}

template<unsigned N >
void
RayListController<N>::clearTally(void) {

	std::cout << "Debug: clearTally called \n";

	if( nFlushs > 0 ) {
		stopTimers();
	}
//	std::cout << "Debug: clearTally nFlushs =" <<nFlushs << " -- starting timers\n";
//	startTimers();
//
//	++nFlushs;
//
//	hipEventRecord(stopGPU,stream1);
//	hipStreamWaitEvent(stream1, stopGPU, 0);

	GPUSync sync;
	pTally->clear();
	bank1->clear();
	bank2->clear();
	sync.sync();
}

template<unsigned N >
void
RayListController<N>::printTotalTime() const{
	std::cout << "Debug: \n";
	std::cout << "Debug: total gpuTime = " << gpuTime << "\n";
	std::cout << "Debug: total cpuTime = " << cpuTime << "\n";
	std::cout << "Debug: total wallTime = " << wallTime << "\n";
}

template<unsigned N >
void
RayListController<N>::printCycleTime(float_t cpu, float_t gpu, float_t wall) const{
	std::cout << "Debug: \n";
	std::cout << "Debug: cycle gpuTime = " << gpu << "\n";
	std::cout << "Debug: cycle cpuTime = " << cpu << "\n";
	std::cout << "Debug: cycle wallTime = " << wall << "\n";
}

template<unsigned N >
unsigned
RayListController<N>::addPointDet( gpuFloatType_t x, gpuFloatType_t y, gpuFloatType_t z ){
	if( ! isUsingNextEventEstimator() ) {
		throw std::runtime_error( "RayListController::addPointDet - Next Event Estimator not enabled." );
	}
	return pNextEventEstimator->add( x, y, z );
}

template<unsigned N >
void
RayListController<N>::setPointDetExclusionRadius(gpuFloatType_t r){
	if( ! isUsingNextEventEstimator() ) {
		throw std::runtime_error( "RayListController::setPointDetExclusionRadius - Next Event Estimator not enabled." );
	}
	pNextEventEstimator->setExclusionRadius( r );
}

template<unsigned N >
void
RayListController<N>::copyPointDetTallyToCPU(void) {
	if( ! isUsingNextEventEstimator() ) {
		throw std::runtime_error( "RayListController::copyPointDetTallyToCPU - Next Event Estimator not enabled." );
	}
	pNextEventEstimator->copyToCPU();
}

template<unsigned N >
gpuTallyType_t
RayListController<N>::getPointDetTally(unsigned i ) const {
	if( ! isUsingNextEventEstimator() ) {
		throw std::runtime_error( "RayListController::getPointDetTally - Next Event Estimator not enabled." );
	}
	return pNextEventEstimator->getTally(i);
}

template<unsigned N >
void
RayListController<N>::copyPointDetToGPU(void) {
	if( ! isUsingNextEventEstimator() ) {
		throw std::runtime_error( "RayListController::getPointDetTally - Next Event Estimator not enabled." );
	}

	pNextEventEstimator->setGeometry( pGrid, pMatProps );
	pNextEventEstimator->setMaterialList( pMatList );
	pNextEventEstimator->copyToGPU();
}

}
template class MonteRay::RayListController<1>;
template class MonteRay::RayListController<3>;
