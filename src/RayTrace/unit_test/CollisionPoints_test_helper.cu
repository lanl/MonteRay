#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "global.h"
#include "CollisionPoints.h"

#include "CollisionPoints_test_helper.hh"


#ifdef CUDA
__global__ void testGetCapacity(CollisionPoints* pXS, CollisionPointsSize_t* results){
	results[0] = capacity(pXS);
	return;
}
#endif

CollisionPointsSize_t
CollisionPointsTester::launchGetCapacity( unsigned nBlocks, unsigned nThreads, CollisionPointsHost& CPs) {
	CollisionPointsSize_t* result_device;
	CollisionPointsSize_t result[1];
	CUDA_CHECK_RETURN( hipMalloc( &result_device, sizeof( CollisionPointsSize_t) * 1 ));
	gpuErrchk( hipPeekAtLastError() );

	hipEvent_t sync;
	hipEventCreate(&sync);
    testGetCapacity<<<nBlocks,nThreads>>>(CPs.ptrPoints_device, result_device);
	hipEventRecord(sync, 0);
	hipEventSynchronize(sync);

    gpuErrchk( hipPeekAtLastError() );

	CUDA_CHECK_RETURN(hipMemcpy(result, result_device, sizeof(CollisionPointsSize_t)*1, hipMemcpyDeviceToHost));
	gpuErrchk( hipPeekAtLastError() );

	hipFree( result_device );
	CollisionPointsSize_t value = *result;
	return value;
}

#ifdef CUDA
__global__ void testSumEnergy(CollisionPoints* pXS, gpuFloatType_t* results){
	gpuFloatType_t total = 0.0f;
	for(unsigned i=0; i<size(pXS); ++i ) {
		total += getEnergy(pXS, i);
	}
	results[0] = total;
	return;
}
#endif

gpuFloatType_t
CollisionPointsTester::launchTestSumEnergy( unsigned nBlocks, unsigned nThreads, CollisionPointsHost& CPs) {
	gpuFloatType_t* result_device;
	gpuFloatType_t result[1];
	CUDA_CHECK_RETURN( hipMalloc( &result_device, sizeof( gpuFloatType_t) * 1 ));
	gpuErrchk( hipPeekAtLastError() );

	hipEvent_t sync;
	hipEventCreate(&sync);
	testSumEnergy<<<nBlocks,nThreads>>>(CPs.ptrPoints_device, result_device);
	hipEventRecord(sync, 0);
	hipEventSynchronize(sync);
	gpuErrchk( hipPeekAtLastError() );

	CUDA_CHECK_RETURN(hipMemcpy(result, result_device, sizeof(gpuFloatType_t)*1, hipMemcpyDeviceToHost));
	gpuErrchk( hipPeekAtLastError() );

	hipFree( result_device );
	return result[0];
}

CollisionPointsTester::CollisionPointsTester(){
	int deviceCount;

	hipInit(0);
	hipGetDeviceCount(&deviceCount);
	if (deviceCount == 0) {
		printf("No CUDA-compatible devices found\n");
		exit(1);
	}
	printf("Number of CUDA devices=%d\n",deviceCount);
	gpuErrchk( hipPeekAtLastError() );
}

CollisionPointsTester::~CollisionPointsTester(){
//		hipDeviceReset();
}

void CollisionPointsTester::setupTimers(){
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
}

void CollisionPointsTester::stopTimers(){
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	float elapsedTime;
	gpuErrchk( hipPeekAtLastError() );

	hipEventElapsedTime(&elapsedTime, start, stop );

	std::cout << "Elapsed time in CUDA kernel=" << elapsedTime << " msec" << std::endl;

	gpuErrchk( hipPeekAtLastError() );

}


