#include <hip/hip_runtime.h>
#include "global.h"
#include "gpuGlobal.h"

#include "gpuDistanceCalculator_test_helper.hh"
#include "cudaGridBins.h"

void
gpuDistanceCalculatorTestHelper::launchGetDistancesToAllCenters( unsigned nBlocks, unsigned nThreads, const Position_t& pos) {
	float_t x = pos[0];
	float_t y = pos[1];
	float_t z = pos[2];

	hipEvent_t sync;
	hipEventCreate(&sync);
	kernelGetDistancesToAllCenters<<<nBlocks,nThreads>>>(grid_device, distances_device, x, y, z);
	hipEventRecord(sync, 0);
	hipEventSynchronize(sync);
	gpuErrchk( hipPeekAtLastError() );
	return;
}

void
gpuDistanceCalculatorTestHelper::launchRayTrace( const Position_t& pos, const Direction_t& dir, float_t distance, bool outsideDistances) {

	hipEvent_t sync;
	hipEventCreate(&sync);
	kernelCudaRayTrace<<<1,1>>>(numCrossings_device,
			                                 grid_device,
			                                 cells_device,
			                                 distances_device,
			                                 pos[0], pos[1], pos[2],
			                                 dir[0], dir[1], dir[2],
			                                 distance,
			                                 outsideDistances );

	hipEventRecord(sync, 0);
	hipEventSynchronize(sync);
	gpuErrchk( hipPeekAtLastError() );
	return;
}

gpuDistanceCalculatorTestHelper::gpuDistanceCalculatorTestHelper(){
	grid_device = NULL;
	distances_device = NULL;
	cells_device = NULL;
	numCrossings_device = NULL;

	nCells = 0;

	int deviceCount;

	hipInit(0);
	hipGetDeviceCount(&deviceCount);
	if (deviceCount == 0) {
		printf("No CUDA-compatible devices found\n");
		exit(1);
	}
	printf("Number of CUDA devices=%d\n",deviceCount);
	gpuErrchk( hipPeekAtLastError() );
}

gpuDistanceCalculatorTestHelper::~gpuDistanceCalculatorTestHelper(){

//	std::cout << "Debug: starting ~gpuDistanceCalculatorTestHelper()" << std::endl;

	if( grid_device != NULL ) {
		hipFree( grid_device );
		gpuErrchk( hipPeekAtLastError() );
	}
	if( distances_device != NULL ) {
		hipFree( distances_device );
		gpuErrchk( hipPeekAtLastError() );
	}
	if( cells_device != NULL ) {
		hipFree( cells_device );
		gpuErrchk( hipPeekAtLastError() );
	}
	if( numCrossings_device != NULL ) {
		hipFree( numCrossings_device );
		gpuErrchk( hipPeekAtLastError() );
	}
//	std::cout << "Debug: exitting ~gpuDistanceCalculatorTestHelper()" << std::endl;
}

void gpuDistanceCalculatorTestHelper::copyGridtoGPU( GridBins* grid){
	// allocate and copy the grid
	CUDA_CHECK_RETURN( hipMalloc((void**) &grid_device, sizeof(GridBins) ));
	CUDA_CHECK_RETURN( hipMemcpy(grid_device, grid, sizeof(GridBins), hipMemcpyHostToDevice ));

	nCells = getNumCells(grid);

	// allocate the distances
	CUDA_CHECK_RETURN(hipMalloc((void**) &distances_device, sizeof(float_t) * nCells ));

	// allocate the cells
	CUDA_CHECK_RETURN(hipMalloc((void**) &cells_device, sizeof(int) * nCells ));

	// allocate the num crossings
	CUDA_CHECK_RETURN(hipMalloc((void**) &numCrossings_device, sizeof(unsigned) ));
}

void gpuDistanceCalculatorTestHelper::copyDistancesFromGPU( float_t* distances){
	// copy distances back to the host
	CUDA_CHECK_RETURN(hipMemcpy(distances, distances_device, sizeof(float_t) * nCells, hipMemcpyDeviceToHost));
}

void gpuDistanceCalculatorTestHelper::copyCellsFromCPU( int* cells){
	// copy cells back to the host
	CUDA_CHECK_RETURN(hipMemcpy(cells, cells_device, sizeof(int) * nCells, hipMemcpyDeviceToHost));
}

unsigned gpuDistanceCalculatorTestHelper::getNumCrossingsFromGPU( void ){
	// copy num crossings
	unsigned num;
	CUDA_CHECK_RETURN(hipMemcpy(&num, numCrossings_device, sizeof(unsigned) * 1, hipMemcpyDeviceToHost));
	return num;
}

void gpuDistanceCalculatorTestHelper::setupTimers(){
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
}

void gpuDistanceCalculatorTestHelper::stopTimers(){
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	float elapsedTime;
	gpuErrchk( hipPeekAtLastError() );

	hipEventElapsedTime(&elapsedTime, start, stop );

	std::cout << "Elapsed time in CUDA kernel=" << elapsedTime << " msec" << std::endl;

	gpuErrchk( hipPeekAtLastError() );

}


