#include <hip/hip_runtime.h>

#include "MonteRayDefinitions.hh"
#include "GPUErrorCheck.hh"
#include "GPUUtilityFunctions.hh"

#include "gpuDistanceCalculator_test_helper.hh"
#include "cudaGridBins.h"

namespace MonteRay{

void
gpuDistanceCalculatorTestHelper::launchGetDistancesToAllCenters( unsigned nBlocks, unsigned nThreads, const Position_t& pos) {
	float_t x = pos[0];
	float_t y = pos[1];
	float_t z = pos[2];

	hipEvent_t sync;
	hipEventCreate(&sync);
	kernelGetDistancesToAllCenters<<<nBlocks,nThreads>>>(grid_device, distances_device, x, y, z);
	gpuErrchk( hipPeekAtLastError() );
	hipEventRecord(sync, 0);
	hipEventSynchronize(sync);

	return;
}

void
gpuDistanceCalculatorTestHelper::launchRayTrace( const Position_t& pos, const Direction_t& dir, float_t distance, bool outsideDistances) {

	hipEvent_t sync;
	hipEventCreate(&sync);
	kernelCudaRayTrace<<<1,1>>>(numCrossings_device,
			                                 grid_device,
			                                 cells_device,
			                                 distances_device,
			                                 pos[0], pos[1], pos[2],
			                                 dir[0], dir[1], dir[2],
			                                 distance,
			                                 outsideDistances );
	gpuErrchk( hipPeekAtLastError() );

	hipEventRecord(sync, 0);
	hipEventSynchronize(sync);

	return;
}

gpuDistanceCalculatorTestHelper::gpuDistanceCalculatorTestHelper(){
	grid_device = NULL;
	distances_device = NULL;
	cells_device = NULL;
	numCrossings_device = NULL;

	nCells = 0;
}

void gpuDistanceCalculatorTestHelper::gpuCheck() {
	MonteRay::gpuCheck();
}

gpuDistanceCalculatorTestHelper::~gpuDistanceCalculatorTestHelper(){

//	std::cout << "Debug: starting ~gpuDistanceCalculatorTestHelper()" << std::endl;

	if( grid_device != NULL ) {
		CUDA_CHECK_RETURN( hipFree( grid_device ));
	}
	if( distances_device != NULL ) {
		CUDA_CHECK_RETURN( hipFree( distances_device ));
	}
	if( cells_device != NULL ) {
		CUDA_CHECK_RETURN( hipFree( cells_device ) );
	}
	if( numCrossings_device != NULL ) {
		CUDA_CHECK_RETURN( hipFree( numCrossings_device ) );
	}
//	std::cout << "Debug: exitting ~gpuDistanceCalculatorTestHelper()" << std::endl;
}

void gpuDistanceCalculatorTestHelper::copyGridtoGPU( GridBins* grid){
	// allocate and copy the grid
	CUDA_CHECK_RETURN( hipMalloc((void**) &grid_device, sizeof(GridBins) ));
	CUDA_CHECK_RETURN( hipMemcpy(grid_device, grid, sizeof(GridBins), hipMemcpyHostToDevice ));

	nCells = getNumCells(grid);

	// allocate the distances
	CUDA_CHECK_RETURN(hipMalloc((void**) &distances_device, sizeof(float_t) * nCells ));

	// allocate the cells
	CUDA_CHECK_RETURN(hipMalloc((void**) &cells_device, sizeof(int) * nCells ));

	// allocate the num crossings
	CUDA_CHECK_RETURN(hipMalloc((void**) &numCrossings_device, sizeof(unsigned) ));
}

void gpuDistanceCalculatorTestHelper::copyDistancesFromGPU( float_t* distances){
	// copy distances back to the host
	CUDA_CHECK_RETURN(hipMemcpy(distances, distances_device, sizeof(float_t) * nCells, hipMemcpyDeviceToHost));
}

void gpuDistanceCalculatorTestHelper::copyCellsFromCPU( int* cells){
	// copy cells back to the host
	CUDA_CHECK_RETURN(hipMemcpy(cells, cells_device, sizeof(int) * nCells, hipMemcpyDeviceToHost));
}

unsigned gpuDistanceCalculatorTestHelper::getNumCrossingsFromGPU( void ){
	// copy num crossings
	unsigned num;
	CUDA_CHECK_RETURN(hipMemcpy(&num, numCrossings_device, sizeof(unsigned) * 1, hipMemcpyDeviceToHost));
	return num;
}

void gpuDistanceCalculatorTestHelper::setupTimers(){
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
}

void gpuDistanceCalculatorTestHelper::stopTimers(){
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	float elapsedTime;

	hipEventElapsedTime(&elapsedTime, start, stop );

	std::cout << "Elapsed time in CUDA kernel=" << elapsedTime << " msec" << std::endl;
}

}
