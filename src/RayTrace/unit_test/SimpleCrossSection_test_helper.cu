#include <hip/hip_runtime.h>
#include "MonteRayDefinitions.hh"

#include "SimpleCrossSection_test_helper.hh"


//SimpleCrossSectionTestHelper::float_t
//SimpleCrossSectionTestHelper::launchGetTotalXS( SimpleCrossSectionHost* pXS, float_t energy){
//	float_t* result_device;
//	float_t result[1];
//	CUDA_CHECK_RETURN( hipMalloc( &result_device, sizeof( float_t) * 1 ));
//
//	hipEvent_t sync;
//	hipEventCreate(&sync);
//	kernelGetTotalXS<<<1,1>>>( pXS->xs_device, energy, result_device);
//  gpuErrchk( hipPeekAtLastError() );
//	hipEventRecord(sync, 0);
//	hipEventSynchronize(sync);
//
//	CUDA_CHECK_RETURN(hipMemcpy(result, result_device, sizeof(float_t)*1, hipMemcpyDeviceToHost));
//
//	hipFree( result_device );
//	return result[0];
//}

SimpleCrossSectionTestHelper::SimpleCrossSectionTestHelper(){
}

SimpleCrossSectionTestHelper::~SimpleCrossSectionTestHelper(){

//	std::cout << "Debug: starting ~SimpleCrossSectionTestHelper()" << std::endl;
//	std::cout << "Debug: exitting ~SimpleCrossSectionTestHelper()" << std::endl;
}

void SimpleCrossSectionTestHelper::setupTimers(){
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
}

void SimpleCrossSectionTestHelper::stopTimers(){
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	float elapsedTime;

	hipEventElapsedTime(&elapsedTime, start, stop );

	std::cout << "Elapsed time in CUDA kernel=" << elapsedTime << " msec" << std::endl;
}


