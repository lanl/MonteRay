#include "hip/hip_runtime.h"
#include "gpuTally_test_helper.hh"
#include "GPUErrorCheck.hh"

GPUTallyTestHelper::GPUTallyTestHelper(){
}

GPUTallyTestHelper::~GPUTallyTestHelper(){
}

__global__ void kernelAddTally(struct MonteRay::gpuTally* pTally, unsigned i, float_t a, float_t b){
    pTally->tally[i] =  a + b;
    return;
}

void GPUTallyTestHelper::launchAddTally( MonteRay::gpuTallyHost* tally, unsigned i, float_t a, float_t b ){

	tally->copyToGPU();

	hipEvent_t sync;
	hipEventCreate(&sync);
	kernelAddTally<<<1,1>>>( tally->ptr_device, i, a, b);
	gpuErrchk( hipPeekAtLastError() );
	hipEventRecord(sync, 0);
	hipEventSynchronize(sync);

    tally->copyToCPU();
}
