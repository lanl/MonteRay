#include "hip/hip_runtime.h"
#include "gpuTally_test_helper.hh"

GPUTallyTestHelper::GPUTallyTestHelper(){
}

GPUTallyTestHelper::~GPUTallyTestHelper(){
}

__global__ void kernelAddTally(struct MonteRay::gpuTally* pTally, unsigned i, float_t a, float_t b){
    pTally->tally[i] =  a + b;
    return;
}

void GPUTallyTestHelper::launchAddTally( MonteRay::gpuTallyHost* tally, unsigned i, float_t a, float_t b ){

	tally->copyToGPU();

	hipEvent_t sync;
	hipEventCreate(&sync);
	kernelAddTally<<<1,1>>>( tally->ptr_device, i, a, b);
	hipEventRecord(sync, 0);
	hipEventSynchronize(sync);

    gpuErrchk( hipPeekAtLastError() );

    tally->copyToCPU();
}
