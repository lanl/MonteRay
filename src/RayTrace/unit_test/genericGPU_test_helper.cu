#include <hip/hip_runtime.h>
#include "global.h"
#include "gpuGlobal.h"

#include "genericGPU_test_helper.hh"

GenericGPUTestHelper::GenericGPUTestHelper(){
	int deviceCount;

	hipInit(0);
	hipGetDeviceCount(&deviceCount);
	if (deviceCount == 0) {
		printf("No CUDA-compatible devices found\n");
		exit(1);
	}
	printf("Number of CUDA devices=%d\n",deviceCount);
	gpuErrchk( hipPeekAtLastError() );
}

GenericGPUTestHelper::~GenericGPUTestHelper(){
}

void GenericGPUTestHelper::setupTimers(){
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
}

void GenericGPUTestHelper::stopTimers(){
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	float elapsedTime;
	gpuErrchk( hipPeekAtLastError() );

	hipEventElapsedTime(&elapsedTime, start, stop );

	std::cout << "Elapsed time in CUDA kernel=" << elapsedTime << " msec" << std::endl;

	gpuErrchk( hipPeekAtLastError() );
}


