#include <hip/hip_runtime.h>
#include "global.h"
#include "gpuGlobal.h"

#include "genericGPU_test_helper.hh"

GenericGPUTestHelper::GenericGPUTestHelper(){
	int deviceCount;
}

GenericGPUTestHelper::~GenericGPUTestHelper(){
}

void GenericGPUTestHelper::setupTimers(){
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
}

void GenericGPUTestHelper::stopTimers(){
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	float elapsedTime;
	gpuErrchk( hipPeekAtLastError() );

	hipEventElapsedTime(&elapsedTime, start, stop );

	std::cout << "Elapsed time in CUDA kernel=" << elapsedTime << " msec" << std::endl;

	gpuErrchk( hipPeekAtLastError() );
}


