#include <iostream>

#include "MonteRayDefinitions.hh"

#include "genericGPU_test_helper.hh"

GenericGPUTestHelper::GenericGPUTestHelper(){}

GenericGPUTestHelper::~GenericGPUTestHelper(){}

void GenericGPUTestHelper::setupTimers(){
#ifdef __HIPCC__
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
#else
	timer.start();
#endif
}

void GenericGPUTestHelper::stopTimers(){
#ifdef __HIPCC__
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	float elapsedTime;

	hipEventElapsedTime(&elapsedTime, start, stop );
	std::cout << "Elapsed time in CUDA kernel=" << elapsedTime << " msec" << std::endl;
#else
	timer.stop();
	std::cout << "Elapsed time in non-CUDA kernel=" << timer.getTime()*1000 << " msec" << std::endl;
#endif
}


