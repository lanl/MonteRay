#include <hip/hip_runtime.h>

#include <iostream>

#include "MonteRayDefinitions.hh"

#include "genericGPU_test_helper.hh"


GenericGPUTestHelper::GenericGPUTestHelper(){}

GenericGPUTestHelper::~GenericGPUTestHelper(){}

void GenericGPUTestHelper::setupTimers(){
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
}

void GenericGPUTestHelper::stopTimers(){
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	float elapsedTime;

	hipEventElapsedTime(&elapsedTime, start, stop );

	std::cout << "Elapsed time in CUDA kernel=" << elapsedTime << " msec" << std::endl;
}


