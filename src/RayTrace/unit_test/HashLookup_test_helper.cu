#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "global.h"
#include "gpuGlobal.h"

#include "HashLookup_test_helper.hh"


#ifdef CUDA
__global__ void kernelGetLowerBoundbyIndex(HashLookup* pHash, unsigned isotope, unsigned bin, unsigned* result){
    result[0] = getLowerBoundbyIndex( pHash, isotope, bin);
    return;
}
#endif

unsigned
HashLookupTestHelper::launchGetLowerBoundbyIndex( HashLookupHost* pHash, unsigned isotope, unsigned bin){
	unsigned* result_device;
	unsigned result[1];
	CUDA_CHECK_RETURN( hipMalloc( &result_device, sizeof( unsigned) * 1 ));
	gpuErrchk( hipPeekAtLastError() );

	hipEvent_t sync;
	hipEventCreate(&sync);
	kernelGetLowerBoundbyIndex<<<1,1>>>( pHash->ptr_device, isotope, bin, result_device);
	hipEventRecord(sync, 0);
	hipEventSynchronize(sync);

    gpuErrchk( hipPeekAtLastError() );

	CUDA_CHECK_RETURN(hipMemcpy(result, result_device, sizeof(unsigned)*1, hipMemcpyDeviceToHost));
	gpuErrchk( hipPeekAtLastError() );

	hipFree( result_device );
	return result[0];
}

HashLookupTestHelper::HashLookupTestHelper(){
}

HashLookupTestHelper::~HashLookupTestHelper(){

//	std::cout << "Debug: starting ~SimpleCrossSectionTestHelper()" << std::endl;
//	std::cout << "Debug: exitting ~SimpleCrossSectionTestHelper()" << std::endl;
}

void HashLookupTestHelper::setupTimers(){
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
}

void HashLookupTestHelper::stopTimers(){
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	float elapsedTime;
	gpuErrchk( hipPeekAtLastError() );

	hipEventElapsedTime(&elapsedTime, start, stop );

	std::cout << "Elapsed time in CUDA kernel=" << elapsedTime << " msec" << std::endl;

	gpuErrchk( hipPeekAtLastError() );

}


