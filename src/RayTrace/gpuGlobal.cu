#include "gpuGlobal.h"

void cudaReset(void) {
#ifdef CUDA
	hipDeviceReset();
	gpuErrchk( hipPeekAtLastError() );
#endif
}
