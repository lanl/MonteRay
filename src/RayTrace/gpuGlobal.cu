#include "gpuGlobal.h"

namespace MonteRay{

void cudaReset(void) {
#ifdef CUDA
	hipDeviceReset();
	gpuErrchk( hipPeekAtLastError() );
#endif
}

void gpuCheck() {
	int deviceCount;

	hipInit(0);
	hipGetDeviceCount(&deviceCount);
	if (deviceCount == 0) {
		printf("No CUDA-compatible devices found\n");
		exit(1);
	}
	printf("Number of CUDA devices=%d\n",deviceCount);
	gpuErrchk( hipPeekAtLastError() );
}

gpuSync::gpuSync(){
	hipEventCreate(&sync_event);
}

gpuSync::~gpuSync(){
	hipEventDestroy(sync_event);
}

void gpuSync::sync(){
	hipEventRecord(sync_event, 0);
	hipEventSynchronize(sync_event);
}

void setCudaPrintBufferSize( size_t size) {
#ifdef CUDA
	hipDeviceSetLimit(hipLimitPrintfFifoSize, size );
#endif
}

}
