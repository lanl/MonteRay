#include <hip/hip_runtime.h>

#include "MonteRayDefinitions.hh"
#include "GPUErrorCheck.hh"
#include "GPUAtomicAdd.hh"
#include "ExpectedPathLength.h"

#include "RayTraceNightly_genericGPU_test_helper.hh"

template<unsigned N>
FIGenericGPUTestHelper<N>::FIGenericGPUTestHelper(unsigned num){
}

template<unsigned N>
FIGenericGPUTestHelper<N>::~FIGenericGPUTestHelper(){
}

template<unsigned N>
void FIGenericGPUTestHelper<N>::setupTimers(){
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
}

template<unsigned N>
void FIGenericGPUTestHelper<N>::stopTimers(){
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	float elapsedTime;
	gpuErrchk( hipPeekAtLastError() );

	hipEventElapsedTime(&elapsedTime, start, stop );

	std::cout << "Elapsed time in CUDA kernel=" << elapsedTime << " msec" << std::endl;

	gpuErrchk( hipPeekAtLastError() );
}

template class FIGenericGPUTestHelper<1>;
template class FIGenericGPUTestHelper<3>;

