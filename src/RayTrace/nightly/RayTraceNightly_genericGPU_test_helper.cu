#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include "MonteRayDefinitions.hh"
#include "GPUErrorCheck.hh"
#include "GPUAtomicAdd.hh"
#include "ExpectedPathLength.h"

#include "RayTraceNightly_genericGPU_test_helper.hh"


FIGenericGPUTestHelper::FIGenericGPUTestHelper(unsigned num){
	nCells = num;
	tally = NULL;
	grid_device = NULL;
}

FIGenericGPUTestHelper::~FIGenericGPUTestHelper(){
	if( tally != NULL ) {
		free( tally );
	}
	if( grid_device != NULL ) {
		hipFree( grid_device );
	}
}

void FIGenericGPUTestHelper::setupTimers(){
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
}

void FIGenericGPUTestHelper::stopTimers(){
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	float elapsedTime;
	gpuErrchk( hipPeekAtLastError() );

	hipEventElapsedTime(&elapsedTime, start, stop );

	std::cout << "Elapsed time in CUDA kernel=" << elapsedTime << " msec" << std::endl;

	gpuErrchk( hipPeekAtLastError() );
}

#ifdef __HIPCC__
__global__ void testTallyCrossSection(CollisionPoints* pCP, MonteRayCrossSection* pXS, gpuTallyType_t* results){

	int tid = threadIdx.x + blockIdx.x*blockDim.x;
	int N = pCP->size();
	while( tid < N ) {
		gpuFloatType_t E = pCP->getEnergy(tid);
		results[tid] = getTotalXS(pXS, E);
		tid += blockDim.x*gridDim.x;
	}
	return;
}
#endif

void FIGenericGPUTestHelper::launchTallyCrossSection(unsigned nBlocks, unsigned nThreads, CollisionPointsHost* pCP, MonteRayCrossSectionHost* pXS ){
	gpuTallyType_t* tally_device;

	unsigned long long allocSize = sizeof(gpuTallyType_t)*nCells;
	tally = (gpuTallyType_t*) malloc ( allocSize );
	CUDA_CHECK_RETURN( hipMalloc( &tally_device, allocSize ));
	CUDA_CHECK_RETURN( hipMemset(tally_device, 0, allocSize));
	gpuErrchk( hipPeekAtLastError() );

	hipEvent_t sync;
	hipEventCreate(&sync);
	testTallyCrossSection<<<nBlocks,nThreads>>>(pCP->ptrPoints_device, pXS->xs_device, tally_device);
	hipEventRecord(sync, 0);
	hipEventSynchronize(sync);
	gpuErrchk( hipPeekAtLastError() );

	CUDA_CHECK_RETURN(hipMemcpy(tally, tally_device, allocSize, hipMemcpyDeviceToHost));
	gpuErrchk( hipPeekAtLastError() );
	hipFree( tally_device );
	return;
}

#ifdef CUDA
__global__ void testTallyCrossSection(CollisionPoints* pCP, SimpleMaterialList* pMatList, unsigned matIndex, HashLookup* pHash, gpuFloatType_t density, gpuTallyType_t* results){

	int tid = threadIdx.x + blockIdx.x*blockDim.x;
	int N = pCP->size();
	while( tid < N ) {
		gpuFloatType_t E = pCP->getEnergy(tid);
		unsigned HashBin = getHashBin( pHash, E);
		results[tid] = getTotalXS(pMatList, matIndex, pHash, HashBin, E, density);
		tid += blockDim.x*gridDim.x;
	}
	return;
}
#endif

void FIGenericGPUTestHelper::launchTallyCrossSection(unsigned nBlocks, unsigned nThreads, CollisionPointsHost* pCP, SimpleMaterialListHost* pMatList, unsigned matIndex, gpuFloatType_t density ){
	gpuTallyType_t* tally_device;

	unsigned long long allocSize = sizeof(gpuTallyType_t)*nCells;
	tally = (gpuTallyType_t*) malloc ( allocSize );
	CUDA_CHECK_RETURN( hipMalloc( &tally_device, allocSize ));
	CUDA_CHECK_RETURN( hipMemset(tally_device, 0, allocSize));
	gpuErrchk( hipPeekAtLastError() );

	hipEvent_t sync;
	hipEventCreate(&sync);
	testTallyCrossSection<<<nBlocks,nThreads>>>(pCP->ptrPoints_device, pMatList->ptr_device, matIndex, pMatList->getHashPtr()->getPtrDevice(), density, tally_device);
	hipEventRecord(sync, 0);
	hipEventSynchronize(sync);
	gpuErrchk( hipPeekAtLastError() );

	CUDA_CHECK_RETURN(hipMemcpy(tally, tally_device, allocSize, hipMemcpyDeviceToHost));
	gpuErrchk( hipPeekAtLastError() );
	hipFree( tally_device );
	return;
}

#ifdef CUDA
__device__ __host__
gpuFloatType_t getTotalXSByMatProp(MonteRay_MaterialProperties_Data* matProps, SimpleMaterialList* pMatList, HashLookup* pHash, unsigned HashBin, unsigned cell, gpuFloatType_t E) {
	gpuFloatType_t total = 0.0f;
	for( unsigned i=0; i< getNumMats(matProps,cell); ++i) {
		gpuFloatType_t density = getDensity(matProps,cell,i);
		unsigned matID = getMatID(matProps,cell,i);
//		unsigned materialIndex = materialIDtoIndex(pMatList, matID);
		total += getTotalXS(pMatList, matID, pHash, HashBin, E, density);
	}
	return total;
}
#endif

gpuFloatType_t nonCudaGetTotalXSByMatProp(MonteRay_MaterialProperties* matProps, SimpleMaterialList* pMatList, HashLookup* pHash, unsigned HashBin, unsigned cell, gpuFloatType_t E) {
	gpuFloatType_t total = 0.0f;
	for( unsigned i=0; i< matProps->getNumMaterials(cell); ++i) {
		gpuFloatType_t density = matProps->getMaterialDensity(cell,i);
		MonteRay_MaterialProperties::MatID_t matID = matProps->getMaterialID(cell,i);
		//unsigned materialIndex = materialIDtoIndex(pMatList, matID);
		total += getTotalXS(pMatList, matID, pHash, HashBin, E, density);
	}
	return total;
}

gpuFloatType_t nonCudaGetTotalXSByMatProp(MonteRay_MaterialProperties* matProps, SimpleMaterialList* pMatList, unsigned cell, gpuFloatType_t E) {
	gpuFloatType_t total = 0.0f;
	for( unsigned i=0; i< matProps->getNumMaterials(cell); ++i) {
		gpuFloatType_t density = matProps->getMaterialDensity(cell,i);
		unsigned matID = matProps->getMaterialID(cell,i);
		//unsigned materialIndex = materialIDtoIndex(pMatList, matID);
		total += getTotalXS(pMatList, matID, E, density);
	}
	return total;
}

gpuFloatType_t FIGenericGPUTestHelper::getTotalXSByMatProp(MonteRay_MaterialProperties* matProps, SimpleMaterialList* pMatList, HashLookup* pHash, unsigned HashBin, unsigned cell, gpuFloatType_t E) {
	return nonCudaGetTotalXSByMatProp( matProps, pMatList, pHash, HashBin, cell, E);
}

gpuFloatType_t FIGenericGPUTestHelper::getTotalXSByMatProp(MonteRay_MaterialProperties* matProps, SimpleMaterialList* pMatList, unsigned cell, gpuFloatType_t E) {
	return nonCudaGetTotalXSByMatProp( matProps, pMatList, cell, E);
}

#ifdef CUDA
__global__ void testTallyCrossSectionAtCollision(CollisionPoints* pCP, SimpleMaterialList* pMatList, MonteRay_MaterialProperties_Data* pMatProps, HashLookup* pHash, gpuTallyType_t* results){

	unsigned tid = threadIdx.x + blockIdx.x*blockDim.x;
	unsigned N = pCP->size();

	while( tid < N ) {
		gpuFloatType_t E = pCP->getEnergy(tid);
		unsigned HashBin = getHashBin( pHash, E);
		unsigned cell = pCP->getIndex(tid);

		results[tid] = getTotalXSByMatProp(pMatProps, pMatList, pHash, HashBin, cell, E);
		tid += blockDim.x*gridDim.x;
	}
	return;
}
#endif

#ifdef CUDA
__global__ void testSumCrossSectionAtCollisionLocation(CollisionPoints* pCP, SimpleMaterialList* pMatList, MonteRay_MaterialProperties_Data* pMatProps, HashLookup* pHash, gpuTallyType_t* results){

	unsigned tid = threadIdx.x + blockIdx.x*blockDim.x;
	unsigned N = pCP->size();

	while( tid < N ) {
		gpuFloatType_t E = pCP->getEnergy(tid);
		unsigned HashBin = getHashBin( pHash, E);
		unsigned cell = pCP->getIndex(tid);

		gpuTallyType_t value = getTotalXSByMatProp(pMatProps, pMatList, pHash, HashBin, cell, E);

		gpu_atomicAdd( &results[cell], value);
		tid += blockDim.x*gridDim.x;
	}
	return;
}
#endif

void FIGenericGPUTestHelper::launchTallyCrossSectionAtCollision(unsigned nBlocks, unsigned nThreads, CollisionPointsHost* pCP, SimpleMaterialListHost* pMatList, MonteRay_MaterialProperties* pMatProps ){
	gpuTallyType_t* tally_device;

	unsigned long long allocSize = sizeof(gpuTallyType_t)*nCells;
	tally = (gpuTallyType_t*) malloc ( allocSize );
	CUDA_CHECK_RETURN( hipMalloc( &tally_device, allocSize ));
	CUDA_CHECK_RETURN( hipMemset(tally_device, 0, allocSize));
	gpuErrchk( hipPeekAtLastError() );

	hipEvent_t sync;
	hipEventCreate(&sync);
	testTallyCrossSectionAtCollision<<<nBlocks,nThreads>>>(pCP->ptrPoints_device, pMatList->ptr_device, pMatProps->ptrData_device, pMatList->getHashPtr()->getPtrDevice(), tally_device);
	hipEventRecord(sync, 0);
	hipEventSynchronize(sync);
	gpuErrchk( hipPeekAtLastError() );

	CUDA_CHECK_RETURN(hipMemcpy(tally, tally_device, allocSize, hipMemcpyDeviceToHost));
	gpuErrchk( hipPeekAtLastError() );
	hipFree( tally_device );
	return;
}

void FIGenericGPUTestHelper::launchSumCrossSectionAtCollisionLocation(unsigned nBlocks, unsigned nThreads, CollisionPointsHost* pCP, SimpleMaterialListHost* pMatList, MonteRay_MaterialProperties* pMatProps ){
	gpuTallyType_t* tally_device;

	unsigned long long allocSize = sizeof(gpuTallyType_t)*nCells;
	tally = (gpuTallyType_t*) malloc ( allocSize );
	CUDA_CHECK_RETURN( hipMalloc( &tally_device, allocSize ));
	CUDA_CHECK_RETURN( hipMemset(tally_device, 0, allocSize));
	gpuErrchk( hipPeekAtLastError() );

	hipEvent_t sync;
	hipEventCreate(&sync);
	testSumCrossSectionAtCollisionLocation<<<nBlocks,nThreads>>>(pCP->ptrPoints_device, pMatList->ptr_device, pMatProps->ptrData_device, pMatList->getHashPtr()->getPtrDevice(), tally_device);
	hipEventRecord(sync, 0);
	hipEventSynchronize(sync);
	gpuErrchk( hipPeekAtLastError() );

	CUDA_CHECK_RETURN(hipMemcpy(tally, tally_device, allocSize, hipMemcpyDeviceToHost));
	gpuErrchk( hipPeekAtLastError() );
	hipFree( tally_device );
	return;
}

void FIGenericGPUTestHelper::launchRayTraceTally(unsigned nBlocks, unsigned nThreads, CollisionPointsHost* pCP, SimpleMaterialListHost* pMatList, MonteRay_MaterialProperties* pMatProps ){
	gpuTallyType_t* tally_device;
	unsigned long long allocSize = sizeof(gpuTallyType_t)*nCells;
	tally = (gpuTallyType_t*) malloc ( allocSize );
	CUDA_CHECK_RETURN( hipMalloc( &tally_device, allocSize ));
	CUDA_CHECK_RETURN( hipMemset(tally_device, 0, allocSize));
	gpuErrchk( hipPeekAtLastError() );

	hipEvent_t sync;
	hipEventCreate(&sync);
	rayTraceTally<<<nBlocks,nThreads>>>(grid_device, pCP->ptrPoints_device, pMatList->ptr_device, pMatProps->ptrData_device, pMatList->getHashPtr()->getPtrDevice(), tally_device);
	hipEventRecord(sync, 0);
	hipEventSynchronize(sync);
	gpuErrchk( hipPeekAtLastError() );

	CUDA_CHECK_RETURN(hipMemcpy(tally, tally_device, allocSize, hipMemcpyDeviceToHost));
	gpuErrchk( hipPeekAtLastError() );
	hipFree( tally_device );
	return;
}

void FIGenericGPUTestHelper::copyGridtoGPU( GridBins* grid){
	// allocate and copy the grid
	CUDA_CHECK_RETURN( hipMalloc( &grid_device, sizeof(GridBins) ));
	CUDA_CHECK_RETURN( hipMemcpy(grid_device, grid, sizeof(GridBins), hipMemcpyHostToDevice ));

	nCells = getNumCells(grid);
}

