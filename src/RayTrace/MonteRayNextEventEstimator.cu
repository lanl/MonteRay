#include "MonteRayNextEventEstimator.hh"

namespace MonteRay {

#ifdef __HIPCC__
template<unsigned N>
CUDA_CALLABLE_KERNEL void kernel_ScoreRayList(MonteRayNextEventEstimator* ptr, const RayList_t<N>* pRayList ) {
	const bool debug = false;

	if( debug ) {
		printf("Debug: MonteRayNextEventEstimator::kernel_ScoreRayList\n");
	}

	unsigned tid = threadIdx.x + blockIdx.x*blockDim.x;

	unsigned num = pRayList->size();
	while( tid < num ) {
		if( debug ) {
			printf("Debug: MonteRayNextEventEstimator::kernel_ScoreRayList -- tid=%d\n", tid);
		}
		ptr->score(pRayList,tid);
		tid += blockDim.x*gridDim.x;
	}
}
#endif

#ifdef __HIPCC__
template<unsigned N>
void
MonteRayNextEventEstimator::launch_ScoreRayList( unsigned nBlocks, unsigned nThreads, hipStream_t& stream, const RayList_t<N>* pRayList ) {
	const bool debug = false;

	if( debug ) {
		printf("Debug: MonteRayNextEventEstimator::launch_ScoreRayList -- calling kernel_ScoreRayList\n");
	}
	kernel_ScoreRayList<<<nBlocks, nThreads, 0, stream>>>( devicePtr, pRayList->devicePtr );
	if( debug ) {
		hipError_t cudaerr = hipDeviceSynchronize();
		if( cudaerr != hipSuccess ) {
			printf("kernel_ScoreRayList launch failed with error \"%s\".\n", hipGetErrorString(cudaerr));
		}
	}
}

template void MonteRayNextEventEstimator::launch_ScoreRayList<1>( unsigned nBlocks, unsigned nThreads, hipStream_t& stream, const RayList_t<1>* pRayList );
template void MonteRayNextEventEstimator::launch_ScoreRayList<3>( unsigned nBlocks, unsigned nThreads, hipStream_t& stream, const RayList_t<3>* pRayList );
#endif

}


