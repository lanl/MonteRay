#include "hip/hip_runtime.h"
#include "MonteRayNextEventEstimator.hh"

namespace MonteRay {

template<unsigned N>
CUDA_CALLABLE_KERNEL void kernel_ScoreRayList(MonteRayNextEventEstimator* ptr, const RayList_t<N>* pRayList ) {
	const bool debug = false;

	if( debug ) {
		printf("Debug: MonteRayNextEventEstimator::kernel_ScoreRayList\n");
	}

#ifdef __HIPCC__
	unsigned tid = threadIdx.x + blockIdx.x*blockDim.x;
#else
	unsigned tid = 0;
#endif

	unsigned num = pRayList->size();
	while( tid < num ) {
		if( debug ) {
			printf("Debug: MonteRayNextEventEstimator::kernel_ScoreRayList -- tid=%d\n", tid);
		}
		ptr->score(pRayList,tid);

#ifdef __HIPCC__
		tid += blockDim.x*gridDim.x;
#else
		++tid;
#endif
	}
}

#ifdef __HIPCC__
template<unsigned N>
void MonteRayNextEventEstimator::launch_ScoreRayList( unsigned nBlocks, unsigned nThreads, hipStream_t& stream, const RayList_t<N>* pRayList )
#else
template<unsigned N>
void MonteRayNextEventEstimator::launch_ScoreRayList( unsigned nBlocks, unsigned nThreads, const RayList_t<N>* pRayList )
#endif
	{
	const bool debug = false;

	const unsigned nRays = pRayList->size();
	if( nThreads > nRays ) {
		nThreads = nRays;
	}
	nThreads = (( nThreads + 32 -1 ) / 32 ) *32;

    const unsigned numThreadOverload = nBlocks;
    nBlocks = std::min(( nRays + numThreadOverload*nThreads -1 ) / (numThreadOverload*nThreads), 65535U);

	if( debug ) {
		printf("Debug: MonteRayNextEventEstimator::launch_ScoreRayList -- launching kernel_ScoreRayList with %d blocks, %d threads, to process %d rays\n", nBlocks, nThreads, nRays);
	}
#ifdef __HIPCC__
	kernel_ScoreRayList<<<nBlocks, nThreads, 0, stream>>>( devicePtr, pRayList->devicePtr );
	if( debug ) {
		hipError_t cudaerr = hipDeviceSynchronize();
		if( cudaerr != hipSuccess ) {
			printf("MonteRayNextEventEstimator::launch_ScoreRayList -- kernel_ScoreRayList launch failed with error \"%s\".\n", hipGetErrorString(cudaerr));
		}
	}
#else
	kernel_ScoreRayList( this, pRayList );
#endif

}

#ifdef __HIPCC__
template void MonteRayNextEventEstimator::launch_ScoreRayList<1>( unsigned nBlocks, unsigned nThreads, hipStream_t& stream, const RayList_t<1>* pRayList );
template void MonteRayNextEventEstimator::launch_ScoreRayList<3>( unsigned nBlocks, unsigned nThreads, hipStream_t& stream, const RayList_t<3>* pRayList );
#else
template void MonteRayNextEventEstimator::launch_ScoreRayList<1>( unsigned nBlocks, unsigned nThreads, const RayList_t<1>* pRayList );
template void MonteRayNextEventEstimator::launch_ScoreRayList<3>( unsigned nBlocks, unsigned nThreads, const RayList_t<3>* pRayList );
#endif

}


