#include "hip/hip_runtime.h"
#include <UnitTest++.h>

#include <memory>
#include <cmath>

#include "MonteRayDefinitions.hh"
#include "GPUUtilityFunctions.hh"
#include "GridBins.hh"
#include "MonteRay_SpatialGrid.hh"
#include "Ray.hh"
#include "MonteRayNextEventEstimator.t.hh"
#include "MonteRayCrossSection.hh"

#include "MonteRayMaterial.hh"
#include "MonteRay_MaterialProperties.hh"
// #define MEMCHECK 1

namespace nextEventEsimator_unittest{

using namespace MonteRay;

SUITE( NextEventEstimator_Tester ) {

#ifndef MEMCHECK
    TEST(  make_a_PointDetRay ) {
        PointDetRay_t ray;
        CHECK_EQUAL(3, ray.getN() );
        CHECK(true);
    }

    TEST(  MonteRayNextEventEstimator_ctor ) {
        MonteRayNextEventEstimator<GridBins> estimator(1);
        CHECK_EQUAL(0, estimator.size() );
        CHECK_EQUAL(1, estimator.capacity() );
        CHECK_CLOSE( 0.0, estimator.getExclusionRadius(), 1e-6 );
    }

    TEST(  MonteRayNextEventEstimator_get_invalid_X ) {
        MonteRayNextEventEstimator<GridBins> estimator(1);
#ifdef DEBUG
        CHECK_THROW( estimator.getX(10), std::runtime_error );
        CHECK_THROW( estimator.getY(10), std::runtime_error );
        CHECK_THROW( estimator.getZ(10), std::runtime_error );
#endif
    }

    TEST( add ) {
        MonteRayNextEventEstimator<GridBins> estimator(1);
        unsigned id = estimator.add( 1.0, 2.0, 3.0);
        CHECK_EQUAL( 0, id);
        CHECK_CLOSE( 1.0, estimator.getX(0), 1e-6 );
        CHECK_CLOSE( 2.0, estimator.getY(0), 1e-6 );
        CHECK_CLOSE( 3.0, estimator.getZ(0), 1e-6 );
    }

    TEST( add_too_many ) {
        MonteRayNextEventEstimator<GridBins> estimator(1);
        unsigned id = estimator.add( 1.0, 2.0, 3.0);
        CHECK_THROW( estimator.add( 1.0, 2.0, 3.0), std::runtime_error );
    }

    TEST( set_exclusion_radius ) {
        MonteRayNextEventEstimator<GridBins> estimator(1);
        estimator.setExclusionRadius( 1.9 );
        CHECK_CLOSE( 1.9, estimator.getExclusionRadius(), 1e-6 );
    }

    TEST( getDistance ) {
        MonteRayNextEventEstimator<GridBins> estimator(1);
        unsigned id = estimator.add( 3.0, 3.0, 3.0);
        gpuFloatType_t expectedDistance = std::sqrt( (3.0f*3.0f)*3 );

        MonteRay::Vector3D<gpuRayFloat_t> pos(0.0, 0.0, 0.0);

        gpuFloatType_t distance = estimator.distance( 0, pos );

        CHECK_CLOSE( expectedDistance, distance, 1e-6 );
    }

    TEST( getDistanceDirection_PosU ) {
        MonteRayNextEventEstimator<GridBins> estimator(1);
        unsigned id = estimator.add( 3.0, 0.0, 0.0);
        gpuFloatType_t expectedDistance = std::sqrt( 3.0f*3.0f );

        MonteRay::Vector3D<gpuRayFloat_t> pos(0.0, 0.0, 0.0);
        MonteRay::Vector3D<gpuRayFloat_t> dir;

        gpuFloatType_t distance = estimator.getDistanceDirection( 0, pos, dir );

        CHECK_CLOSE( expectedDistance, distance, 1e-6 );
        CHECK_CLOSE( 1.0, dir[0], 1e-6 );
    }

    TEST( getDistanceDirection_NegU ) {
        MonteRayNextEventEstimator<GridBins> estimator(1);
        unsigned id = estimator.add( -3.0, 0.0, 0.0);
        gpuFloatType_t expectedDistance = std::sqrt( 3.0f*3.0f );

        MonteRay::Vector3D<gpuRayFloat_t> pos(0.0, 0.0, 0.0);
        MonteRay::Vector3D<gpuRayFloat_t> dir;

        gpuFloatType_t distance = estimator.getDistanceDirection( 0, pos, dir );

        CHECK_CLOSE( expectedDistance, distance, 1e-6 );
        CHECK_CLOSE( -1.0, dir[0], 1e-6 );
    }

    TEST( getDistanceDirection_PosV ) {
        MonteRayNextEventEstimator<GridBins> estimator(1);
        unsigned id = estimator.add( 0.0, 3.0, 0.0);
        gpuFloatType_t expectedDistance = std::sqrt( 3.0f*3.0f );

        MonteRay::Vector3D<gpuRayFloat_t> pos(0.0, 0.0, 0.0);
        MonteRay::Vector3D<gpuRayFloat_t> dir;

        gpuFloatType_t distance = estimator.getDistanceDirection( 0, pos, dir );

        CHECK_CLOSE( expectedDistance, distance, 1e-6 );
        CHECK_CLOSE( 1.0, dir[1], 1e-6 );
    }

    TEST( getDistanceDirection_NegV ) {
        MonteRayNextEventEstimator<GridBins> estimator(1);
        unsigned id = estimator.add( 0.0, -3.0, 0.0);
        gpuFloatType_t expectedDistance = std::sqrt( 3.0f*3.0f );

        MonteRay::Vector3D<gpuRayFloat_t> pos(0.0, 0.0, 0.0);
        MonteRay::Vector3D<gpuRayFloat_t> dir;

        gpuFloatType_t distance = estimator.getDistanceDirection( 0, pos, dir );

        CHECK_CLOSE( expectedDistance, distance, 1e-6 );
        CHECK_CLOSE( -1.0, dir[1], 1e-6 );
    }

    TEST( getDistanceDirection_PosW ) {
        MonteRayNextEventEstimator<GridBins> estimator(1);
        unsigned id = estimator.add( 0.0, 0.0, 3.0);
        gpuFloatType_t expectedDistance = std::sqrt( 3.0f*3.0f );

        MonteRay::Vector3D<gpuRayFloat_t> pos(0.0, 0.0, 0.0);
        MonteRay::Vector3D<gpuRayFloat_t> dir;

        gpuFloatType_t distance = estimator.getDistanceDirection( 0, pos, dir );

        CHECK_CLOSE( expectedDistance, distance, 1e-6 );
        CHECK_CLOSE( 1.0, dir[2], 1e-6 );
    }

    TEST( getDistanceDirection_NegW ) {
        MonteRayNextEventEstimator<GridBins> estimator(1);
        unsigned id = estimator.add( 0.0, 0.0, -3.0);
        gpuFloatType_t expectedDistance = std::sqrt( 3.0f*3.0f );

        MonteRay::Vector3D<gpuRayFloat_t> pos(0.0, 0.0, 0.0);
        MonteRay::Vector3D<gpuRayFloat_t> dir;

        gpuFloatType_t distance = estimator.getDistanceDirection( 0, pos, dir );

        CHECK_CLOSE( expectedDistance, distance, 1e-6 );
        CHECK_CLOSE( -1.0, dir[2], 1e-6 );
    }

    TEST( getDistanceDirection_PosUV ) {
        MonteRayNextEventEstimator<GridBins> estimator(1);
        unsigned id = estimator.add( 3.0, 3.0, 0.0);
        gpuFloatType_t expectedDistance = std::sqrt( (3.0f*3.0f)*2 );

        MonteRay::Vector3D<gpuRayFloat_t> pos(0.0, 0.0, 0.0);
        MonteRay::Vector3D<gpuRayFloat_t> dir;

        gpuFloatType_t distance = estimator.getDistanceDirection( 0, pos, dir );

        CHECK_CLOSE( expectedDistance, distance, 1e-6 );
        CHECK_CLOSE( 1.0/sqrt(2.0), dir[0], 1e-6 );
        CHECK_CLOSE( 1.0/sqrt(2.0), dir[1], 1e-6 );
    }
#endif

    class CalcScore_test {
    public:
        CalcScore_test(){

            // Two 1-cm think slabs in x direction
            grid.setVertices( 0, 0.0, 2.0, 2);
            grid.setVertices( 1, -10.0, 10.0, 1);
            grid.setVertices( 2, -10.0, 10.0, 1);
            grid.finalize();
            grid.copyToGPU();

            cell1.add( 0, 0.0); // vacuum
            matProps.add( cell1 );

            cell2.add( 0, 1.0); // density = 1.0
            matProps.add( cell2 );

            matProps.setupPtrData();

            // setup a material list
            pXS = std::unique_ptr<MonteRayCrossSectionHost> ( new MonteRayCrossSectionHost(4) );
            pXS->setParticleType( photon );
            pXS->setTotalXS(0, 1e-11, 1.0 );
            pXS->setTotalXS(1, 0.75, 1.0 );
            pXS->setTotalXS(2, 1.00, 2.0 );
            pXS->setTotalXS(3, 3.00, 4.0 );
            pXS->setAWR( gpu_AvogadroBarn / gpu_neutron_molar_mass );

            pMat = std::unique_ptr<MonteRayMaterialHost>( new MonteRayMaterialHost(1) );
            pMat->add( 0, *pXS, 1.0);
            pMat->normalizeFractions();
            pMat->copyToGPU();

            pMatList = std::unique_ptr<MonteRayMaterialListHost>( new MonteRayMaterialListHost(1,1,3) );
            pMatList->add(0, *pMat, 0);
            pMatList->copyToGPU();

            matProps.renumberMaterialIDs(*pMatList);
            matProps.copyToGPU();

            pXS->copyToGPU();

            pEstimator = std::unique_ptr<MonteRayNextEventEstimator<GridBins>>( new MonteRayNextEventEstimator<GridBins>(10) );
            pEstimator->setGeometry( &grid, &matProps );
            pEstimator->setMaterialList( pMatList.get() );
        }
        ~CalcScore_test(){}

    public:
        GridBins grid;
        MonteRay_CellProperties cell1, cell2;
        std::unique_ptr<MonteRayMaterialListHost> pMatList;
        std::unique_ptr<MonteRayMaterialHost> pMat;
        std::unique_ptr<MonteRayCrossSectionHost> pXS;
        MonteRay_MaterialProperties matProps;

        std::unique_ptr<MonteRayNextEventEstimator<GridBins>> pEstimator;
    };

#ifndef MEMCHECK
    TEST_FIXTURE(CalcScore_test, calcScore_vacuum ) {
        CHECK_CLOSE( 1.0, pXS->getTotalXS( 0.5 ), 1e-6 );
        CHECK_CLOSE( 1.0, pMat->getTotalXS( 0.5 ), 1e-6 );

        unsigned id = pEstimator->add( 1.0, 0.0, 0.0);
        pEstimator->initialize();

        gpuFloatType_t x = 0.0;
        gpuFloatType_t y = 0.0;
        gpuFloatType_t z = 0.0;
        gpuFloatType_t u = 1.0;
        gpuFloatType_t v = 0.0;
        gpuFloatType_t w = 0.0;

        gpuFloatType_t energy[1];
        energy[0]= 0.5;

        gpuFloatType_t weight[1];
        weight[0] = 0.5;  // isotropic

        Ray_t<> ray;
        ray.pos[0] = x;
        ray.pos[1] = y;
        ray.pos[2] = z;
        ray.dir[0] = u;
        ray.dir[1] = v;
        ray.dir[2] = w;
        ray.energy[0] = energy[0];
        ray.weight[0] = weight[0];
        ray.detectorIndex = 0;
        ray.particleType = photon;

        unsigned particleID = 0;
        RayWorkInfo<> rayInfo(1,true);
        gpuFloatType_t score = pEstimator->calcScore<1>(particleID, ray, rayInfo );

        gpuFloatType_t expected = ( 1/ (4.0f * MonteRay::pi ) ) * exp(-0.0);
        CHECK_CLOSE( expected, score, 1e-6);
    }

    TEST_FIXTURE(CalcScore_test, calcScore_thru_material ) {
        CHECK_CLOSE( 1.0, pXS->getTotalXS( 0.5 ) , 1e-6 );
        CHECK_CLOSE( 1.0, pMat->getTotalXS( 0.5 ) , 1e-6 );

        unsigned id = pEstimator->add( 2.0, 0.0, 0.0);
        pEstimator->initialize();

        gpuFloatType_t x = 1.0;
        gpuFloatType_t y = 0.0;
        gpuFloatType_t z = 0.0;
        gpuFloatType_t u = 1.0;
        gpuFloatType_t v = 0.0;
        gpuFloatType_t w = 0.0;

        gpuFloatType_t energy[1];
        energy[0]= 0.5;

        gpuFloatType_t weight[1];
        weight[0] = 0.5;  // isotropic

        Ray_t<> ray;
        ray.pos[0] = x;
        ray.pos[1] = y;
        ray.pos[2] = z;
        ray.dir[0] = u;
        ray.dir[1] = v;
        ray.dir[2] = w;
        ray.energy[0] = energy[0];
        ray.weight[0] = weight[0];
        ray.detectorIndex = 0;
        ray.particleType = photon;

        unsigned particleID = 0;
        RayWorkInfo<> rayInfo(1,true);
        gpuFloatType_t score = pEstimator->calcScore<1>(particleID, ray, rayInfo );

        gpuFloatType_t expected = ( 1/ (4.0f * MonteRay::pi ) ) * exp(-1.0);
        CHECK_CLOSE( expected, score, 1e-6);
    }

    TEST_FIXTURE(CalcScore_test, calcScore_thru_vacuum_and_material ) {
        unsigned id = pEstimator->add( 2.0, 0.0, 0.0);
        pEstimator->initialize();

        gpuFloatType_t x = 0.0;
        gpuFloatType_t y = 0.0;
        gpuFloatType_t z = 0.0;
        gpuFloatType_t u = 1.0;
        gpuFloatType_t v = 0.0;
        gpuFloatType_t w = 0.0;

        gpuFloatType_t energy[1];
        energy[0]= 0.5;

        gpuFloatType_t weight[1];
        weight[0] = 0.5;  // isotropic

        Ray_t<> ray;
        ray.pos[0] = x;
        ray.pos[1] = y;
        ray.pos[2] = z;
        ray.dir[0] = u;
        ray.dir[1] = v;
        ray.dir[2] = w;
        ray.energy[0] = energy[0];
        ray.weight[0] = weight[0];
        ray.detectorIndex = 0;
        ray.particleType = photon;

        unsigned particleID = 0;
        RayWorkInfo<> rayInfo(1,true);
        gpuFloatType_t score = pEstimator->calcScore<1>(particleID, ray, rayInfo );

        gpuFloatType_t expected = ( 1/ (4.0f * MonteRay::pi * 2.0f*2.0f ) ) * exp(-1.0);
        CHECK_CLOSE( expected, score, 1e-6);
    }

    TEST_FIXTURE(CalcScore_test, getTally ) {
        unsigned id = pEstimator->add( 2.0, 0.0, 0.0);
        pEstimator->initialize();

        gpuFloatType_t x = 0.0;
        gpuFloatType_t y = 0.0;
        gpuFloatType_t z = 0.0;
        gpuFloatType_t u = 1.0;
        gpuFloatType_t v = 0.0;
        gpuFloatType_t w = 0.0;

        gpuFloatType_t energy[1];
        energy[0]= 0.5;

        gpuFloatType_t weight[1];
        weight[0] = 0.5;  // isotropic

        Ray_t<> ray;
        ray.pos[0] = x;
        ray.pos[1] = y;
        ray.pos[2] = z;
        ray.dir[0] = u;
        ray.dir[1] = v;
        ray.dir[2] = w;
        ray.energy[0] = energy[0];
        ray.weight[0] = weight[0];
        ray.detectorIndex = 0;
        ray.particleType = photon;

        unsigned particleID = 0;
        RayWorkInfo<> rayInfo(1,true);
        pEstimator->calcScore<1>(particleID, ray, rayInfo );

        gpuFloatType_t score = pEstimator->getTally(0,0);

        gpuFloatType_t expected = ( 1/ (4.0f * MonteRay::pi * 2.0f*2.0f ) ) * exp(-1.0);
        CHECK_CLOSE( expected, score, 1e-6);
    }

    TEST_FIXTURE(CalcScore_test, addTimeBins ) {
        gpuFloatType_t distance1 = 2.0; // score at t=0.006
        gpuFloatType_t distance2 = 400.0; // score at t=1.33

        pEstimator->add( distance1, 0.0, 0.0);
        pEstimator->add( distance2, 0.0, 0.0);
        std::vector<MonteRay::gpuFloatType_t> timeEdges= { 1.0, 2.0, 10.0, 20.0, 100.0 };
        pEstimator->setTimeBinEdges( timeEdges );
        pEstimator->initialize();

        gpuFloatType_t x = 0.0;
        gpuFloatType_t y = 0.0;
        gpuFloatType_t z = 0.0;
        gpuFloatType_t u = 1.0;
        gpuFloatType_t v = 0.0;
        gpuFloatType_t w = 0.0;

        gpuFloatType_t energy[1];
        energy[0]= 0.5;

        gpuFloatType_t weight[1];
        weight[0] = 0.5;  // isotropic

        Ray_t<> ray;
        ray.pos[0] = x;
        ray.pos[1] = y;
        ray.pos[2] = z;
        ray.dir[0] = u;
        ray.dir[1] = v;
        ray.dir[2] = w;
        ray.energy[0] = energy[0];
        ray.weight[0] = weight[0];
        ray.time = 0.0;
        ray.detectorIndex = 0;
        ray.particleType = photon;

        unsigned particleID = 0;
        RayWorkInfo<> rayInfo(1,true);
        pEstimator->calcScore<1>(particleID, ray, rayInfo );

        gpuFloatType_t expected1 = ( 1/ (4.0f * MonteRay::pi * distance1*distance1 ) ) * exp(-1.0);
        CHECK_CLOSE( expected1, pEstimator->getTally(0,0), 1e-6);
        CHECK_CLOSE( 0.0,      pEstimator->getTally(0,1), 1e-6);
        CHECK_CLOSE( 0.0,      pEstimator->getTally(0,2), 1e-6);
        CHECK_CLOSE( 0.0,      pEstimator->getTally(0,3), 1e-6);
        CHECK_CLOSE( 0.0,      pEstimator->getTally(0,4), 1e-6);
        CHECK_CLOSE( 0.0,      pEstimator->getTally(0,5), 1e-6);
        CHECK_CLOSE( 0.0,      pEstimator->getTally(1,0), 1e-6);
        CHECK_CLOSE( 0.0,      pEstimator->getTally(1,1), 1e-6);
        CHECK_CLOSE( 0.0,      pEstimator->getTally(1,2), 1e-6);
        CHECK_CLOSE( 0.0,      pEstimator->getTally(1,3), 1e-6);
        CHECK_CLOSE( 0.0,      pEstimator->getTally(1,4), 1e-6);
        CHECK_CLOSE( 0.0,      pEstimator->getTally(1,5), 1e-6);

        ray.detectorIndex = 1;
        rayInfo.clear();
        pEstimator->calcScore<1>(particleID, ray, rayInfo );

        gpuFloatType_t expected2 = ( 1/ (4.0f * MonteRay::pi * distance2*distance2 ) ) * exp(-1.0);

        CHECK_CLOSE( expected1, pEstimator->getTally(0,0), 1e-6);
        CHECK_CLOSE( 0.0,       pEstimator->getTally(0,1), 1e-6);
        CHECK_CLOSE( 0.0,       pEstimator->getTally(0,2), 1e-6);
        CHECK_CLOSE( 0.0,       pEstimator->getTally(0,3), 1e-6);
        CHECK_CLOSE( 0.0,       pEstimator->getTally(0,4), 1e-6);
        CHECK_CLOSE( 0.0,       pEstimator->getTally(0,5), 1e-6);
        CHECK_CLOSE( 0.0,       pEstimator->getTally(1,0), 1e-6);
        CHECK_CLOSE( expected2, pEstimator->getTally(1,1), 1e-6);
        CHECK_CLOSE( 0.0,       pEstimator->getTally(1,2), 1e-6);
        CHECK_CLOSE( 0.0,       pEstimator->getTally(1,3), 1e-6);
        CHECK_CLOSE( 0.0,       pEstimator->getTally(1,4), 1e-6);
        CHECK_CLOSE( 0.0,       pEstimator->getTally(1,5), 1e-6);
    }

    TEST_FIXTURE(CalcScore_test, calcScore_thru_material_3_probabilities ) {
        CHECK_CLOSE( 1.0, pXS->getTotalXS( 0.5 ) , 1e-6 );
        CHECK_CLOSE( 1.0, pMat->getTotalXS( 0.5 ) , 1e-6 );
        CHECK_CLOSE( 2.0, pXS->getTotalXS( 1.0 ) , 1e-6 );
        CHECK_CLOSE( 2.0, pMat->getTotalXS( 1.0 ) , 1e-6 );
        CHECK_CLOSE( 4.0, pXS->getTotalXS( 3.0 ) , 1e-6 );
        CHECK_CLOSE( 4.0, pMat->getTotalXS( 3.0 ) , 1e-6 );

        unsigned id = pEstimator->add( 2.0, 0.0, 0.0);
        pEstimator->initialize();

        const unsigned N = 3;

        gpuFloatType_t x = 0.0;
        gpuFloatType_t y = 0.0;
        gpuFloatType_t z = 0.0;
        gpuFloatType_t u = 1.0;
        gpuFloatType_t v = 0.0;
        gpuFloatType_t w = 0.0;

        gpuFloatType_t energy[N];
        energy[0]= 0.5;
        energy[1]= 1.0;
        energy[2]= 3.0;

        gpuFloatType_t weight[N];
        weight[0] = 0.3;  // isotropic
        weight[1] = 1.0;
        weight[2] = 2.0;

        Ray_t<N> ray;
        ray.pos[0] = x;
        ray.pos[1] = y;
        ray.pos[2] = z;
        ray.dir[0] = u;
        ray.dir[1] = v;
        ray.dir[2] = w;
        ray.energy[0] = energy[0];
        ray.energy[1] = energy[1];
        ray.energy[2] = energy[2];
        ray.weight[0] = weight[0];
        ray.weight[1] = weight[1];
        ray.weight[2] = weight[2];
        ray.detectorIndex = 0;
        ray.particleType = photon;

        //std:: cout << "Debug: *************************\n";
        unsigned particleID = 0;
        RayWorkInfo<N> rayInfo(1,true);
        gpuFloatType_t score = pEstimator->calcScore<N>(particleID, ray, rayInfo );
        //std:: cout << "Debug: *************************\n";

        gpuFloatType_t expected1 = ( 0.3f / (2.0f * MonteRay::pi * 4.0f ) ) * exp( -1.0*1.0 );
        gpuFloatType_t expected2 = ( 1.0f / (2.0f * MonteRay::pi * 4.0f ) ) * exp( -1.0*2.0 );
        gpuFloatType_t expected3 = ( 2.0f / (2.0f * MonteRay::pi * 4.0f ) ) * exp( -1.0*4.0 );
        CHECK_CLOSE( 0.00439124, expected1, 1e-7);
        CHECK_CLOSE( 0.00538482, expected2, 1e-7);
        CHECK_CLOSE( 0.00145751, expected3, 1e-7);
        CHECK_CLOSE( expected1+expected2+expected3, score, 1e-7);

    }

    TEST_FIXTURE(CalcScore_test, calcScore_with_RayList ) {
        const unsigned N = 3;
        unsigned id = pEstimator->add( 2.0, 0.0, 0.0);
        pEstimator->initialize();

        gpuFloatType_t x = 0.0;
        gpuFloatType_t y = 0.0;
        gpuFloatType_t z = 0.0;
        gpuFloatType_t u = 1.0;
        gpuFloatType_t v = 0.0;
        gpuFloatType_t w = 0.0;

        gpuFloatType_t energy[N];
        energy[0]= 0.5;
        energy[1]= 1.0;
        energy[2]= 3.0;

        gpuFloatType_t weight[N];
        weight[0] = 0.3;  // isotropic
        weight[1] = 1.0;
        weight[2] = 2.0;

        Ray_t<N> ray;
        ray.pos[0] = x;
        ray.pos[1] = y;
        ray.pos[2] = z;
        ray.dir[0] = u;
        ray.dir[1] = v;
        ray.dir[2] = w;

        for( unsigned i=0;i<N;++i) {
            ray.energy[i] = energy[i];
            ray.weight[i] = weight[i];
        }
        ray.index = 0;
        ray.detectorIndex = 0;
        ray.particleType = photon;

        std::unique_ptr<RayList_t<N>> pBank =  std::unique_ptr<RayList_t<N>>( new RayList_t<N>(2) );
        pBank->add( ray );
        pBank->add( ray );

        RayWorkInfo<N> rayInfo(pBank->size(),true);

        //std:: cout << "Debug: **********calcScore_with_RayList***************\n";
        CHECK_CLOSE( 0.0, pEstimator->getTally(0), 1e-7);
        pEstimator->cpuScoreRayList(pBank.get(), &rayInfo);
        gpuTallyType_t value = pEstimator->getTally(0);
        //std:: cout << "Debug: ************************************************\n";

        gpuFloatType_t expected1 = ( 0.3f / (2.0f * MonteRay::pi * 4.0f ) ) * exp( -1.0*1.0 );
        gpuFloatType_t expected2 = ( 1.0f / (2.0f * MonteRay::pi * 4.0f ) ) * exp( -1.0*2.0 );
        gpuFloatType_t expected3 = ( 2.0f / (2.0f * MonteRay::pi * 4.0f ) ) * exp( -1.0*4.0 );

        CHECK_CLOSE( 2*(expected1+expected2+expected3), value, 1e-7);

    }
#endif

    TEST_FIXTURE(CalcScore_test, calc_with_rayList_on_GPU ) {

//        std::cout << "Debug: **********************\n";
//        std::cout << "Debug: MonteRayNextEventEstimator_unittest.cc -- TEST calc_with_rayList_on_GPU\n";
        const unsigned N = 3;
        unsigned id = pEstimator->add( 2.0, 0.0, 0.0);
        pEstimator->initialize();

        gpuFloatType_t x = 0.0;
        gpuFloatType_t y = 0.0;
        gpuFloatType_t z = 0.0;
        gpuFloatType_t u = 1.0;
        gpuFloatType_t v = 0.0;
        gpuFloatType_t w = 0.0;

        gpuFloatType_t energy[N];
        energy[0]= 0.5;
        energy[1]= 1.0;
        energy[2]= 3.0;

        gpuFloatType_t weight[N];
        weight[0] = 0.3;  // isotropic
        weight[1] = 1.0;
        weight[2] = 2.0;

        Ray_t<N> ray;
        ray.pos[0] = x;
        ray.pos[1] = y;
        ray.pos[2] = z;
        ray.dir[0] = u;
        ray.dir[1] = v;
        ray.dir[2] = w;

        for( unsigned i=0;i<N;++i) {
            ray.energy[i] = energy[i];
            ray.weight[i] = weight[i];
        }
        ray.index = 0;
        ray.detectorIndex = 0;
        ray.particleType = photon;

        std::unique_ptr<RayList_t<N>> pBank =  std::unique_ptr<RayList_t<N>>( new RayList_t<N>(2) );
        pBank->add( ray );
        pBank->add( ray );

#ifdef __HIPCC__
        hipEvent_t start, stop;
        hipEventCreate(&start);

        pBank->copyToGPU();
        pEstimator->copyToGPU();

        RayWorkInfo<N> rayInfo(1);
        rayInfo.copyToGPU();

        hipStream_t* stream = NULL;
        stream = new hipStream_t;
        stream[0] = 0;  // use the default stream

        hipEventRecord(start, 0);
        hipEventSynchronize(start);

        hipEventCreate(&stop);

        hipStreamSynchronize(*stream);
        pEstimator->launch_ScoreRayList(1, 1, pBank.get(), &rayInfo, stream );

        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipStreamSynchronize(*stream);
        pEstimator->copyToCPU();

        delete stream;
#else
        pEstimator->launch_ScoreRayList(1,1,pBank.get());
#endif
        gpuTallyType_t value = pEstimator->getTally(0);

        gpuFloatType_t expected1 = ( 0.3f / (2.0f * MonteRay::pi * 4.0f ) ) * exp( -1.0*1.0 );
        gpuFloatType_t expected2 = ( 1.0f / (2.0f * MonteRay::pi * 4.0f ) ) * exp( -1.0*2.0 );
        gpuFloatType_t expected3 = ( 2.0f / (2.0f * MonteRay::pi * 4.0f ) ) * exp( -1.0*4.0 );

        CHECK_CLOSE( 2*(expected1+expected2+expected3), value, 1e-7);
//        std::cout << "Debug: **********************\n";
    }

    TEST_FIXTURE(CalcScore_test, rayListOnGPU_withTimeBins ) {

 //        std::cout << "Debug: **********************\n";
 //        std::cout << "Debug: MonteRayNextEventEstimator_unittest.cc -- TEST calc_with_rayList_on_GPU\n";
         const unsigned N = 3;
         gpuFloatType_t distance1 = 2.0; // score at t=0.006
         gpuFloatType_t distance2 = 400.0; // score at t=1.33

         pEstimator->add( distance1, 0.0, 0.0);
         pEstimator->add( distance2, 0.0, 0.0);
         std::vector<MonteRay::gpuFloatType_t> timeEdges= { 1.0, 2.0, 10.0, 20.0, 100.0 };
         pEstimator->setTimeBinEdges( timeEdges );
         pEstimator->initialize();

         gpuFloatType_t x = 0.0;
         gpuFloatType_t y = 0.0;
         gpuFloatType_t z = 0.0;
         gpuFloatType_t u = 1.0;
         gpuFloatType_t v = 0.0;
         gpuFloatType_t w = 0.0;

         gpuFloatType_t energy[N];
         energy[0]= 0.5;
         energy[1]= 1.0;
         energy[2]= 3.0;

         gpuFloatType_t weight[N];
         weight[0] = 0.3;  // isotropic
         weight[1] = 1.0;
         weight[2] = 2.0;

         Ray_t<N> ray;
         ray.pos[0] = x;
         ray.pos[1] = y;
         ray.pos[2] = z;
         ray.dir[0] = u;
         ray.dir[1] = v;
         ray.dir[2] = w;

         for( unsigned i=0;i<N;++i) {
             ray.energy[i] = energy[i];
             ray.weight[i] = weight[i];
         }
         ray.index = 0;
         ray.detectorIndex = 0;
         ray.particleType = photon;

         std::unique_ptr<RayList_t<N>> pBank =  std::unique_ptr<RayList_t<N>>( new RayList_t<N>(4) );
         pBank->add( ray );
         pBank->add( ray );

         ray.detectorIndex = 1;
         pBank->add( ray );
         pBank->add( ray );

 #ifdef __HIPCC__
         hipEvent_t start, stop;
         hipEventCreate(&start);

         pBank->copyToGPU();
         pEstimator->copyToGPU();

         RayWorkInfo<N> rayInfo(1);
         rayInfo.copyToGPU();

         hipStream_t* stream = NULL;
         stream = new hipStream_t;
         stream[0] = 0;  // use the default stream

         hipEventRecord(start, 0);
         hipEventSynchronize(start);

         hipEventCreate(&stop);

         hipStreamSynchronize(*stream);
         pEstimator->launch_ScoreRayList(1, 1, pBank.get(), &rayInfo, stream );

         hipEventRecord(stop, 0);
         hipEventSynchronize(stop);
         hipStreamSynchronize(*stream);
         pEstimator->copyToCPU();

         delete stream;
 #else
         pEstimator->launch_ScoreRayList(1,1,pBank.get());
 #endif

         gpuFloatType_t expected1  = ( 0.3f / (2.0f * MonteRay::pi * distance1*distance1 ) ) * exp( -1.0*1.0 ) +
                                     ( 1.0f / (2.0f * MonteRay::pi * distance1*distance1 ) ) * exp( -1.0*2.0 ) +
                                     ( 2.0f / (2.0f * MonteRay::pi * distance1*distance1 ) ) * exp( -1.0*4.0 );

         gpuFloatType_t expected2  = ( 0.3f / (2.0f * MonteRay::pi * distance2*distance2 ) ) * exp( -1.0*1.0 ) +
                                     ( 1.0f / (2.0f * MonteRay::pi * distance2*distance2 ) ) * exp( -1.0*2.0 ) +
                                     ( 2.0f / (2.0f * MonteRay::pi * distance2*distance2 ) ) * exp( -1.0*4.0 );

         CHECK_CLOSE( 2*expected1, pEstimator->getTally(0,0), 1e-7);
         CHECK_CLOSE(         0.0, pEstimator->getTally(0,1), 1e-7);
         CHECK_CLOSE(         0.0, pEstimator->getTally(0,2), 1e-7);
         CHECK_CLOSE(         0.0, pEstimator->getTally(0,3), 1e-7);
         CHECK_CLOSE(         0.0, pEstimator->getTally(0,4), 1e-7);
         CHECK_CLOSE(         0.0, pEstimator->getTally(0,5), 1e-7);

         CHECK_CLOSE(         0.0, pEstimator->getTally(1,0), 1e-7);
         CHECK_CLOSE( 2*expected2, pEstimator->getTally(1,1), 1e-7);
         CHECK_CLOSE(         0.0, pEstimator->getTally(1,2), 1e-7);
         CHECK_CLOSE(         0.0, pEstimator->getTally(1,3), 1e-7);
         CHECK_CLOSE(         0.0, pEstimator->getTally(1,4), 1e-7);
         CHECK_CLOSE(         0.0, pEstimator->getTally(1,5), 1e-7);
 //        std::cout << "Debug: **********************\n";
     }

    TEST_FIXTURE(CalcScore_test, write ) {

         const unsigned N = 3;
         gpuFloatType_t distance1 = 2.0; // score at t=0.006
         gpuFloatType_t distance2 = 400.0; // score at t=1.33

         pEstimator->add( distance1, 0.0, 0.0);
         pEstimator->add( distance2, 0.0, 0.0);
         std::vector<MonteRay::gpuFloatType_t> timeEdges= { 1.0, 2.0, 10.0, 20.0, 100.0 };
         pEstimator->setTimeBinEdges( timeEdges );
         pEstimator->initialize();

         gpuFloatType_t x = 0.0;
         gpuFloatType_t y = 0.0;
         gpuFloatType_t z = 0.0;
         gpuFloatType_t u = 1.0;
         gpuFloatType_t v = 0.0;
         gpuFloatType_t w = 0.0;

         gpuFloatType_t energy[N];
         energy[0]= 0.5;
         energy[1]= 1.0;
         energy[2]= 3.0;

         gpuFloatType_t weight[N];
         weight[0] = 0.3;  // isotropic
         weight[1] = 1.0;
         weight[2] = 2.0;

         Ray_t<N> ray;
         ray.pos[0] = x;
         ray.pos[1] = y;
         ray.pos[2] = z;
         ray.dir[0] = u;
         ray.dir[1] = v;
         ray.dir[2] = w;

         for( unsigned i=0;i<N;++i) {
             ray.energy[i] = energy[i];
             ray.weight[i] = weight[i];
         }
         ray.index = 0;
         ray.detectorIndex = 0;
         ray.particleType = photon;

         std::unique_ptr<RayList_t<N>> pBank =  std::unique_ptr<RayList_t<N>>( new RayList_t<N>(4) );
         pBank->add( ray );
         pBank->add( ray );

         ray.detectorIndex = 1;
         pBank->add( ray );
         pBank->add( ray );

         // write out state of MonteRayNextEventEstimator class
         std::string filename = std::string("nee_write_state_test.bin");
         pEstimator->writeToFile( filename );

         // read
         {
             MonteRayNextEventEstimator<GridBins> estimator(0);

             // test file exists
             std::ifstream exists(filename.c_str());
             CHECK_EQUAL( true, exists.good() );
             exists.close();

             estimator.readFromFile( filename );

             CHECK_EQUAL( 2, estimator.size() );
             CHECK_EQUAL( 10, estimator.capacity() );

             CHECK_CLOSE( 2.0, estimator.getX(0), 1e-6 );
             CHECK_CLOSE( 0.0, estimator.getY(0), 1e-6 );
             CHECK_CLOSE( 0.0, estimator.getZ(0), 1e-6 );

             CHECK_CLOSE( 400.0, estimator.getX(1), 1e-6 );
             CHECK_CLOSE( 0.0, estimator.getY(1), 1e-6 );
             CHECK_CLOSE( 0.0, estimator.getZ(1), 1e-6 );

             std::vector<gpuFloatType_t> timeBins = estimator.getTimeBinEdges();
             CHECK_EQUAL( 5, timeBins.size() );
         }
    }

    TEST( run_leak_report ) {

#ifdef MEMCHECK
        std:: cout << "Debug: ********************************\n";
        std:: cout << "Debug: ****** Leak report *************\n";
        AllocationTracker::getInstance().reportLeakedMemory();

        hipDeviceReset(); // enable leak checking.
        std:: cout << "Debug: ********************************\n";
#endif
    }


}

} // end namespace

