#include "RayList.hh"

namespace MonteRay {

template<unsigned N>
CUDAHOST_CALLABLE_MEMBER
RayList_t<N>::RayList_t(RayListSize_t num) {
	if( num == 0 ) { num = 1; }
	if( Base::debug ) {
		std::cout << "RayList_t::RayList_t(n), n=" << num << " \n";
	}
	init();
	points = (RAY_T*) MONTERAYHOSTALLOC( num*sizeof( RAY_T ), Base::isManagedMemory, "host RayList_t::points" );
	nAllocated = num;
}

template<unsigned N>
CUDAHOST_CALLABLE_MEMBER
RayList_t<N>::~RayList_t(){
	if( ! Base::isCudaIntermediate ) {
		MonteRayHostFree(points, Base::isManagedMemory );
	} else {
		MonteRayDeviceFree( points );
	}
}

/// Copy constructor
template<unsigned N>
CUDAHOST_CALLABLE_MEMBER
RayList_t<N>::RayList_t(const RayList_t<N>& rhs) :
	RayList_t<N>::RayList_t( rhs.nAllocated )
{
	nUsed = rhs.nUsed;
	std::memcpy( points, rhs.points, rhs.nUsed * sizeof( RAY_T) );
}

template<unsigned N>
CUDAHOST_CALLABLE_MEMBER void
RayList_t<N>::copy(const RayList_t<N>* rhs) {

#ifdef __HIPCC__
	if( Base::debug ) {
		std::cout << "Debug: RayList_t::copy (const RayList_t<N>& rhs) \n";
	}

	if( Base::isCudaIntermediate && rhs->isCudaIntermediate ) {
		throw std::runtime_error("RayList_t::copy -- can NOT copy CUDA intermediate to CUDA intermediate.");
	}

	if( !Base::isCudaIntermediate && !rhs->isCudaIntermediate ) {
		throw std::runtime_error("RayList_t::copy -- can NOT copy CUDA non-intermediate to CUDA non-intermediate.");
	}

	if( nAllocated > 0 && nAllocated != rhs->nAllocated) {
		throw std::runtime_error("RayList_t::copy -- can NOT change the size of the RayList.");
	}

	if( Base::isCudaIntermediate ) {
		// target is the intermediate, origin is the host
		if( points == NULL ) {
			points = (RAY_T*) MONTERAYDEVICEALLOC( rhs->nAllocated*sizeof(RAY_T), "device - RayList_t::points" );
		}
		MonteRayMemcpy(points, rhs->points, rhs->nAllocated*sizeof(RAY_T), hipMemcpyHostToDevice);
	} else {
		// target is the host, origin is the intermediate
		MonteRayMemcpy(rhs->points, points, rhs->nAllocated*sizeof(RAY_T), hipMemcpyDeviceToHost);
	}

	nAllocated = rhs->nAllocated;
	nUsed = rhs->nUsed;
#else
	throw std::runtime_error("RayList_t::copy -- Only valid when compiling with CUDA.");
#endif
}

} // end namespace

template class MonteRay::RayList_t<1>;
template class MonteRay::RayList_t<3>;
