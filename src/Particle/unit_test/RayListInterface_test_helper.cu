#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <iostream>

#include "MonteRayDefinitions.hh"
#include "GPUErrorCheck.hh"
#include "RayListInterface.hh"

#include "RayListInterface_test_helper.hh"

namespace MonteRay {

#ifdef CUDA
template< unsigned N> __global__ void
testGetCapacity(RayList_t<N>* pRayList, MonteRay::RayListSize_t* results){
	results[0] = pRayList->capacity();
	return;
}
#endif

template< unsigned N>
MonteRay::RayListSize_t
RayListInterfaceTester<N>::launchGetCapacity( unsigned nBlocks, unsigned nThreads, RayListInterface<N>& CPs) {
	MonteRay::RayListSize_t* result_device;
	MonteRay::RayListSize_t* result;
	size_t allocSize = sizeof( MonteRay::RayListSize_t ) * 1;
	CUDA_CHECK_RETURN( hipMalloc( &result_device, allocSize ));
	result = (MonteRay::RayListSize_t*) malloc( allocSize );

	hipEvent_t sync;
	hipEventCreate(&sync);
    testGetCapacity<<<nBlocks,nThreads>>>(CPs.getPtrPoints()->devicePtr, result_device);
    gpuErrchk( hipPeekAtLastError() );
	hipEventRecord(sync, 0);
	hipEventSynchronize(sync);

	CUDA_CHECK_RETURN(hipMemcpy(result, result_device, sizeof(MonteRay::RayListSize_t)*1, hipMemcpyDeviceToHost));

	hipFree( result_device );
	MonteRay::RayListSize_t value = *result;
	free(result);
	return value;
}

#ifdef CUDA
template< unsigned N>
__global__ void testSumEnergy(MonteRay::RayList_t<N>* ParticleRayList, gpuFloatType_t* results){
	gpuFloatType_t total = 0.0f;
	for(unsigned i=0; i< ParticleRayList->size(); ++i ) {
		total += ParticleRayList->getEnergy(i);
	}
	results[0] = total;
	return;
}
#endif

template< unsigned N>
gpuFloatType_t
RayListInterfaceTester<N>::launchTestSumEnergy( unsigned nBlocks, unsigned nThreads, RayListInterface<N>& CPs) {
	gpuFloatType_t* result_device;
	gpuFloatType_t result[1];
	CUDA_CHECK_RETURN( hipMalloc( &result_device, sizeof( gpuFloatType_t) * 1 ));

	hipEvent_t sync;
	hipEventCreate(&sync);
	testSumEnergy<<<nBlocks,nThreads>>>(CPs.getPtrPoints()->devicePtr, result_device);
	gpuErrchk( hipPeekAtLastError() );
	hipEventRecord(sync, 0);
	hipEventSynchronize(sync);

	CUDA_CHECK_RETURN(hipMemcpy(result, result_device, sizeof(gpuFloatType_t)*1, hipMemcpyDeviceToHost));

	hipFree( result_device );
	return result[0];
}

template< unsigned N>
RayListInterfaceTester<N>::RayListInterfaceTester(){
}

template< unsigned N>
RayListInterfaceTester<N>::~RayListInterfaceTester(){
//		hipDeviceReset();
}

template< unsigned N>
void RayListInterfaceTester<N>::setupTimers(){
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
}

template< unsigned N>
void RayListInterfaceTester<N>::stopTimers(){
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	float elapsedTime;

	hipEventElapsedTime(&elapsedTime, start, stop );

	std::cout << "Elapsed time in CUDA kernel=" << elapsedTime << " msec" << std::endl;

}

} //end namespace

template class MonteRay::RayListInterfaceTester<1>;
template class MonteRay::RayListInterfaceTester<3>;
