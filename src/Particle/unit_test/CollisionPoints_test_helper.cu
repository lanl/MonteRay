#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <iostream>

#include "MonteRayDefinitions.hh"
#include "GPUErrorCheck.hh"
#include "CollisionPoints.h"

#include "CollisionPoints_test_helper.hh"

using namespace MonteRay;

#ifdef CUDA
__global__ void testGetCapacity(CollisionPoints* pXS, CollisionPointsHost::CollisionPointsSize_t* results){
	results[0] = pXS->capacity();
	return;
}
#endif

CollisionPointsHost::CollisionPointsSize_t
CollisionPointsTester::launchGetCapacity( unsigned nBlocks, unsigned nThreads, CollisionPointsHost& CPs) {
	CollisionPointsHost::CollisionPointsSize_t* result_device;
	CollisionPointsHost::CollisionPointsSize_t* result;
	size_t allocSize = sizeof( CollisionPointsHost::CollisionPointsSize_t) * 1;
	CUDA_CHECK_RETURN( hipMalloc( &result_device, allocSize ));
	result = (CollisionPointsHost::CollisionPointsSize_t*) malloc( allocSize );

	hipEvent_t sync;
	hipEventCreate(&sync);
    testGetCapacity<<<nBlocks,nThreads>>>(CPs.ptrPoints_device, result_device);
    gpuErrchk( hipPeekAtLastError() );
	hipEventRecord(sync, 0);
	hipEventSynchronize(sync);

	CUDA_CHECK_RETURN(hipMemcpy(result, result_device, sizeof(CollisionPointsHost::CollisionPointsSize_t)*1, hipMemcpyDeviceToHost));

	hipFree( result_device );
	CollisionPointsHost::CollisionPointsSize_t value = *result;
	free(result);
	return value;
}

#ifdef CUDA
__global__ void testSumEnergy(CollisionPoints* pXS, gpuFloatType_t* results){
	gpuFloatType_t total = 0.0f;
	for(unsigned i=0; i< pXS->size(); ++i ) {
		total += pXS->getEnergy(i);
	}
	results[0] = total;
	return;
}
#endif

gpuFloatType_t
CollisionPointsTester::launchTestSumEnergy( unsigned nBlocks, unsigned nThreads, CollisionPointsHost& CPs) {
	gpuFloatType_t* result_device;
	gpuFloatType_t result[1];
	CUDA_CHECK_RETURN( hipMalloc( &result_device, sizeof( gpuFloatType_t) * 1 ));

	hipEvent_t sync;
	hipEventCreate(&sync);
	testSumEnergy<<<nBlocks,nThreads>>>(CPs.ptrPoints_device, result_device);
	gpuErrchk( hipPeekAtLastError() );
	hipEventRecord(sync, 0);
	hipEventSynchronize(sync);

	CUDA_CHECK_RETURN(hipMemcpy(result, result_device, sizeof(gpuFloatType_t)*1, hipMemcpyDeviceToHost));

	hipFree( result_device );
	return result[0];
}

CollisionPointsTester::CollisionPointsTester(){
}

CollisionPointsTester::~CollisionPointsTester(){
//		hipDeviceReset();
}

void CollisionPointsTester::setupTimers(){
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
}

void CollisionPointsTester::stopTimers(){
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	float elapsedTime;

	hipEventElapsedTime(&elapsedTime, start, stop );

	std::cout << "Elapsed time in CUDA kernel=" << elapsedTime << " msec" << std::endl;

}


