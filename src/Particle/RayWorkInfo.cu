#include "RayWorkInfo.hh"
#include "MonteRayCopyMemory.t.hh"

namespace MonteRay {

CUDAHOST_CALLABLE_MEMBER
RayWorkInfo::RayWorkInfo(unsigned num, bool cpuAllocate) {
    if( num == 0 ) { num = 1; }
    if( num > MONTERAY_MAX_THREADS ) {
        std::cout << "WARNING: Limiting MonteRay RayWorkInfo size, requested size="
                  << num << ", limit=" << MONTERAY_MAX_THREADS << "\n";
        num = MONTERAY_MAX_THREADS;
    }

#ifdef DEBUG
    if( Base::debug ) {
        std::cout << "RayWorkInfo::RayWorkInfo(n), n=" << num << " \n";
    }
#endif
    allocateOnCPU = cpuAllocate;

    reallocate( num );
}

CUDAHOST_CALLABLE_MEMBER
void
RayWorkInfo::reallocate(unsigned n) {
    if( indices          != NULL ) { MonteRayHostFree( indices,          Base::isManagedMemory ); }
    if( rayCastSize      != NULL ) { MonteRayHostFree( rayCastSize,      Base::isManagedMemory ); }
    if( rayCastCell      != NULL ) { MonteRayHostFree( rayCastCell,      Base::isManagedMemory ); }
    if( rayCastDistance  != NULL ) { MonteRayHostFree( rayCastDistance,  Base::isManagedMemory ); }
    if( crossingSize     != NULL ) { MonteRayHostFree( crossingSize,     Base::isManagedMemory ); }
    if( crossingCell     != NULL ) { MonteRayHostFree( crossingCell,     Base::isManagedMemory ); }
    if( crossingDistance != NULL ) { MonteRayHostFree( crossingDistance, Base::isManagedMemory ); }

    init();

    if( allocateOnCPU ) {
        // only allocate if not using CUDA -- very big
        indices = (int*) MONTERAYHOSTALLOC( n*3*sizeof( int ), Base::isManagedMemory, "host RayWorkInfo::indices" );
        rayCastSize = (int*) MONTERAYHOSTALLOC( n*sizeof( int ), Base::isManagedMemory, "host RayWorkInfo::rayCastSize" );
        rayCastCell = (int*) MONTERAYHOSTALLOC( n*MAXNUMRAYCELLS*sizeof( int ), Base::isManagedMemory, "host RayWorkInfo::rayCastCell" );
        rayCastDistance = (gpuRayFloat_t*) MONTERAYHOSTALLOC( n*MAXNUMRAYCELLS*sizeof( gpuRayFloat_t ), Base::isManagedMemory, "host RayWorkInfo::rayCastDistance" );
        crossingSize = (int*) MONTERAYHOSTALLOC( n*3*sizeof( int ), Base::isManagedMemory, "host RayWorkInfo::crossingSize" );
        crossingCell = (int*) MONTERAYHOSTALLOC( n*MAXNUMVERTICES*3*sizeof( int ), Base::isManagedMemory, "host RayWorkInfo::crossingCell" );
        crossingDistance = (gpuRayFloat_t*) MONTERAYHOSTALLOC( n*MAXNUMVERTICES*3*sizeof( gpuRayFloat_t ), Base::isManagedMemory, "host RayWorkInfo::crossingDistance" );
    }

    nAllocated = n;
    clear();
}

CUDAHOST_CALLABLE_MEMBER
RayWorkInfo::~RayWorkInfo(){
    if( ! Base::isCudaIntermediate ) {
        MonteRayHostFree(indices, Base::isManagedMemory );
        MonteRayHostFree(rayCastSize, Base::isManagedMemory );
        MonteRayHostFree(rayCastCell, Base::isManagedMemory );
        MonteRayHostFree(rayCastDistance, Base::isManagedMemory );
        MonteRayHostFree(crossingSize, Base::isManagedMemory );
        MonteRayHostFree(crossingCell, Base::isManagedMemory );
        MonteRayHostFree(crossingDistance, Base::isManagedMemory );
    } else {
        MonteRayDeviceFree( indices );
        MonteRayDeviceFree( rayCastSize );
        MonteRayDeviceFree( rayCastCell );
        MonteRayDeviceFree( rayCastDistance );
        MonteRayDeviceFree( crossingSize );
        MonteRayDeviceFree( crossingCell );
        MonteRayDeviceFree( crossingDistance );
    }
}

CUDAHOST_CALLABLE_MEMBER void
RayWorkInfo::copy(const RayWorkInfo* rhs) {

#ifdef __HIPCC__

#ifdef DEBUG
    if( Base::debug ) {
        std::cout << "Debug: RayWorkInfo::copy (const RayWorkInfo& rhs) \n";
    }
#endif

    if( Base::isCudaIntermediate && rhs->isCudaIntermediate ) {
        throw std::runtime_error("RayWorkInfo::copy -- can NOT copy CUDA intermediate to CUDA intermediate.");
    }

    if( !Base::isCudaIntermediate && !rhs->isCudaIntermediate ) {
        throw std::runtime_error("RayWorkInfo::copy -- can NOT copy CUDA non-intermediate to CUDA non-intermediate.");
    }

    if( nAllocated > 0 && nAllocated != rhs->nAllocated) {
        throw std::runtime_error("RayWorkInfo::copy -- can NOT change the size of the RayList.");
    }

    if( Base::isCudaIntermediate ) {
        // target is the intermediate, origin is the host
        if( indices == NULL ) {
            indices = (int*) MONTERAYDEVICEALLOC( rhs->nAllocated*sizeof(int)*3, "device - RayWorkInfo::indices" );
        }

        if( rayCastSize == NULL ) {
            rayCastSize = (int*) MONTERAYDEVICEALLOC( rhs->nAllocated*sizeof(int), "device - RayWorkInfo::rayCastSize" );
        }
        //hipMemset(rayCastSize, 0, rhs->nAllocated*sizeof(int));

        if( rayCastCell == NULL ) {
            rayCastCell = (int*) MONTERAYDEVICEALLOC( rhs->nAllocated*sizeof(int)*MAXNUMRAYCELLS, "device - RayWorkInfo::rayCastCell" );
        }

        if( rayCastDistance == NULL ) {
            rayCastDistance = (gpuRayFloat_t*) MONTERAYDEVICEALLOC( rhs->nAllocated*sizeof(gpuRayFloat_t)*MAXNUMRAYCELLS, "device - RayWorkInfo::rayCastDistance" );
        }

        if( crossingSize == NULL ) {
            crossingSize = (int*) MONTERAYDEVICEALLOC( rhs->nAllocated*sizeof(int)*3, "device - RayWorkInfo::crossingSize" );
        }
        //hipMemset(crossingSize, 0, rhs->nAllocated*3*sizeof(int));

        if( crossingCell == NULL ) {
            crossingCell = (int*) MONTERAYDEVICEALLOC( rhs->nAllocated*sizeof(int)*MAXNUMVERTICES*3, "device - RayWorkInfo::crossingCell" );
        }

        if( crossingDistance == NULL ) {
            crossingDistance = (gpuRayFloat_t*) MONTERAYDEVICEALLOC( rhs->nAllocated*sizeof(gpuRayFloat_t)*MAXNUMVERTICES*3, "device - RayWorkInfo::crossingDistance" );
        }

        // initialize the crossing info
//        MonteRayMemcpy(rayCastSize, rhs->rayCastSize, rhs->nAllocated*sizeof(int), hipMemcpyHostToDevice);
//        MonteRayMemcpy(crossingSize, rhs->crossingSize, rhs->nAllocated*sizeof(int)*3, hipMemcpyHostToDevice);
    }else {
        // device to host
        if( allocateOnCPU ) {
            MonteRayMemcpy( indices, rhs->indices, rhs->nAllocated*sizeof(int)*3, hipMemcpyDeviceToHost );
            MonteRayMemcpy( rayCastSize, rhs->rayCastSize, rhs->nAllocated*sizeof(int), hipMemcpyDeviceToHost );
            MonteRayMemcpy( rayCastCell, rhs->rayCastCell, rhs->nAllocated*sizeof(int)*MAXNUMRAYCELLS, hipMemcpyDeviceToHost );
            MonteRayMemcpy( rayCastDistance, rhs->rayCastDistance, rhs->nAllocated*sizeof(gpuRayFloat_t)*MAXNUMRAYCELLS, hipMemcpyDeviceToHost );
            MonteRayMemcpy( crossingSize, rhs->crossingSize, rhs->nAllocated*sizeof(int)*3, hipMemcpyDeviceToHost );
            MonteRayMemcpy( crossingCell, rhs->crossingCell, rhs->nAllocated*sizeof(int)*MAXNUMVERTICES*3, hipMemcpyDeviceToHost );
            MonteRayMemcpy( crossingDistance, rhs->crossingDistance, rhs->nAllocated*sizeof(gpuRayFloat_t)*MAXNUMVERTICES*3, hipMemcpyDeviceToHost );
        }
    }

    nAllocated = rhs->nAllocated;
#else
    throw std::runtime_error("RayWorkInfo::copy -- Only valid when compiling with CUDA.");
#endif
}

CUDAHOST_CALLABLE_MEMBER void
RayWorkInfo::addRayCastCell(unsigned i, int cellID, gpuRayFloat_t dist) {
#ifdef DEBUG
    if( dist < 0.0 ) {
        printf("Debug:  ERROR: RayWorkInfo::addRayCastCell, distance < 0, threadID=%u, cellID=%d, distance=%f\n", i, cellID, dist );
    }
#endif
    MONTERAY_ASSERT_MSG( dist >= 0, "distance must be > 0.0!" );
    getRayCastCell(i, getRayCastSize(i) ) = cellID;
    getRayCastDist(i, getRayCastSize(i) ) = dist;
    ++(getRayCastSize(i));
}

CUDAHOST_CALLABLE_MEMBER void
RayWorkInfo::addCrossingCell(unsigned dim, unsigned i, int cellID, gpuRayFloat_t dist) {
#ifdef DEBUG
    if( dist < 0.0 ) {
        printf("Debug:  ERROR: RayWorkInfo::addCrossingCell, distance < 0, threadID=%u, cellID=%d, distance=%f\n", i, cellID, dist );
    }
#endif
    MONTERAY_ASSERT_MSG( dist >= 0, "distance must be > 0.0!" );
    getCrossingCell(dim, i, getCrossingSize(dim,i) ) = cellID;
    getCrossingDist(dim, i, getCrossingSize(dim,i) ) = dist;
    ++(getCrossingSize(dim,i));
}

template class CopyMemoryBase<RayWorkInfo>;

} // end namespace
