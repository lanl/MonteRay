#include "MonteRayDefinitions.hh"
#include "GPUErrorCheck.hh"
#include "GPUAtomicAdd.hh"
#include "ExpectedPathLength.t.hh"
#include "HashLookup.hh"
#include "RayList.hh"
#include "GPUUtilityFunctions.hh"

#include "fi_genericGPU_test_helper.hh"

template<unsigned N>
FIGenericGPUTestHelper<N>::FIGenericGPUTestHelper(unsigned num){
    nCells = num;
}

template<unsigned N>
FIGenericGPUTestHelper<N>::~FIGenericGPUTestHelper(){
    if( tally ) {
        free( tally );
    }
}

template<unsigned N>
void FIGenericGPUTestHelper<N>::setupTimers(){
#ifdef __HIPCC__
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
#else
    timer.start();
#endif
}

template<unsigned N>
void FIGenericGPUTestHelper<N>::stopTimers(){
#ifdef __HIPCC__
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float elapsedTime;
    gpuErrchk( hipPeekAtLastError() );

    hipEventElapsedTime(&elapsedTime, start, stop );

    std::cout << "Elapsed time in CUDA kernel=" << elapsedTime << " msec" << std::endl;

    gpuErrchk( hipPeekAtLastError() );
#else
    timer.stop();
    std::cout << "Elapsed time in non-CUDA kernel=" << timer.getTime()*1000.0 << " msec" << std::endl;
#endif
}

template<unsigned N>
CUDA_CALLABLE_KERNEL  testTallyCrossSection(const RayList_t<N>* pCP, const MonteRayCrossSection* pXS, gpuTallyType_t* results){

#ifdef __HIPCC__
    int tid = threadIdx.x + blockIdx.x*blockDim.x;
#else
    int tid = 0;
#endif

    int num = pCP->size();
    while( tid < num ) {
        gpuFloatType_t E = pCP->getEnergy(tid);
        results[tid] = getTotalXS(pXS, E);
#ifdef __HIPCC__
        tid += blockDim.x*gridDim.x;
#else
        tid++;
#endif
    }
    return;
}

template CUDA_CALLABLE_KERNEL 
testTallyCrossSection<1>(const RayList_t<1>* pCP, const MonteRayCrossSection* pXS, gpuTallyType_t* results);

template CUDA_CALLABLE_KERNEL 
testTallyCrossSection<3>(const RayList_t<3>* pCP, const MonteRayCrossSection* pXS, gpuTallyType_t* results);

template<unsigned N>
void FIGenericGPUTestHelper<N>::launchTallyCrossSection(
        unsigned nBlocks, unsigned nThreads,
        const RayListInterface<N>* pCP,
        const MonteRayCrossSectionHost* pXS )
        {
    unsigned long long allocSize = sizeof(gpuTallyType_t)*nCells;

    if( tally ) {
        free( tally );
    }
    tally = (gpuTallyType_t*) malloc ( allocSize );
    for( unsigned i=0; i<nCells; ++i) {
        tally[i] = 0.0;
    }

#ifdef __HIPCC__
    auto launchBounds = setLaunchBounds( nThreads, nBlocks, pCP->getPtrPoints()->size() );
    nBlocks = launchBounds.first;
    nThreads = launchBounds.second;

    gpuTallyType_t* tally_device;
    CUDA_CHECK_RETURN( hipMalloc( &tally_device, allocSize ));
    CUDA_CHECK_RETURN( hipMemset(tally_device, 0, allocSize));
    gpuErrchk( hipPeekAtLastError() );

    hipEvent_t sync;
    hipEventCreate(&sync);
    testTallyCrossSection<N><<<nBlocks,nThreads>>>(pCP->getPtrPoints()->devicePtr, pXS->xs_device, tally_device);
    hipEventRecord(sync, 0);
    hipEventSynchronize(sync);
    gpuErrchk( hipPeekAtLastError() );

    CUDA_CHECK_RETURN(hipMemcpy(tally, tally_device, allocSize, hipMemcpyDeviceToHost));
    gpuErrchk( hipPeekAtLastError() );
    hipFree( tally_device );
#else
    testTallyCrossSection<N>(pCP->getPtrPoints(), pXS->getPtr(), tally);
#endif
    return;
        }

template< unsigned N>
CUDA_CALLABLE_KERNEL  testTallyCrossSection(
        const RayList_t<N>* pCP,
        const MonteRayMaterialList* pMatList,
        unsigned matIndex,
        const HashLookup* pHash,
        gpuFloatType_t density,
        gpuTallyType_t* results)
{
#ifdef __HIPCC__
    int tid = threadIdx.x + blockIdx.x*blockDim.x;
#else
    int tid = 0;
#endif
    int num = pCP->size();
    while( tid < num ) {
        gpuFloatType_t E = pCP->getEnergy(tid);
        unsigned HashBin = getHashBin( pHash, E);
        results[tid] = getTotalXS(pMatList, matIndex, pHash, HashBin, E, density);
#ifdef __HIPCC__
        tid += blockDim.x*gridDim.x;
#else
        tid++;
#endif
    }
    return;
}

template<unsigned N>
void FIGenericGPUTestHelper<N>::launchTallyCrossSection(
        unsigned nBlocks, unsigned nThreads,
        const RayListInterface<N>* pCP,
        const MonteRayMaterialListHost* pMatList,
        unsigned matIndex,
        gpuFloatType_t density )
        {
    unsigned long long allocSize = sizeof(gpuTallyType_t)*nCells;
    tally = (gpuTallyType_t*) malloc ( allocSize );

#ifdef __HIPCC__
    auto launchBounds = setLaunchBounds( nThreads, nBlocks, pCP->getPtrPoints()->size() );
    nBlocks = launchBounds.first;
    nThreads = launchBounds.second;

    gpuTallyType_t* tally_device;
    CUDA_CHECK_RETURN( hipMalloc( &tally_device, allocSize ));
    CUDA_CHECK_RETURN( hipMemset(tally_device, 0, allocSize));
    gpuErrchk( hipPeekAtLastError() );

    hipEvent_t sync;
    hipEventCreate(&sync);
    testTallyCrossSection<N><<<nBlocks,nThreads>>>(pCP->getPtrPoints()->devicePtr, pMatList->ptr_device,
            matIndex, pMatList->getHashPtr()->getPtrDevice(), density, tally_device);
    hipEventRecord(sync, 0);
    hipEventSynchronize(sync);
    gpuErrchk( hipPeekAtLastError() );

    CUDA_CHECK_RETURN(hipMemcpy(tally, tally_device, allocSize, hipMemcpyDeviceToHost));
    gpuErrchk( hipPeekAtLastError() );
    hipFree( tally_device );
#else
    testTallyCrossSection<N>(pCP->getPtrPoints(),
            pMatList->getPtr(),
            matIndex,
            pMatList->getHashPtr()->getPtr(),
            density,
            tally);
#endif
return;
        }


CUDA_CALLABLE_MEMBER
gpuFloatType_t getTotalXSByMatProp(
        const MaterialProperties* pMatProps,
        const MonteRayMaterialList* pMatList,
        const HashLookup* pHash,
        unsigned HashBin,
        unsigned cell,
        gpuFloatType_t E)
{
    gpuFloatType_t total = 0.0f;
    for( unsigned i=0; i< pMatProps->numMaterials(cell); ++i) {
        gpuFloatType_t density = pMatProps->getMaterialDensity(cell,i);
        unsigned matID = pMatProps->getMaterialID(cell,i);
        //		unsigned materialIndex = materialIDtoIndex(pMatList, matID);
        total += getTotalXS(pMatList, matID, pHash, HashBin, E, density);
    }
    return total;
}

gpuFloatType_t nonCudaGetTotalXSByMatProp(
        const MaterialProperties* pMatProps,
        const MonteRayMaterialList* pMatList,
        const HashLookup* pHash,
        unsigned HashBin,
        unsigned cell,
        gpuFloatType_t E)
{
    gpuFloatType_t total = 0.0f;
    for( unsigned i=0; i< pMatProps->numMaterials(cell); ++i) {
        gpuFloatType_t density = pMatProps->getMaterialDensity(cell,i);
        MaterialProperties::MatID_t matID = pMatProps->getMaterialID(cell,i);
        //unsigned materialIndex = materialIDtoIndex(pMatList, matID);
        total += getTotalXS(pMatList, matID, pHash, HashBin, E, density);
    }
    return total;
}

gpuFloatType_t nonCudaGetTotalXSByMatProp(
        const MaterialProperties* pMatProps,
        const MonteRayMaterialList* pMatList,
        unsigned cell,
        gpuFloatType_t E)
{
    gpuFloatType_t total = 0.0f;
    for( unsigned i=0; i< pMatProps->numMaterials(cell); ++i) {
        gpuFloatType_t density = pMatProps->getMaterialDensity(cell,i);
        unsigned matID = pMatProps->getMaterialID(cell,i);
        //unsigned materialIndex = materialIDtoIndex(pMatList, matID);
        total += getTotalXS(pMatList, matID, E, density);
    }
    return total;
}

template<unsigned N>
gpuFloatType_t
FIGenericGPUTestHelper<N>::getTotalXSByMatProp(
        const MaterialProperties* pMatProps,
        const MonteRayMaterialList* pMatList,
        const HashLookup* pHash,
        unsigned HashBin,
        unsigned cell,
        gpuFloatType_t E)
        {
    return nonCudaGetTotalXSByMatProp( pMatProps, pMatList, pHash, HashBin, cell, E);
        }

template<unsigned N>
gpuFloatType_t
FIGenericGPUTestHelper<N>::getTotalXSByMatProp(
        const MaterialProperties* pMatProps,
        const MonteRayMaterialList* pMatList,
        unsigned cell,
        gpuFloatType_t E)
        {
    return nonCudaGetTotalXSByMatProp( pMatProps, pMatList, cell, E);
        }

template<unsigned N>
CUDA_CALLABLE_KERNEL 
testTallyCrossSectionAtCollision(
        const RayList_t<N>* pCP,
        const MonteRayMaterialList* pMatList,
        const MaterialProperties* pMatProps,
        const HashLookup* pHash,
        gpuTallyType_t* results)
{
#ifdef __HIPCC__
    unsigned tid = threadIdx.x + blockIdx.x*blockDim.x;
#else
    unsigned tid = 0;
#endif

    unsigned num = pCP->size();

    while( tid < num ) {
        gpuFloatType_t E = pCP->getEnergy(tid);
        unsigned HashBin = getHashBin( pHash, E);
        unsigned cell = pCP->getIndex(tid);

        results[tid] = getTotalXSByMatProp(pMatProps, pMatList, pHash, HashBin, cell, E);
#ifdef __HIPCC__
        tid += blockDim.x*gridDim.x;
#else
        tid++;
#endif
    }
    return;
}


template<unsigned N>
CUDA_CALLABLE_KERNEL  testSumCrossSectionAtCollisionLocation(
        const RayList_t<N>* pCP,
        const MonteRayMaterialList* pMatList,
        const MaterialProperties* pMatProps,
        const HashLookup* pHash,
        gpuTallyType_t* results)
{
#ifdef __HIPCC__
    unsigned tid = threadIdx.x + blockIdx.x*blockDim.x;
#else
    unsigned tid = 0;
#endif

    unsigned num = pCP->size();

    while( tid < num ) {
        gpuFloatType_t E = pCP->getEnergy(tid);
        unsigned HashBin = getHashBin( pHash, E);
        unsigned cell = pCP->getIndex(tid);

        gpuTallyType_t value = getTotalXSByMatProp(pMatProps, pMatList, pHash, HashBin, cell, E);

        gpu_atomicAdd( &results[cell], value);
#ifdef __HIPCC__
        tid += blockDim.x*gridDim.x;
#else
        tid++;
#endif
    }
    return;
}

template<unsigned N>
void FIGenericGPUTestHelper<N>::launchTallyCrossSectionAtCollision(
        unsigned nBlocks, unsigned nThreads,
        const RayListInterface<N>* pCP,
        const MonteRayMaterialListHost* pMatList,
        const MaterialProperties* pMatProps )
        {

    unsigned long long allocSize = sizeof(gpuTallyType_t)*nCells;
    tally = (gpuTallyType_t*) malloc ( allocSize );

#ifdef __HIPCC__
    auto launchBounds = setLaunchBounds( nThreads, nBlocks, pCP->getPtrPoints()->size() );
    nBlocks = launchBounds.first;
    nThreads = launchBounds.second;

    gpuTallyType_t* tally_device;
    CUDA_CHECK_RETURN( hipMalloc( &tally_device, allocSize ));
    CUDA_CHECK_RETURN( hipMemset(tally_device, 0, allocSize));
    gpuErrchk( hipPeekAtLastError() );

    hipEvent_t sync;
    hipEventCreate(&sync);
    testTallyCrossSectionAtCollision<<<nBlocks,nThreads>>>(pCP->getPtrPoints()->devicePtr,
            pMatList->ptr_device, pMatProps, pMatList->getHashPtr()->getPtrDevice(),
            tally_device);
    hipEventRecord(sync, 0);
    hipEventSynchronize(sync);
    gpuErrchk( hipPeekAtLastError() );

    CUDA_CHECK_RETURN(hipMemcpy(tally, tally_device, allocSize, hipMemcpyDeviceToHost));
    gpuErrchk( hipPeekAtLastError() );
    hipFree( tally_device );
#else
    testTallyCrossSectionAtCollision(
            pCP->getPtrPoints(),
            pMatList->getPtr(),
            pMatProps,
            pMatList->getHashPtr()->getPtr(),
            tally);
#endif
    return;
        }

template<unsigned N>
void FIGenericGPUTestHelper<N>::launchSumCrossSectionAtCollisionLocation(
        unsigned nBlocks,
        unsigned nThreads,
        const RayListInterface<N>* pCP,
        const MonteRayMaterialListHost* pMatList,
        const MaterialProperties* pMatProps )
        {

    unsigned long long allocSize = sizeof(gpuTallyType_t)*nCells;
    if( tally ) {
        free( tally );
    }
    tally = (gpuTallyType_t*) malloc ( allocSize );
    for( unsigned i=0; i < nCells; ++i ) {
        tally[i] = 0.0;
    }

#ifdef __HIPCC__
    auto launchBounds = setLaunchBounds( nThreads, nBlocks, pCP->getPtrPoints()->size() );
    nBlocks = launchBounds.first;
    nThreads = launchBounds.second;

    gpuTallyType_t* tally_device;
    CUDA_CHECK_RETURN( hipMalloc( &tally_device, allocSize ));
    CUDA_CHECK_RETURN( hipMemset(tally_device, 0, allocSize));
    gpuErrchk( hipPeekAtLastError() );

    hipEvent_t sync;
    hipEventCreate(&sync);
    testSumCrossSectionAtCollisionLocation<<<nBlocks,nThreads>>>(pCP->getPtrPoints()->devicePtr,
            pMatList->ptr_device, pMatProps, pMatList->getHashPtr()->getPtrDevice(),
            tally_device);
    hipEventRecord(sync, 0);
    hipEventSynchronize(sync);
    gpuErrchk( hipPeekAtLastError() );

    CUDA_CHECK_RETURN(hipMemcpy(tally, tally_device, allocSize, hipMemcpyDeviceToHost));
    gpuErrchk( hipPeekAtLastError() );
    hipFree( tally_device );
#else
    testSumCrossSectionAtCollisionLocation(
            pCP->getPtrPoints(),
            pMatList->getPtr(),
            pMatProps,
            pMatList->getHashPtr()->getPtr(),
            tally);
#endif

    return;
        }

template<unsigned N>
void FIGenericGPUTestHelper<N>::launchRayTraceTally(
        unsigned nBlocks,
        unsigned nThreads,
        const RayListInterface<N>* pCP,
        const MonteRayMaterialListHost* pMatList,
        const MaterialProperties* pMatProps )
        {

#ifdef __HIPCC__
    gpuErrchk( hipPeekAtLastError() );
#endif

    unsigned long long allocSize = sizeof(gpuTallyType_t)*nCells;
    tally = (gpuTallyType_t*) malloc ( allocSize );
    for( unsigned i = 0; i < nCells; ++i ) {
        tally[i] = 0.0;
    }

#ifdef __HIPCC__
    gpuTallyType_t* tally_device;
    CUDA_CHECK_RETURN( hipMalloc( &tally_device, allocSize ));
    CUDA_CHECK_RETURN(hipMemcpy(tally_device, tally, allocSize, hipMemcpyHostToDevice));

    std::cout << "Debug: FIGenericGPUTestHelper::launchRayTraceTally, requesting kernel with " <<
                    nBlocks << " blocks, " << nThreads << " threads, nBlocks*nThreads= " <<
                    nBlocks*nThreads << ", to process " << pCP->getPtrPoints()->size() << "rays. \n";

    auto launchBounds = setLaunchBounds( nThreads, nBlocks,  pCP->getPtrPoints()->size() );
    nThreads = launchBounds.second;
    nBlocks = launchBounds.first;

    std::cout << "Debug: FIGenericGPUTestHelper::launchRayTraceTally, launching kernel with " <<
                 nBlocks << " blocks, " << nThreads << " threads, nBlocks*nThreads= " <<
                 nBlocks*nThreads << "\n";

    RayWorkInfo rayInfo( nThreads*nBlocks );
    auto pRayInfo = std::make_unique<RayWorkInfo>(nThreads*nBlocks);

    gpuErrchk( hipPeekAtLastError() );

    hipEvent_t sync;
    hipEventCreate(&sync);
    rayTraceTally<<<nBlocks,nThreads>>>(
            grid_device,
            pCP->getPtrPoints()->devicePtr,
            pMatList->ptr_device,
            pMatProps,
            pMatList->getHashPtr()->getPtrDevice(),
            pRayInfo.get(),
            tally_device);
    hipEventRecord(sync, 0);
    hipEventSynchronize(sync);
    gpuErrchk( hipPeekAtLastError() );

    CUDA_CHECK_RETURN(hipMemcpy(tally, tally_device, allocSize, hipMemcpyDeviceToHost));
    gpuErrchk( hipPeekAtLastError() );
    hipFree( tally_device );
#else
    RayWorkInfo rayInfo( 1 );

    rayTraceTally(
            grid_device,
            pCP->getPtrPoints(),
            pMatList->getPtr(),
            pMatProps,
            pMatList->getHashPtr()->getPtr(),
            &rayInfo,
            tally);
#endif
    return;
        }

template<unsigned N>
void FIGenericGPUTestHelper<N>::copyGridtoGPU(GridBins* grid){
    // copy the grid to the device
#ifdef __HIPCC__
    grid->copyToGPU();
    grid_device = grid->devicePtr;
#else
    grid_device = grid;
#endif

    nCells = grid->getNumCells();
}

template class FIGenericGPUTestHelper<1>;
template class FIGenericGPUTestHelper<3>;


