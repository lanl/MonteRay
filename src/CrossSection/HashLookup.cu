#include "hip/hip_runtime.h"
#include "HashLookup.h"

#include <math.h>

#include "GPUErrorCheck.hh"
#include "MonteRayDefinitions.hh"
#include "MonteRayConstants.hh"

#include "MonteRay_binaryIO.hh"
#include "MonteRayCrossSection.hh"

namespace MonteRay{

void ctor(HashLookup* ptr, unsigned num, unsigned nBins ) {
    if( num <=0 ) { num = 1; }
    if( nBins <=2 ) { nBins = 3; }

    ptr->maxNumIsotopes = num;
    ptr->numIsotopes = 0;
    ptr->eMax = 0.0f;
    ptr->eMin = MonteRay::inf;
    ptr->N = nBins;

    unsigned allocSize = sizeof(unsigned)*nBins*num;

    ptr->binBounds  = (unsigned*) malloc( allocSize);
    if(ptr->binBounds == 0) abort ();

    for( unsigned i=0; i<num*nBins; ++i ){
    	ptr->binBounds[i] = 0U;
    }
}


void cudaCtor(HashLookup* pCopy, unsigned num, unsigned nBins) {
#ifdef __HIPCC__
	pCopy->maxNumIsotopes = num;
	pCopy->N = nBins;

	// binBounds
	unsigned allocSize = sizeof(unsigned)*num*nBins;
	CUDA_CHECK_RETURN( hipMalloc(&pCopy->binBounds, allocSize ));
#endif
}

void cudaCtor(struct HashLookup* pCopy, struct HashLookup* pOrig){
#ifdef __HIPCC__
	cudaCtor( pCopy, pOrig->maxNumIsotopes, pOrig->N);
	pCopy->maxNumIsotopes = pOrig->maxNumIsotopes;
	pCopy->numIsotopes = pOrig->numIsotopes;
	pCopy->eMin = pOrig->eMin;
	pCopy->eMax = pOrig->eMax;
	pCopy->delta = pOrig->delta;
#endif
}

void dtor(HashLookup* ptr) {
    if( ptr->binBounds != 0 ) {
        free( ptr->binBounds );
        ptr->binBounds = 0;
    }
}

void cudaDtor(HashLookup* ptr) {
#ifdef __HIPCC__
	hipFree( ptr->binBounds );
#endif
}

HashLookupHost::HashLookupHost(unsigned num, unsigned nBins) {
     ptr = new HashLookup;
     ctor( ptr, num, nBins);
     temp = NULL;
     ptr_device = NULL;
     cudaCopyMade = false;
}


HashLookupHost::~HashLookupHost() {
     dtor( ptr );
     delete ptr;

#ifdef __HIPCC__
     if( cudaCopyMade ) {
       	cudaDtor( temp );
       	delete temp;
     	hipFree( ptr_device );
     }
#endif
}


void HashLookupHost::copyToGPU(void) {
#ifdef __HIPCC__
	cudaCopyMade = true;
    temp = new HashLookup;
//    copy(temp, ptr);

	unsigned num = ptr->maxNumIsotopes;

	// allocate target struct
	CUDA_CHECK_RETURN( hipMalloc(&ptr_device, sizeof( HashLookup ) ));

	// allocate target dynamic memory
	cudaCtor( temp, ptr);

	unsigned allocSize = sizeof(unsigned)*num*ptr->N;
	CUDA_CHECK_RETURN( hipMemcpy(temp->binBounds, ptr->binBounds, allocSize, hipMemcpyHostToDevice));

	// copy data
	CUDA_CHECK_RETURN( hipMemcpy(ptr_device, temp, sizeof( HashLookup ), hipMemcpyHostToDevice));
#endif
}

void copy(HashLookup* pCopy, const HashLookup* const pOrig ) {
    unsigned num = pOrig->maxNumIsotopes;
    if( num <=0 ) { num = 1; }

    ctor( pCopy, num, pOrig->N);

	pCopy->numIsotopes = pOrig->numIsotopes;
	pCopy->eMin = pOrig->eMin;
	pCopy->eMax = pOrig->eMax;
	pCopy->delta = pOrig->delta;

    for( unsigned i=0; i<num*pOrig->N; ++i ){
        pCopy->binBounds[i] = pOrig->binBounds[i];
    }
}

CUDA_CALLABLE_MEMBER
unsigned getMaxNumIsotopes(const HashLookup* ptr ) {
    return ptr->maxNumIsotopes;
}

CUDA_CALLABLE_MEMBER
unsigned getNumIsotopes(const HashLookup* ptr ) {
    return ptr->numIsotopes;
}

CUDA_CALLABLE_MEMBER
gpuFloatType_t getMaxEnergy(const HashLookup* ptr ) {
	return std::exp(ptr->eMax);
}

CUDA_CALLABLE_MEMBER
gpuFloatType_t getMinEnergy(const HashLookup* ptr ) {
	return std::exp(ptr->eMin);
}

CUDA_CALLABLE_MEMBER
unsigned getNBins(const HashLookup* ptr ) {
	return ptr->N;
}

unsigned HashLookupHost::getNBins(void) {
	return MonteRay::getNBins( ptr );
}

CUDA_CALLABLE_MEMBER
bool setHashMinMax(HashLookup* ptr, MonteRayCrossSection* xs ) {
	setID(xs, ptr->numIsotopes );

	ptr->numIsotopes++;
	unsigned numIsotopes = ptr->numIsotopes;
	unsigned numBins = ptr->N;
	if( numIsotopes > ptr->maxNumIsotopes ) {
		printf("Error: HasLookup::addIsotope -- exceeded max number of isotopes. %s %d", __FILE__, __LINE__);
		return true;
	}
	if(xs->energies[0] <= 0.0  ) {
		printf("Error: HasLookup::addIsotope -- minimum cross-section can not be zero or less. %s %d", __FILE__, __LINE__);
		return true;
	}

	if( logf(xs->energies[0]) < ptr->eMin ) { ptr->eMin = logf(xs->energies[0]); }
	if( logf(xs->energies[ xs->numPoints -1]) > ptr->eMax ) { ptr->eMax = logf(xs->energies[xs->numPoints-1]); }

	if( ptr->eMin > ptr->eMax ) {
		printf("Error: HasLookup::addIsotope -- min energy > max energy %s %d", __FILE__, __LINE__);
		return true;
	}
	ptr->delta = (ptr->eMax - ptr->eMin)/numBins;
	return false;
}

CUDA_CALLABLE_MEMBER
void setHashBinBounds(HashLookup* ptr, MonteRayCrossSection* xs, unsigned j ) {
	for( unsigned i = 0; i < ptr->N; ++i ){
		unsigned index = getBinBoundIndex(ptr, j, i);
		gpuFloatType_t hashEnergy = std::exp( ptr->eMin + i*ptr->delta);
		ptr->binBounds[index] = getIndex(xs, hashEnergy);
	}
}

CUDA_CALLABLE_MEMBER
unsigned getBinBoundIndex(const HashLookup* ptr, unsigned isotope, unsigned index ){
	if( isotope > ptr->numIsotopes) {
		printf("Error: HasLookup::getBinBoundIndex -- isotope ( = %d )  > numIsotopes (= %d), %s %d\n", isotope, ptr->numIsotopes, __FILE__, __LINE__);
		ABORT( "HashLookup.cu -- getBinBoundIndex" );
	}
	if( index > ptr->N) {
		printf("Error: HasLookup::getBinBoundIndex -- index ( = %d )  > numBins (= %d), %s %d\n", index, ptr->N, __FILE__, __LINE__);
		ABORT( "HashLookup.cu -- getBinBoundIndex" );
	}
	unsigned i = isotope + index*ptr->maxNumIsotopes;
	if( i >= ptr->maxNumIsotopes*ptr->N ){
		printf("Error: HasLookup::getBinBoundIndex -- index outside of range. isotope = %d, index=%d, %s %d\n", isotope, index, __FILE__, __LINE__);
		printf("Error: HasLookup::getBinBoundIndex -- index outside of range. i = %d, N*maxNumIsotopes=%d,\n", i, ptr->maxNumIsotopes*ptr->N  );
		printf("Error: HasLookup::getBinBoundIndex -- index outside of range. N = %d, maxNumIsotopes=%d,\n", ptr->N, ptr->maxNumIsotopes  );
		ABORT( "HashLookup.cu -- getBinBoundIndex" );
	}
	return i;
}


void HashLookupHost::addIsotope( MonteRayCrossSectionHost* xs ) {
	addIsotope( xs->getXSPtr() );
}

void HashLookupHost::addIsotope( MonteRayCrossSection* xs ) {
	xsList.push_back(xs);
	if( xs->id < 0 ) {
		bool err = MonteRay::setHashMinMax(ptr, xs );
		if( err ) {
			throw std::runtime_error( "Error:  HashLookupHost::addIsotope -- setHashMinMax failure.\n");
		}
	}
	for( unsigned i=0; i<xsList.size(); ++i) {
		MonteRay::setHashBinBounds( ptr, xsList.at(i), i);
	}
}

CUDA_CALLABLE_MEMBER
unsigned getHashBin(const HashLookup* ptr, gpuFloatType_t energy ) {
	gpuFloatType_t logE = logf(energy);
	if( logE <= ptr->eMin) { return 0; }
	if( logE >= ptr->eMax ) { return ptr->N-1; }
	return (logE-ptr->eMin)/ptr->delta;
}

CUDA_CALLABLE_MEMBER
unsigned getLowerBoundbyIndex(const HashLookup* ptr, unsigned isotope, unsigned index ){
	return ptr->binBounds[getBinBoundIndex( ptr, isotope, index )];
}

CUDA_CALLABLE_MEMBER
unsigned getUpperBoundbyIndex(const HashLookup* ptr, unsigned isotope, unsigned index ){
	if( index < ptr->N - 1 ) {
		return getLowerBoundbyIndex( ptr, isotope, index+1) + 1;
	}
	return index;
}

unsigned HashLookupHost::getLowerBoundbyIndex( unsigned isotope, unsigned index) const {
	return MonteRay::getLowerBoundbyIndex(ptr, isotope, index);
}

unsigned HashLookupHost::getUpperBoundbyIndex( unsigned isotope, unsigned index) const {
	return MonteRay::getUpperBoundbyIndex(ptr, isotope, index);
}

}
