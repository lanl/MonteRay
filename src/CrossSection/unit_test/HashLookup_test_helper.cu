#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <iostream>

#include "MonteRayDefinitions.hh"
#include "GPUErrorCheck.hh"

#include "HashLookup_test_helper.hh"


#ifdef CUDA
__global__ void kernelGetLowerBoundbyIndex(HashLookup* pHash, unsigned isotope, unsigned bin, unsigned* result){
    result[0] = getLowerBoundbyIndex( pHash, isotope, bin);
    return;
}
#endif

unsigned
HashLookupTestHelper::launchGetLowerBoundbyIndex( HashLookupHost* pHash, unsigned isotope, unsigned bin){
	unsigned* result_device;
	unsigned result[1];
	CUDA_CHECK_RETURN( hipMalloc( &result_device, sizeof( unsigned) * 1 ));

	hipEvent_t sync;
	hipEventCreate(&sync);
	kernelGetLowerBoundbyIndex<<<1,1>>>( pHash->ptr_device, isotope, bin, result_device);
	gpuErrchk( hipPeekAtLastError() );
	hipEventRecord(sync, 0);
	hipEventSynchronize(sync);

	CUDA_CHECK_RETURN(hipMemcpy(result, result_device, sizeof(unsigned)*1, hipMemcpyDeviceToHost));

	hipFree( result_device );
	return result[0];
}

HashLookupTestHelper::HashLookupTestHelper(){
}

HashLookupTestHelper::~HashLookupTestHelper(){

//	std::cout << "Debug: starting ~MonteRayCrossSectionTestHelper()" << std::endl;
//	std::cout << "Debug: exitting ~MonteRayCrossSectionTestHelper()" << std::endl;
}

void HashLookupTestHelper::setupTimers(){
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
}

void HashLookupTestHelper::stopTimers(){
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	float elapsedTime;

	hipEventElapsedTime(&elapsedTime, start, stop );

	std::cout << "Elapsed time in CUDA kernel=" << elapsedTime << " msec" << std::endl;
}


