#include "MonteRayDefinitions.hh"

#include "MonteRayCrossSection_test_helper.hh"


//MonteRayCrossSectionTestHelper::float_t
//MonteRayCrossSectionTestHelper::launchGetTotalXS( MonteRayCrossSectionHost* pXS, float_t energy){
//	float_t* result_device;
//	float_t result[1];
//	CUDA_CHECK_RETURN( hipMalloc( &result_device, sizeof( float_t) * 1 ));
//
//	hipEvent_t sync;
//	hipEventCreate(&sync);
//	kernelGetTotalXS<<<1,1>>>( pXS->xs_device, energy, result_device);
//  gpuErrchk( hipPeekAtLastError() );
//	hipEventRecord(sync, 0);
//	hipEventSynchronize(sync);
//
//	CUDA_CHECK_RETURN(hipMemcpy(result, result_device, sizeof(float_t)*1, hipMemcpyDeviceToHost));
//
//	hipFree( result_device );
//	return result[0];
//}

MonteRayCrossSectionTestHelper::MonteRayCrossSectionTestHelper(){
}

MonteRayCrossSectionTestHelper::~MonteRayCrossSectionTestHelper(){

//	std::cout << "Debug: starting ~MonteRayCrossSectionTestHelper()" << std::endl;
//	std::cout << "Debug: exitting ~MonteRayCrossSectionTestHelper()" << std::endl;
}

void MonteRayCrossSectionTestHelper::setupTimers(){
#ifdef __HIPCC__
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
#else
	timer.start();
#endif
}

void MonteRayCrossSectionTestHelper::stopTimers(){
	float elapsedTime;

#ifdef __HIPCC__
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop );
	std::cout << "Elapsed time in CUDA kernel=" << elapsedTime << " msec" << std::endl;
#else
	timer.stop();
	std::cout << "Elapsed time in non-CUDA kernel=" << timer.getTime()*1000.0 << " msec" << std::endl;
#endif


}


