#include "hip/hip_runtime.h"
#include "MonteRayCrossSection.hh"

#include <stdlib.h>
#include <iostream>
#include <fstream>

#include "GPUErrorCheck.hh"
#include "MonteRay_binaryIO.hh"

namespace MonteRay{

void ctor(struct MonteRayCrossSection* pXS, unsigned num) {
    if( num <=0 ) { num = 1; }

    pXS->id = -1;
    pXS->numPoints = num;
    pXS->AWR = 0.0;

    unsigned allocSize = sizeof(gpuFloatType_t)*num;
    pXS->energies  = (gpuFloatType_t*) malloc( allocSize);
    if (pXS->energies == 0) abort ();

    pXS->totalXS   = (gpuFloatType_t*) malloc( allocSize );
    if (pXS->totalXS == 0) abort ();

    for( unsigned i=0; i<num; ++i ){
        pXS->energies[i] = -1.0;
        pXS->totalXS[i] =  0.0;
    }
}

void dtor(struct MonteRayCrossSection* pXS) {
    if( pXS->energies != 0 ) {
        free(pXS->energies);
        pXS->energies = 0;
    }
    if( pXS->totalXS != 0 ) {
        free(pXS->totalXS);
        pXS->totalXS = 0;
    }
}

#ifdef CUDA
void cudaCtor(MonteRayCrossSection* ptr, unsigned num) {

	 ptr->numPoints = num;
     unsigned allocSize = sizeof( gpuFloatType_t ) * num;

     CUDA_CHECK_RETURN( hipMalloc(&ptr->energies, allocSize ));

     CUDA_CHECK_RETURN( hipMalloc(&ptr->totalXS, allocSize ));
}

void cudaCtor(MonteRayCrossSection* pCopy, MonteRayCrossSection* pOrig) {
	unsigned num = pOrig->numPoints;
	cudaCtor( pCopy, num);

	pCopy->id = pOrig->id;
	pCopy->AWR = pOrig->AWR;

	unsigned allocSize = sizeof( gpuFloatType_t ) * pOrig->numPoints;

    CUDA_CHECK_RETURN( hipMemcpy(pCopy->energies, pOrig->energies, allocSize, hipMemcpyHostToDevice));

    CUDA_CHECK_RETURN( hipMemcpy(pCopy->totalXS, pOrig->totalXS, allocSize, hipMemcpyHostToDevice));
}

void cudaDtor(MonteRayCrossSection* ptr) {
    hipFree( ptr->energies );
    hipFree( ptr->totalXS );
}
#endif

void copy(struct MonteRayCrossSection* pCopy, struct MonteRayCrossSection* pOrig ) {
    unsigned num = pOrig->numPoints;
    if( num <=0 ) { num = 1; }

    ctor( pCopy, num);
    pCopy->id = pOrig->id;
    pCopy->AWR = pOrig->AWR;
    for( unsigned i=0; i<num; ++i ){
        pCopy->energies[i] = pOrig->energies[i];
        pCopy->totalXS[i] =  pOrig->totalXS[i];
    }
}

#ifdef CUDA
__device__ __host__
#endif
int getID(struct MonteRayCrossSection* pXS) {
	return pXS->id;
}

#ifdef CUDA
__device__ __host__
#endif
void setID(struct MonteRayCrossSection* pXS, unsigned i) {
	if( pXS->id < 0 ) {
		pXS->id = i;
	}
}

#ifdef CUDA
__device__ __host__
#endif
gpuFloatType_t getEnergy(struct MonteRayCrossSection* pXS, unsigned i ) {
    return pXS->energies[i];
}


#ifdef CUDA
__device__ __host__
#endif
gpuFloatType_t getTotalXSByIndex(struct MonteRayCrossSection* pXS, unsigned i ){
    return pXS->totalXS[i];
}

#ifdef CUDA
__device__ __host__
#endif
unsigned getIndex(struct MonteRayCrossSection* pXS, gpuFloatType_t value ){
    // modified from http://en.cppreference.com/w/cpp/algorithm/upper_bound
	return getIndexBinary( pXS, 0, pXS->numPoints-1, value);
}

#ifdef CUDA
__device__ __host__
#endif
unsigned getIndexBinary(struct MonteRayCrossSection* pXS, unsigned lower, unsigned upper, gpuFloatType_t value ){
    // modified from http://en.cppreference.com/w/cpp/algorithm/upper_bound
    unsigned it, step;
    unsigned first = lower;
    unsigned count = upper-lower+1;

    while (count > 0U) {
        it = first;
        step = count / 2;
        it += step;
        if(!(value < pXS->energies[it])) {
            first = ++it;
            count -= step + 1;
        } else {
            count = step;
        }
    }
    if( first > 0 ) { --first; }
    return first;
}

#ifdef CUDA
__device__ __host__
#endif
unsigned getIndexLinear(struct MonteRayCrossSection* pXS, unsigned lower, unsigned upper, gpuFloatType_t value ){

    for( unsigned i=lower+1; i < upper+1; ++i ){
    	if( value < pXS->energies[ i ] ) {
    		return i-1;
    	}
    }
    if( value < pXS->energies[ lower ] ) { return lower; }
    return upper;
}

#ifdef CUDA
__device__ __host__
#endif
unsigned getIndex(struct MonteRayCrossSection* pXS, struct HashLookup* pHash, unsigned hashBin, gpuFloatType_t E ){
	unsigned isotope = MonteRay::getID(pXS);
	unsigned lowerBin = MonteRay::getLowerBoundbyIndex(pHash, isotope, hashBin);
	unsigned upperBin = MonteRay::getUpperBoundbyIndex(pHash, isotope, hashBin);

	if( upperBin-lowerBin+1 <= 8 ){
		return getIndexLinear( pXS, lowerBin, upperBin, E);
	} else {
		return getIndexBinary( pXS, lowerBin, upperBin, E);
	}

}


#ifdef CUDA
__device__ __host__
#endif
gpuFloatType_t getAWR(struct MonteRayCrossSection* pXS) {
    return pXS->AWR;
}

#ifdef CUDA
__device__ __host__
#endif
gpuFloatType_t getTotalXSByIndex(struct MonteRayCrossSection* pXS, unsigned i, gpuFloatType_t E ) {

    gpuFloatType_t lower =  pXS->totalXS[i];
    gpuFloatType_t upper =  pXS->totalXS[i+1];
    gpuFloatType_t deltaE = pXS->energies[i+1] - pXS->energies[i];

    gpuFloatType_t value = lower + (upper-lower) * (E - pXS->energies[i])/deltaE;
    return value;
}

#ifdef CUDA
__device__ __host__
#endif
gpuFloatType_t getTotalXS(struct MonteRayCrossSection* pXS, gpuFloatType_t E ) {

    if( E > pXS->energies[ pXS->numPoints-1] ) {
        return pXS->totalXS[ pXS->numPoints-1];
    }

    if( E < pXS->energies[ 0 ] ) {
        return pXS->totalXS[ 0 ];
    }

    unsigned i = getIndex(pXS, E);
    return getTotalXSByIndex( pXS, i, E);
}

#ifdef CUDA
__device__ __host__
#endif
gpuFloatType_t getTotalXS(struct MonteRayCrossSection* pXS, struct HashLookup* pHash, unsigned hashBin, gpuFloatType_t E ) {

    if( E > pXS->energies[ pXS->numPoints-1] ) {
        return pXS->totalXS[ pXS->numPoints-1];
    }

    if( E < pXS->energies[ 0 ] ) {
        return pXS->totalXS[ 0 ];
    }

    unsigned i = getIndex(pXS, pHash, hashBin, E);
    return getTotalXSByIndex( pXS, i, E);
}

#ifdef CUDA
__global__ void kernelGetTotalXS(struct MonteRayCrossSection* pXS, HashLookup* pHash, unsigned HashBin, gpuFloatType_t E, gpuFloatType_t* results){
    results[0] = getTotalXS(pXS, pHash, HashBin, E);
    return;
}
#endif

#ifdef CUDA
__global__ void kernelGetTotalXS(struct MonteRayCrossSection* pXS,  gpuFloatType_t E, gpuFloatType_t* results){
    results[0] = getTotalXS(pXS, E);
    return;
}
#endif

gpuFloatType_t
launchGetTotalXS( MonteRayCrossSectionHost* pXS, gpuFloatType_t energy){
#ifdef CUDA
	gpuFloatType_t* result_device;
	gpuFloatType_t result[1];
	CUDA_CHECK_RETURN( hipMalloc( &result_device, sizeof( gpuFloatType_t) * 1 ));

	hipEvent_t sync;
	hipEventCreate(&sync);
	kernelGetTotalXS<<<1,1>>>( pXS->xs_device, energy, result_device);
	gpuErrchk( hipPeekAtLastError() );
	hipEventRecord(sync, 0);
	hipEventSynchronize(sync);

	CUDA_CHECK_RETURN(hipMemcpy(result, result_device, sizeof(gpuFloatType_t)*1, hipMemcpyDeviceToHost));

	hipFree( result_device );
	return result[0];
#else
	return -100.0;
#endif
}


#if !defined( CUDA )
#include "ContinuousNeutron.hh"
void MonteRayCrossSectionHost::load( const ContinuousNeutron& cn){
    unsigned num = cn.getEnergyGrid().GridSize();
    dtor( xs );
    ctor( xs, num );

    gpuFloatType_t ratio = cn.getAWR();
    setAWR( ratio );

    for( unsigned i=0; i<num; ++i ){
        gpuFloatType_t energy = (cn.getEnergyGrid())[i];
        gpuFloatType_t totalXS = cn.TotalXsec( energy, -1.0, i);
        xs->energies[i] = energy;
        xs->totalXS[i] = totalXS;
    }

}
#endif

MonteRayCrossSectionHost::MonteRayCrossSectionHost(unsigned num){
    xs = (struct MonteRayCrossSection*) malloc( sizeof(struct MonteRayCrossSection) );
    ctor(xs,num);

    cudaCopyMade = false;
    temp = NULL;

#ifdef CUDA
    CUDA_CHECK_RETURN( hipMalloc(&xs_device, sizeof( MonteRayCrossSection) ));
#endif
}

MonteRayCrossSectionHost::~MonteRayCrossSectionHost(){
    dtor(xs);

    if( xs != 0 ) {
        free(xs);
        xs = 0;
    }

    if( cudaCopyMade ) {
        cudaDtor( temp );
        delete temp;
    }
#ifdef CUDA
    hipFree( xs_device );
#endif
}

gpuFloatType_t MonteRayCrossSectionHost::getTotalXS( struct HashLookup* pHash, unsigned hashBin, gpuFloatType_t E ) const {
	return MonteRay::getTotalXS(xs, pHash, hashBin, E);
}

gpuFloatType_t MonteRayCrossSectionHost::getTotalXSByHashIndex(struct HashLookup* pHash, unsigned i, gpuFloatType_t E) const {
	return MonteRay::getTotalXS(xs, pHash, i, E);
}

unsigned MonteRayCrossSectionHost::getIndex( HashLookupHost* pHost, unsigned hashBin, gpuFloatType_t e ) const {
	return MonteRay::getIndex( xs, pHost->getPtr(), hashBin, e);
}


void MonteRayCrossSectionHost::copyToGPU(void) {
#ifdef CUDA
    cudaCopyMade = true;
    temp = new MonteRayCrossSection;
    cudaCtor(temp, xs );
    CUDA_CHECK_RETURN( hipMemcpy(xs_device, temp, sizeof( MonteRayCrossSection ), hipMemcpyHostToDevice));
#endif
}

void MonteRayCrossSectionHost::load(struct MonteRayCrossSection* ptrXS ) {
    unsigned num = ptrXS->numPoints;
    dtor( xs );
    ctor( xs, num );

    setAWR( ptrXS->AWR );

    for( unsigned i=0; i<num; ++i ){
        gpuFloatType_t energy = ptrXS->energies[i];
        gpuFloatType_t totalXS = ptrXS->totalXS[i];
        xs->energies[i] = energy;
        xs->totalXS[i] = totalXS;
    }
}

void MonteRayCrossSectionHost::write(std::ostream& outf) const{
    binaryIO::write(outf, xs->numPoints );
    binaryIO::write(outf, xs->AWR );
    for( unsigned i=0; i<xs->numPoints; ++i ){
        binaryIO::write(outf, xs->energies[i] );
    }
    for( unsigned i=0; i<xs->numPoints; ++i ){
        binaryIO::write(outf, xs->totalXS[i] );
    }
}

void MonteRayCrossSectionHost::read(std::istream& infile) {
    unsigned num;
    binaryIO::read(infile, num);
    dtor( xs );
    ctor( xs, num );

    binaryIO::read(infile, xs->AWR );
    for( unsigned i=0; i<num; ++i ){
        binaryIO::read(infile, xs->energies[i] );
    }

    for( unsigned i=0; i<num; ++i ){
        binaryIO::read(infile, xs->totalXS[i] );
    }
}


void MonteRayCrossSectionHost::write( const std::string& filename ) {
    std::ofstream outfile;

    outfile.open( filename.c_str(), std::ios::binary | std::ios::out);
    if( ! outfile.is_open() ) {
        fprintf(stderr, "MonteRayCrossSectionHost::write -- Failure to open file,  filename=%s  %s %d\n", filename.c_str(), __FILE__, __LINE__);
        exit(1);
    }
    assert( outfile.good() );
    outfile.exceptions(std::ios_base::failbit | std::ios_base::badbit );
    write( outfile );
    outfile.close();
}

void MonteRayCrossSectionHost::read( const std::string& filename ) {
    std::ifstream infile;
    if( infile.is_open() ) {
        infile.close();
    }
    infile.open( filename.c_str(), std::ios::binary | std::ios::in);

    if( ! infile.is_open() ) {
        fprintf(stderr, "MonteRayCrossSectionHost::read -- Failure to open file,  filename=%s  %s %d\n", filename.c_str(), __FILE__, __LINE__);
        exit(1);
    }
    assert( infile.good() );
    infile.exceptions(std::ios_base::failbit | std::ios_base::badbit );
    read(infile);
    infile.close();
}

}
