#include "hip/hip_runtime.h"
#include <UnitTest++.h>

#include "CudaWrappers.hh"


SUITE(CudaStream_test) {
  TEST(ConstructorAndAssignment){
    MonteRay::cuda::StreamPointer sp;
    auto spCopy = sp;
    CHECK_EQUAL(sp.get(), spCopy.get());
    auto spMove = std::move(spCopy);
    CHECK_EQUAL(sp.get(), spMove.get());
    MonteRay::cuda::StreamPointer spCopyConstructed(sp);
    CHECK_EQUAL(sp.get(), spCopyConstructed.get());
    MonteRay::cuda::StreamPointer spMoveConstructed(std::move(spCopyConstructed));
    CHECK_EQUAL(sp.get(), spMoveConstructed.get());
  }
}

SUITE(CudaEvent_test) {
  TEST(ConstructorAndAssignment){
    MonteRay::cuda::EventPointer ep;
    auto epCopy = ep;
    CHECK_EQUAL(ep.get(), epCopy.get());
    auto epMove = std::move(epCopy);
    CHECK_EQUAL(ep.get(), epMove.get());
    MonteRay::cuda::EventPointer epCopyConstructed(ep);
    CHECK_EQUAL(ep.get(), epCopyConstructed.get());
    MonteRay::cuda::EventPointer epMoveConstructed(std::move(epCopyConstructed));
    CHECK_EQUAL(ep.get(), epMoveConstructed.get());
  }
}


#ifdef __HIPCC__
__global__ void testKernel(int* val){
  atomicAdd(val, 1);
}

SUITE(CudaStreamAndEventGPU_test){
  TEST(StreamAndEventUsage){
    MonteRay::cuda::StreamPointer pStream1;
    MonteRay::cuda::StreamPointer pStream2;
    MonteRay::cuda::EventPointer pEvent1;
    MonteRay::cuda::EventPointer pEvent2;

    int* val;
    hipMallocManaged(&val, sizeof(int));
    *val = 0;
    testKernel<<<1, 1, 0, *pStream1>>>(val);
    hipEventRecord(*pEvent1, *pStream1);
    hipEventSynchronize(*pEvent1);
    CHECK_EQUAL(1, *val);
    testKernel<<<1, 1, 0, *pStream2>>>(val);
    hipEventRecord(*pEvent2, *pStream2);
    hipEventSynchronize(*pEvent2);
    hipStreamWaitEvent(*pStream2, *pEvent2, 0);
    hipDeviceSynchronize();
    CHECK_EQUAL(2, *val);
    hipFree(val);
    CHECK(pEvent1.get() != pEvent2.get());
    CHECK(pStream1.get() != pStream2.get());
  }
}

#endif
