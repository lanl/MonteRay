#include "StreamAndEvent.hh"

namespace MonteRay{
namespace cuda{
StreamPointer::StreamPointer(): pStream_(std::make_shared<hipStream_t>()) {
#ifdef __HIPCC__
  hipStreamCreate(pStream_.get());
#else
  *pStream_ = 0;
#endif
}

StreamPointer::~StreamPointer() {
#ifdef __HIPCC__
  if (pStream_.use_count() == 1) {
    hipStreamDestroy(*pStream_);
  }
#endif
}

EventPointer::~EventPointer(){
#ifdef __HIPCC__
  if (pEvent_.use_count() == 1) {
    hipEventDestroy(*pEvent_);
  }
#endif
}

EventPointer::EventPointer(): pEvent_(std::make_shared<hipEvent_t>()) {
#ifdef __HIPCC__
  hipEventCreate(pEvent_.get());
#else
  *pEvent_ = 0;
#endif
}
}
}
