#include <iostream>

#include "MonteRayDefinitions.hh"
#include "GPUErrorCheck.hh"

#include "HashLookup_test_helper.hh"

CUDA_CALLABLE_KERNEL  kernelGetLowerBoundbyIndex(const HashLookup* pHash, unsigned isotope, unsigned bin, unsigned* result){
    result[0] = getLowerBoundbyIndex( pHash, isotope, bin);
    return;
}

unsigned
HashLookupTestHelper::launchGetLowerBoundbyIndex( const HashLookupHost* pHash, unsigned isotope, unsigned bin){
	unsigned result[1];

#ifdef __HIPCC__
	unsigned* result_device;
	CUDA_CHECK_RETURN( hipMalloc( &result_device, sizeof( unsigned) * 1 ));

	hipEvent_t sync;
	hipEventCreate(&sync);
	kernelGetLowerBoundbyIndex<<<1,1>>>( pHash->ptr_device, isotope, bin, result_device);
	gpuErrchk( hipPeekAtLastError() );
	hipEventRecord(sync, 0);
	hipEventSynchronize(sync);

	CUDA_CHECK_RETURN(hipMemcpy(result, result_device, sizeof(unsigned)*1, hipMemcpyDeviceToHost));

	hipFree( result_device );
#else
	kernelGetLowerBoundbyIndex( pHash->getPtr(), isotope, bin, result);
#endif

	return result[0];
}

HashLookupTestHelper::HashLookupTestHelper(){
}

HashLookupTestHelper::~HashLookupTestHelper(){

//	std::cout << "Debug: starting ~MonteRayCrossSectionTestHelper()" << std::endl;
//	std::cout << "Debug: exitting ~MonteRayCrossSectionTestHelper()" << std::endl;
}

void HashLookupTestHelper::setupTimers(){
#ifdef __HIPCC__
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
#else
	timer.start();
#endif
}

void HashLookupTestHelper::stopTimers(){
	float elapsedTime;

#ifdef __HIPCC__
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop );
	std::cout << "Elapsed time in CUDA kernel=" << elapsedTime << " msec" << std::endl;
#else
	timer.stop();
	std::cout << "Elapsed time in non-CUDA kernel=" << timer.getTime()*1000.0 << " msec" << std::endl;
#endif


}


