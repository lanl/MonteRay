#include "ManagedResource.hh"

#ifdef __HIPCC__
#include <thrust/system_error.h>
#include <thrust/system/cuda/error.h>
#endif

namespace MonteRay{

  void* ManagedResource::allocate(size_t n) {
#ifdef __HIPCC__
    void* result = nullptr;

    hipError_t error = hipMallocManaged( &result, n, hipMemAttachGlobal);

    if(error != hipSuccess) {
      throw thrust::system_error(error, thrust::cuda_category(), "managed_allocator::allocate(): hipMallocManaged");
    }
    return result;
#else
    return ::operator new(n);
#endif
  }

  void ManagedResource::deallocate(void* ptr, size_t) {
#ifdef __HIPCC__
    hipError_t error = hipFree(ptr);

    if(error != hipSuccess) {
      throw thrust::system_error(error, thrust::cuda_category(), "managed_allocator::deallocate(): hipFree");
    }
#else
    ::operator delete(ptr);
#endif
  }

}

