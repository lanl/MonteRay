#include "ManagedAllocator.hh"

#ifdef __HIPCC__
#include <thrust/system_error.h>
#include <thrust/system/cuda/error.h>
#endif

namespace MonteRay{

void* Managed::operator new(size_t len) {
#ifdef __HIPCC__
  void *ptr;
  hipMallocManaged(&ptr, len);
  hipDeviceSynchronize();
  return ptr;
#else
  return ::operator new(len);
#endif
}

void Managed::operator delete(void *ptr) {
#ifdef __HIPCC__
  hipDeviceSynchronize();
  hipFree(ptr);
#else
  ::operator delete(ptr);
#endif
}

} // end namespace MonteRay
