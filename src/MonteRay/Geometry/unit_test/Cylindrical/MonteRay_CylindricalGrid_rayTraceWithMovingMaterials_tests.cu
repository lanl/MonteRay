#include "hip/hip_runtime.h"
#include <UnitTest++.h>

#include "MonteRay_CylindricalGrid.t.hh"
#include "MonteRay_SpatialGrid.hh"
#include "MonteRayVector3D.hh"
#include "MonteRay_GridBins.hh"
#include "MonteRayDefinitions.hh"
#include "MonteRayCopyMemory.t.hh"
#include "RayWorkInfo.hh"
#include "MaterialProperties.hh"

namespace MonteRay_CylindricalGrid_rayTraceWithMovingMaterials_tests{

using namespace MonteRay;

SUITE( MonteRay_CylindricalGrid_rayTraceWithMovingMaterials_Tests) {

  using Position_t = Vector3D<gpuRayFloat_t>;
  using Direction_t = Vector3D<gpuRayFloat_t>;
  using GridBins_t = MonteRay_GridBins;

  enum cartesian_coord {x=0,y=1,z=2};
  enum cylindrical_coord {R=0,CZ=1,Theta=2,DIM=2};

  class GridTestData {
  public:

    GridTestData(){
      std::vector<gpuRayFloat_t> Rverts = { 1.0, 2.0 };
      std::vector<gpuRayFloat_t> Zverts = { -1, 0, 1};

      pGridInfo[R] = new GridBins_t();
      pGridInfo[CZ] = new GridBins_t();

      pGridInfo[R]->initialize( Rverts );
      pGridInfo[CZ]->initialize( Zverts );

      }
      ~GridTestData(){
          delete pGridInfo[R];
          delete pGridInfo[CZ];
      }

      MonteRay_SpatialGrid::pArrayOfpGridInfo_t pGridInfo;
  };

  class CylindricalGrid{
    public:
      gpuRayFloat_t two = 2.0;
      gpuRayFloat_t one = 1.0;
      GridTestData gridTestData;
      MonteRay_CylindricalGrid cyl;
      CylindricalGrid(): gridTestData(GridTestData{}), cyl(2,gridTestData.pGridInfo){
        int size = cyl.getNumBins(0)*cyl.getNumBins(1);
        CHECK_EQUAL(4, size);
      }
  };

  TEST_FIXTURE( CylindricalGrid, ConvertToCellReferenceFrame ) {
    Position_t pos(0.5, 0.0, 0.5);
    Direction_t dir(1.0,   0.0,    0.0);
    gpuRayFloat_t speed = 1.0;
    Direction_t velocity(1.0, 2.0, 3.0);
    auto newDirAndSpeed = cyl.convertToCellReferenceFrame(velocity, pos, dir, speed);
    CHECK_CLOSE(2.0, newDirAndSpeed.speed(), 1e-6);
    CHECK_CLOSE(0.0, newDirAndSpeed.direction()[0], 1e-6);
    CHECK_CLOSE(-1.0, newDirAndSpeed.direction()[2], 1e-6);
  }

  TEST_FIXTURE( CylindricalGrid, CalcIndices ){
    Position_t pos( 0.5, 0.0, 0.5 );
    auto indices = cyl.calcIndices(pos);
    CHECK_EQUAL(0, indices[0]);
    CHECK_EQUAL(1, indices[1]);
  }

  TEST_FIXTURE( CylindricalGrid, getMinRadialDistAndDir){
    Position_t pos = { -1.0, -1.0, 0.5};
    Direction_t dir = Direction_t{1.0, 1.0, 0.0}.normalize();
    auto indices = cyl.calcIndices(pos);
    auto distAndDir = cyl.getMinRadialDistAndDir(pos, dir, indices[R]);
    CHECK_CLOSE(sqrt(2) - 1.0, distAndDir.distance(), 1e-6);
    CHECK_EQUAL(R, distAndDir.dimension());
    CHECK_EQUAL(false, distAndDir.isPositiveDir());
  }

  TEST_FIXTURE( CylindricalGrid, GetMinDistToSurface){
    Position_t pos( 0.5, 0.0,  0.5 );
    Position_t dir(1, 0, 0);
    auto indices = cyl.calcIndices(pos);
    auto distAndDir = cyl.getMinDistToSurface(pos, dir, indices.data());
    CHECK_CLOSE(0.5, distAndDir.distance(), 1e-6);
    CHECK_EQUAL(R, distAndDir.dimension());
    CHECK_EQUAL(true, distAndDir.isPositiveDir());

    pos = { 0.0, 0.5,  0.5 };
    dir = {0, -1, 0};
    distAndDir = cyl.getMinDistToSurface(pos, dir, indices.data());
    CHECK_CLOSE(1.5, distAndDir.distance(), 1e-6);
    CHECK_EQUAL(R, distAndDir.dimension());
    CHECK_EQUAL(true, distAndDir.isPositiveDir());

    pos = { 0.0, 0.0,  0.5 };
    dir = {0, 0, 1};
    distAndDir = cyl.getMinDistToSurface(pos, dir, indices.data());
    CHECK_CLOSE(0.5, distAndDir.distance(), 1e-6);
    CHECK_EQUAL(CZ, distAndDir.dimension());
    CHECK_EQUAL(true, distAndDir.isPositiveDir());

    pos = { 0.0, 0.0,  0.5 };
    dir = {0, 0, -1};
    distAndDir = cyl.getMinDistToSurface(pos, dir, indices.data());
    CHECK_CLOSE(0.5, distAndDir.distance(), 1e-6);
    CHECK_EQUAL(CZ, distAndDir.dimension());
    CHECK_EQUAL(false, distAndDir.isPositiveDir());

    pos = { 1.1, 1.1,  0.5 };
    dir = {-1, 0, 0};
    indices = cyl.calcIndices(pos);
    distAndDir = cyl.getMinDistToSurface(pos, dir, indices.data());
    CHECK_CLOSE(1.1 + Math::sqrt(4.0 - 1.1*1.1), distAndDir.distance(), 1e-6);
    CHECK_EQUAL(R, distAndDir.dimension());
    CHECK_EQUAL(true, distAndDir.isPositiveDir());
  }

  TEST_FIXTURE( CylindricalGrid, isMovingInward){
    Position_t pos = { -1.5, -1.5, -0.75 };
    Direction_t dir = Direction_t{-Math::sqrt(two), -Math::sqrt(two), one}.normalize();
    CHECK(not cyl.isMovingInward(pos, dir));

    dir = Direction_t{Math::sqrt(two), Math::sqrt(two), one}.normalize();
    CHECK(cyl.isMovingInward(pos, dir));
  }

  TEST_FIXTURE( CylindricalGrid, DistanceToInsideOfMesh){
    Position_t pos = { -1.5, -1.5, -0.75 };
    Direction_t dir = Direction_t{-1.0, -1.0, 0.0};
    auto distanceToInsideOfMesh = cyl.getDistanceToInsideOfMesh(pos, dir);
    CHECK_EQUAL(std::numeric_limits<gpuRayFloat_t>::infinity(), distanceToInsideOfMesh);

    dir = Direction_t{Math::sqrt(two), Math::sqrt(two), one}.normalize();
    distanceToInsideOfMesh = cyl.getDistanceToInsideOfMesh(pos, dir);
    auto answer = (Math::sqrt(2*1.5*1.5) - 2.0)/Math::sqrt(dir[0]*dir[0] + dir[1]*dir[1]);
    CHECK_CLOSE(answer, distanceToInsideOfMesh, 1e-6);

    pos = { -1.5, -1.5, -1.5 };
    distanceToInsideOfMesh = cyl.getDistanceToInsideOfMesh(pos, dir);
    answer = 0.5/dir[2];
    CHECK_CLOSE(answer, distanceToInsideOfMesh, 1e-6);

    dir = Direction_t{Math::sqrt(two), Math::sqrt(two), -one}.normalize();
    distanceToInsideOfMesh = cyl.getDistanceToInsideOfMesh(pos, dir);
    CHECK_EQUAL(std::numeric_limits<gpuRayFloat_t>::infinity(), distanceToInsideOfMesh);
  }

  TEST_FIXTURE( CylindricalGrid, RayTraceWithNonMovingMaterials){
    gpuRayFloat_t dist = 10;

    std::unique_ptr<MaterialProperties> pMatProps;
    auto mpb = MaterialProperties::Builder();
    using Cell = MaterialProperties::Builder::Cell;
    Cell cell{ {1}, {1.0} }; // set material IDs and densities
    int size = 1;
    for (int i = 0; i < DIM; i++){
      size *= cyl.getNumBins(i);
    }
    for (int i = 0; i < size; i++){
      mpb.addCell(cell);
      mpb.setCellVelocity(i, {0.0, 0.0, 0.0});
    }
    pMatProps = std::make_unique<MaterialProperties>(mpb.build());

    RayWorkInfo rayInfo(1);
    const double speed = 1.0;
    const int threadID = 0;

    Position_t pos ( -2.5/Math::sqrt(2.0), -2.5/Math::sqrt(2.0), -1.0);
    Direction_t dir = Direction_t{Math::sqrt(two), Math::sqrt(two), one}.normalize();
    cyl.rayTraceWithMovingMaterials(threadID, rayInfo, pos, dir, dist, speed, *pMatProps);

    auto radialComponent = Math::sqrt(dir[0]*dir[0] + dir[1]*dir[1]);
    rayTraceList_t distances( rayInfo, threadID );
    CHECK_EQUAL( 4, distances.size() );

    CHECK_EQUAL( 1, distances.id(0) );
    CHECK_CLOSE( 1.0/radialComponent, distances.dist(0), 1e-6 );

    CHECK_EQUAL( 0, distances.id(1) );
    CHECK_CLOSE( 0.25/dir[z], distances.dist(1), 1e-6 );

    CHECK_EQUAL( 2, distances.id(2) );
    CHECK_CLOSE( 1.5/radialComponent, distances.dist(2), 1e-6 );

    CHECK_EQUAL( 3, distances.id(3) );
    CHECK_CLOSE( 0.25/dir[z], distances.dist(3), 1e-6 );

    rayInfo = RayWorkInfo{1};
    gpuRayFloat_t xPos = -2.5/Math::sqrt(2.0);
    pos = Position_t{ xPos, xPos, -1.0 };
    dir = Direction_t{-Math::sqrt(two), -Math::sqrt(two), one}.normalize();
    cyl.rayTraceWithMovingMaterials(threadID, rayInfo, pos, dir, dist, speed, *pMatProps);
    distances = rayTraceList_t( rayInfo, threadID );
    CHECK_EQUAL(0, distances.size());

    rayInfo = RayWorkInfo{1};
    pos = Position_t{ 100, 0.0, 0.5};
    dir = Direction_t{-1.0, 0.0, 0.0}.normalize();
    cyl.rayTraceWithMovingMaterials(threadID, rayInfo, pos, dir, dist, speed, *pMatProps);
    distances = rayTraceList_t( rayInfo, threadID );
    CHECK_EQUAL(0, distances.size());
  }

  TEST_FIXTURE( CylindricalGrid, RayTraceWithMovingMaterials){

    std::unique_ptr<MaterialProperties> pMatProps;
    auto mpb = MaterialProperties::Builder();
    using Cell = MaterialProperties::Builder::Cell;
    Cell cell{ {1}, {1.0} }; // set material IDs and densities
    // velocities are (r, z, t), t is not used
    mpb.addCell(cell);
    mpb.setCellVelocity(0, {0.0, -0.5, 0.0});
    mpb.addCell(cell);
    mpb.setCellVelocity(1, {0.0,  0.5, 0.0});
    mpb.addCell(cell);
    mpb.setCellVelocity(2, {-2.0, -0.5, 0.0});
    mpb.addCell(cell);
    mpb.setCellVelocity(3, {-2.0, 0.5, 0.0});

    pMatProps = std::make_unique<MaterialProperties>(mpb.build());

    RayWorkInfo rayInfo(1);
    const double speed = 1.0/sqrt(4.0/5.0);
    const int threadID = 0;
    gpuRayFloat_t dist = 10;

    Position_t pos ( -2.5/Math::sqrt(2.0), -2.5/Math::sqrt(2.0), -1.0);
    Direction_t dir = Direction_t{Math::sqrt(two), Math::sqrt(two), one}.normalize();
    cyl.rayTraceWithMovingMaterials(threadID, rayInfo, pos, dir, dist, speed, *pMatProps);

    rayTraceList_t distances( rayInfo, threadID );

    CHECK_EQUAL( 4, distances.size() );

    CHECK_EQUAL( 1, distances.id(0) );
    CHECK_CLOSE( 1.0, distances.dist(0), 1e-6 );

    CHECK_EQUAL( 0, distances.id(1) );
    CHECK_CLOSE( 0.75*sqrt(2.0), distances.dist(1), 1e-6 );

    CHECK_EQUAL( 2, distances.id(2) );
    CHECK_CLOSE( 0.75*sqrt(2.0), distances.dist(2), 1e-6 );

    CHECK_EQUAL( 3, distances.id(3) );
    CHECK_CLOSE( 1.0, distances.dist(3), 1e-6 );
  }
}

} // end namespace

