#include "hip/hip_runtime.h"
#include <UnitTest++.h>

#include "../MonteRay_SpatialGrid_GPU_helper.hh"
#include "MonteRay_CylindricalGrid.hh"

namespace MonteRay_CylindricalGrid_crossingDistance_GPU_tests{

using namespace MonteRay;
using namespace MonteRay_SpatialGrid_helper;

SUITE( MonteRay_CylindricalGrid_crossingDistance_GPU_Tests) {
#ifdef __HIPCC__
    using Grid_t = MonteRay_CylindricalGrid;
    using GridBins_t = MonteRay_GridBins;
    using GridBins_t = Grid_t::GridBins_t;
    using pGridInfo_t = GridBins_t*;
    using pArrayOfpGridInfo_t = Grid_t::pArrayOfpGridInfo_t;
    using Position_t = MonteRay::Vector3D<gpuRayFloat_t>;

    const gpuFloatType_t s2 = std::sqrt(2.0);

    enum coord {R=0,Z=1,Theta=2,DIM=3};

    inline void checkDistances( const char *file, int line,
            const std::vector<unsigned>& expectedIndex,
            const std::vector<gpuFloatType_t>& expectedDistance, const singleDimRayTraceMap_t& distances )
    {
        char const* const errorFormat = "%s(%d): error: Failure \n";
        if( expectedIndex.size() != expectedDistance.size() ) {
            printf(errorFormat, file, line);
        }
        CHECK_EQUAL( expectedIndex.size(), expectedDistance.size() );

        if( expectedIndex.size() != distances.size() ) {
            printf(errorFormat, file, line);
        }
        CHECK_EQUAL( expectedIndex.size(), distances.size() );

        for( auto i=0; i<distances.size(); ++i ) {
            if( expectedIndex[i] != distances.id(i) ) {
                printf("%s(%d): error: Failure in cell id #%d \n", file, line, i);
            }
            CHECK_EQUAL( expectedIndex   [i], distances.id(i) );

            if( std::abs( expectedDistance[i] - distances.dist(i) ) > 1.0e-5  ) {
                printf("%s(%d): error: Failure in distance #%d \n", file, line, i);
            }
            CHECK_CLOSE( expectedDistance[i], distances.dist(i), 1e-5 );
        }
    }

#define checkDistances(expectedIndex, expectedDistance, distances) { checkDistances(__FILE__, __LINE__, expectedIndex, expectedDistance, distances); }

    typedef singleDimRayTraceMap_t distances_t;
    typedef singleDimRayTraceMap_t rayTraceMap_t;
    typedef rayTraceList_t rayTrace_t;
    typedef MonteRay_CylindricalGrid CylindricalGrid;

    TEST( setup ) {
        //gpuReset();
    }


    // ************************ rayTrace Testing ****************************

    TEST_FIXTURE(SpatialGridGPUTester, CrossingDistance_in_1D_R_inward_from_outside_to_outside ) {
        std::vector<gpuRayFloat_t> Rverts = { 1.0, 2.0, 3.0, 5.0 };
        std::vector<gpuRayFloat_t> Zverts = { 0.0, 1.0, 2.0, 3.0, 5.0 };

        cylindricalGrid_setup(Rverts, Zverts);

        Position_t position (  -6.5, 0.0,  0.5 );
        Position_t direction(    1,   0,    0 );
        gpuFloatType_t distance = 100.0;

        //distances_t distances = crossingDistance( 0, position[0], direction[0], distance );
        distances_t distances = crossingDistance( R, position, direction, distance );

        CHECK_EQUAL( 9,  distances.size() );
        checkDistances( std::vector<unsigned>({4,3,2,1,0,1,2,3,4}),
                std::vector<gpuFloatType_t>({1.5,3.5,4.5,5.5,7.5,8.5,9.5,11.5,distance}),
                distances );
    }


    TEST_FIXTURE(SpatialGridGPUTester, CrossingDistance_in_1D_R_inward_from_outside_to_inside_stop_inward ) {

        std::vector<gpuRayFloat_t> Rverts = { 1.0, 2.0, 3.0, 5.0 };
        std::vector<gpuRayFloat_t> Zverts = { 0.0, 1.0, 2.0, 3.0, 5.0 };

        cylindricalGrid_setup(Rverts, Zverts);

        Position_t position (  -6.5, 0.0,  0.5 );
        Position_t direction(    1,   0,    0 );
        gpuFloatType_t distance = 6.0;

        distances_t distances = crossingDistance( R, position, direction, distance );

        CHECK_EQUAL( 5,  distances.size() );
        checkDistances( std::vector<unsigned>({4,3,2,1,0}),
                std::vector<gpuFloatType_t>({1.5,3.5,4.5,5.5,6.0}),
                distances );
    }

    TEST_FIXTURE(SpatialGridGPUTester, CrossingDistance_in_1D_R_inward_from_outside_to_inside_stop_outward ) {

        std::vector<gpuRayFloat_t> Rverts = { 1.0, 2.0, 3.0, 5.0 };
        std::vector<gpuRayFloat_t> Zverts = { 0.0, 1.0, 2.0, 3.0, 5.0 };

        cylindricalGrid_setup(Rverts, Zverts);

        Position_t position (  -6.5, 0.0,  0.5 );
        Position_t direction(    1,   0,    0 );
        gpuFloatType_t distance = 9.0;

        distances_t distances = crossingDistance( R, position, direction, distance );

        CHECK_EQUAL( 7,  distances.size() );
        checkDistances( std::vector<unsigned>({4,3,2,1,0,1,2}),
                std::vector<gpuFloatType_t>({1.5,3.5,4.5,5.5,7.5,8.5,9.0}),
                distances );
    }

    TEST_FIXTURE(SpatialGridGPUTester, CrossingDistance_through_a_single_cylinder_in_2D_R_inward_from_inside_to_outside ) {

        std::vector<gpuRayFloat_t> Rverts = { 1.0, 2.0, 3.0, 5.0 };
        std::vector<gpuRayFloat_t> Zverts = { 0.0, 1.0, 2.0, 3.0, 5.0 };

        cylindricalGrid_setup(Rverts, Zverts);

        gpuFloatType_t y = 3.0f / std::sqrt(2.0f );
        gpuFloatType_t last_dist = std::sqrt( 25 - y*y );
        Position_t position (  -4.0, y,  0.5 );
        Position_t direction(    1,   0,    0 );
        gpuFloatType_t distance = 9.0;

        distances_t distances = crossingDistance( R, position, direction, distance );

        CHECK_EQUAL( 4,  distances.size() );
        checkDistances( std::vector<unsigned>({3,2,3,4}),
                std::vector<gpuFloatType_t>({4.0f-y,4.0f+y,4.0f+last_dist,9.0}),
                distances );
    }


    TEST_FIXTURE(SpatialGridGPUTester, CrossingDistance_tanget_to_first_inner_cylinder_posY ) {
        std::vector<gpuRayFloat_t> Rverts = { 1.0, 2.0, 3.0, 5.0 };
        std::vector<gpuRayFloat_t> Zverts = { 0.0, 1.0, 2.0, 3.0, 5.0 };

        cylindricalGrid_setup(Rverts, Zverts);

        gpuFloatType_t x = -3.5;
        gpuFloatType_t y = 3.0;
        gpuFloatType_t last_dist = std::sqrt( 25 - y*y );

        Position_t position (  x, y, 0.5 );
        Position_t direction(  1, 0,   0 );
        gpuFloatType_t distance = 9.0;

        distances_t distances = crossingDistance( R, position, direction, distance );

        CHECK_EQUAL( 4, distances.size() );
        CHECK_EQUAL( 3, distances.id(0) );
        CHECK_CLOSE( 3.5, distances.dist(0), 1e-5 );
        CHECK_EQUAL( 2, distances.id(1) );
        CHECK_CLOSE( 3.5, distances.dist(1), 1e-5 );
        CHECK_EQUAL( 3, distances.id(2) );
        CHECK_CLOSE( 7.5, distances.dist(2), 1e-5 );
        CHECK_EQUAL( 4, distances.id(3) );
        CHECK_CLOSE( 9.0, distances.dist(3), 1e-5 );

    }

    TEST_FIXTURE(SpatialGridGPUTester, CrossingDistance_tanget_to_first_inner_cylinder_negY ) {
        std::vector<gpuRayFloat_t> Rverts = { 1.0, 2.0, 3.0, 5.0 };
        std::vector<gpuRayFloat_t> Zverts = { 0.0, 1.0, 2.0, 3.0, 5.0 };

        cylindricalGrid_setup(Rverts, Zverts);

        gpuFloatType_t x = -3.5;
        gpuFloatType_t y = -3.0;
        gpuFloatType_t last_dist = std::sqrt( 25 - y*y );

        Position_t position (  x, y, 0.5 );
        Position_t direction(    1,   0,    0 );
        gpuFloatType_t distance = 9.0;

        distances_t distances = crossingDistance( R, position, direction, distance );

        CHECK_EQUAL( 4, distances.size() );
        CHECK_EQUAL( 3, distances.id(0) );
        CHECK_CLOSE( 3.5, distances.dist(0), 1e-5 );
        CHECK_EQUAL( 2, distances.id(1) );
        CHECK_CLOSE( 3.5, distances.dist(1), 1e-5 );
        CHECK_EQUAL( 3, distances.id(2) );
        CHECK_CLOSE( 7.5, distances.dist(2), 1e-5 );
        CHECK_EQUAL( 4, distances.id(3) );
        CHECK_CLOSE( 9.0, distances.dist(3), 1e-5 );


//        checkDistances( std::vector<unsigned>({3,4}),
//                std::vector<gpuFloatType_t>({3.5f+last_dist, 9.0}),
//                distances );
    }

    TEST_FIXTURE(SpatialGridGPUTester, CrossingDistance_tanget_to_first_second_cylinder_posY ) {
        std::vector<gpuRayFloat_t> Rverts = { 1.0, 2.0, 3.0, 5.0 };
        std::vector<gpuRayFloat_t> Zverts = { 0.0, 1.0, 2.0, 3.0, 5.0 };

        cylindricalGrid_setup(Rverts, Zverts);

        gpuFloatType_t y = 2.0;
        Position_t position (  -4.0, y,  0.5 );
        Position_t direction(    1,   0,    0 );
        gpuFloatType_t distance = 9.0;

        distances_t distances = crossingDistance( R, position, direction, distance );

        CHECK_EQUAL( 6, distances.size() );
        CHECK_EQUAL( 3, distances.id(0) );
        CHECK_CLOSE( 4.0 - std::sqrt(9.0-4.0), distances.dist(0), 1e-5 );
        CHECK_EQUAL( 2, distances.id(1) );
        CHECK_CLOSE( 4.0, distances.dist(1), 1e-5 );
        CHECK_EQUAL( 1, distances.id(2) );
        CHECK_CLOSE( 4.0, distances.dist(2), 1e-5 );
        CHECK_EQUAL( 2, distances.id(3) );
        CHECK_CLOSE( 4.0 + std::sqrt(9.0-4.0), distances.dist(3), 1e-5 );
        CHECK_EQUAL( 3, distances.id(4) );
        CHECK_CLOSE( 4.0 + std::sqrt(25.0-4.0), distances.dist(4), 1e-5 );
        CHECK_EQUAL( 4, distances.id(5) );
        CHECK_CLOSE( distance, distances.dist(5), 1e-5 );
    }

    TEST_FIXTURE(SpatialGridGPUTester, CrossingDistance_outward_from_Origin_posX_to_outside ) {
        //        std::cout << "Debug: ---------------------------------------------------------" << std::endl;
        std::vector<gpuRayFloat_t> Rverts = { 1.0, 2.0, 3.0, 5.0 };
        std::vector<gpuRayFloat_t> Zverts = { 0.0, 1.0, 2.0, 3.0, 5.0 };

        cylindricalGrid_setup(Rverts, Zverts);

        Position_t position (  0.0, 0.0,  0.5 );
        Position_t direction(    1,   0,    0 );
        gpuFloatType_t distance = 9.0;

        distances_t distances = crossingDistance( R, position, direction, distance );

        CHECK_EQUAL( 5, distances.size() );
        CHECK_EQUAL( 0, distances.id(0) );
        CHECK_CLOSE( 1.0, distances.dist(0), 1e-5 );
        CHECK_EQUAL( 1, distances.id(1) );
        CHECK_CLOSE( 2.0, distances.dist(1), 1e-5 );
        CHECK_EQUAL( 2, distances.id(2) );
        CHECK_CLOSE( 3.0, distances.dist(2), 1e-5 );
        CHECK_EQUAL( 3, distances.id(3) );
        CHECK_CLOSE( 5.0, distances.dist(3), 1e-5 );
        CHECK_EQUAL( 4, distances.id(4) );
        CHECK_CLOSE( 9.0, distances.dist(4), 1e-5 );
    }

    TEST_FIXTURE(SpatialGridGPUTester, CrossingDistance_outward_from_Origin_posX_to_inside ) {
        //        std::cout << "Debug: ---------------------------------------------------------" << std::endl;
        std::vector<gpuRayFloat_t> Rverts = { 1.0, 2.0, 3.0, 5.0 };
        std::vector<gpuRayFloat_t> Zverts = { 0.0, 1.0, 2.0, 3.0, 5.0 };

        cylindricalGrid_setup(Rverts, Zverts);

        Position_t position (  0.0, 0.0,  0.5 );
        Position_t direction(    1,   0,    0 );
        gpuFloatType_t distance = 4.5;

        distances_t distances = crossingDistance( R, position, direction, distance );

        CHECK_EQUAL( 4, distances.size() );
        CHECK_EQUAL( 0, distances.id(0) );
        CHECK_CLOSE( 1.0, distances.dist(0), 1e-5 );
        CHECK_EQUAL( 1, distances.id(1) );
        CHECK_CLOSE( 2.0, distances.dist(1), 1e-5 );
        CHECK_EQUAL( 2, distances.id(2) );
        CHECK_CLOSE( 3.0, distances.dist(2), 1e-5 );
        CHECK_EQUAL( 3, distances.id(3) );
        CHECK_CLOSE( 4.5, distances.dist(3), 1e-5 );
    }

    TEST_FIXTURE(SpatialGridGPUTester, CrossingDistance_outward_from_posX_Postion_negX_Direction ) {
        //        std::cout << "Debug: ---------------------------------------------------------" << std::endl;
        std::vector<gpuRayFloat_t> Rverts = { 1.0, 2.0, 3.0, 5.0 };
        std::vector<gpuRayFloat_t> Zverts = { 0.0, 1.0, 2.0, 3.0, 5.0 };

        cylindricalGrid_setup(Rverts, Zverts);

        Position_t position (  3.5, 0.0,  0.5 );
        Position_t direction(   -1,   0,    0 );
        gpuFloatType_t distance = 9.0;

        distances_t distances = crossingDistance( R, position, direction, distance );

        CHECK_EQUAL( 8,  distances.size() );
        checkDistances( std::vector<unsigned>({3,2,1,0,1,2,3,4}),
                std::vector<gpuFloatType_t>({0.5, 1.5, 2.5, 4.5, 5.5, 6.5, 8.5, 9.0}),
                distances );
    }

    TEST_FIXTURE(SpatialGridGPUTester, CrossingDistance_outward_from_posX_Postion_negX_Direction_not_outside ) {
        // std::cout << "Debug: ---------------------------------------------------------" << std::endl;
        std::vector<gpuRayFloat_t> Rverts = { 1.0, 2.0, 3.0, 5.0 };
        std::vector<gpuRayFloat_t> Zverts = { 0.0, 1.0, 2.0, 3.0, 5.0 };

        cylindricalGrid_setup(Rverts, Zverts);

        Position_t position (  3.5, 0.0,  0.5 );
        Position_t direction(   -1,   0,    0 );
        gpuFloatType_t distance = 7.5;

        distances_t distances = crossingDistance( R, position, direction, distance );

        CHECK_EQUAL( 7,  distances.size() );
        checkDistances( std::vector<unsigned>({3,2,1,0,1,2,3}),
                std::vector<gpuFloatType_t>({0.5, 1.5, 2.5, 4.5, 5.5, 6.5, 7.5}),
                distances );
    }

    TEST_FIXTURE(SpatialGridGPUTester, radialCrossingDistances_inside_thru_to_outside ) {
        std::vector<gpuRayFloat_t> Rverts = { 1.0, 2.0, 3.0, 5.0 };
        std::vector<gpuRayFloat_t> Zverts = { 0.0, 1.0, 2.0, 3.0, 5.0 };

        cylindricalGrid_setup(Rverts, Zverts);

        Position_t position (  -4.5, 0.0,  0.5 );
        Position_t direction(    1,   0,    0 );
        gpuFloatType_t distance = 100.0;

        distances_t distances = crossingDistance( R, position, direction, distance );

        CHECK_EQUAL( 8, distances.size() );
        CHECK_EQUAL( 3, distances.id(0) );
        CHECK_CLOSE( 1.5, distances.dist(0), 1e-5 );
        CHECK_EQUAL( 2, distances.id(1) );
        CHECK_CLOSE( 2.5, distances.dist(1), 1e-5 );
        CHECK_EQUAL( 1, distances.id(2) );
        CHECK_CLOSE( 3.5, distances.dist(2), 1e-5 );
        CHECK_EQUAL( 0, distances.id(3) );
        CHECK_CLOSE( 5.5, distances.dist(3), 1e-5 );
        CHECK_EQUAL( 1, distances.id(4) );
        CHECK_CLOSE( 6.5, distances.dist(4), 1e-5 );
        CHECK_EQUAL( 2, distances.id(5) );
        CHECK_CLOSE( 7.5, distances.dist(5), 1e-5 );
        CHECK_EQUAL( 3, distances.id(6) );
        CHECK_CLOSE( 9.5, distances.dist(6), 1e-5 );
        CHECK_EQUAL( 4, distances.id(7) );
        CHECK_CLOSE( 100.0, distances.dist(7), 1e-5 );
    }

    TEST_FIXTURE(SpatialGridGPUTester, radialCrossingDistances_inside_misses_inner_cells ) {
        std::vector<gpuRayFloat_t> Rverts = { 1.0, 2.0, 3.0, 5.0 };
        std::vector<gpuRayFloat_t> Zverts = { 0.0, 1.0, 2.0, 3.0, 5.0 };

        cylindricalGrid_setup(Rverts, Zverts);

        Position_t position (  -3.5, 3.1,  0.5 );
        Position_t direction(    1,   0,    0 );
        gpuFloatType_t distance = 100.0;

        distances_t distances = crossingDistance( R, position, direction, distance );

        CHECK_EQUAL( 2, distances.size() );
        CHECK_EQUAL( 3, distances.id(0) );
        CHECK_CLOSE( 3.5+std::sqrt(5.0*5.0-3.1*3.1), distances.dist(0), 1e-5 );
        CHECK_EQUAL( 4, distances.id(1) );
        CHECK_CLOSE( 100.0, distances.dist(1), 1e-5 );
    }

    TEST_FIXTURE(SpatialGridGPUTester, radialCrossingDistances_twice_through_a_single_cylinder_going_inward_single_crossing_outward  ) {
        std::vector<gpuRayFloat_t> Rverts = { 1.0, 2.0, 3.0, 5.0 };
        std::vector<gpuRayFloat_t> Zverts = { 0.0, 1.0, 2.0, 3.0, 5.0 };

        cylindricalGrid_setup(Rverts, Zverts);

        gpuFloatType_t y = 3.0 / std::sqrt(2.0 );
        Position_t position (  -4.0, y,  0.5 );
        Position_t direction(    1,   0,    0 );
        gpuFloatType_t distance = 9.0;

        distances_t distances = crossingDistance( R, position, direction, distance );

        CHECK_EQUAL( 4, distances.size() );
        CHECK_EQUAL( 3, distances.id(0) );
        CHECK_CLOSE( 4.0 - y, distances.dist(0), 1e-5 );
        CHECK_EQUAL( 2, distances.id(1) );
        CHECK_CLOSE( 4.0 + y, distances.dist(1), 1e-5 );
        CHECK_EQUAL( 3, distances.id(2) );
        CHECK_CLOSE( 4.0 + std::sqrt(5.0*5.0-y*y) , distances.dist(2), 1e-5 );
        CHECK_EQUAL( 4, distances.id(3) );
        CHECK_CLOSE( 9.0, distances.dist(3), 1e-5 );
    }

#endif /* __HIPCC__ */
}

}

