#include "hip/hip_runtime.h"
#include <UnitTest++.h>

#include "MonteRay_CartesianGrid.t.hh"
#include "MonteRay_SpatialGrid.hh"
#include "MonteRayVector3D.hh"
#include "MonteRay_GridBins.hh"
#include "MonteRayDefinitions.hh"
#include "MonteRayCopyMemory.t.hh"

namespace MonteRay_CartesianGrid_crossingDistance_tests{

using namespace MonteRay;

SUITE( MonteRay_CartesianGrid_crossingDistance_Tests) {
    typedef Vector3D<gpuRayFloat_t> Position_t;
    using GridBins_t = MonteRay_GridBins;
    enum coord {X,Y,Z,DIM};

    class gridTestData {
    public:

        gridTestData(){
            std::vector<gpuRayFloat_t> vertices{
                -10, -9, -8, -7, -6, -5, -4, -3, -2, -1,
                  0,  1,  2,  3,  4,  5,  6,  7,  8,  9,  10};

            pGridInfo[X] = new GridBins_t();
            pGridInfo[Y] = new GridBins_t();
            pGridInfo[Z] = new GridBins_t();

            pGridInfo[X]->initialize( vertices );
            pGridInfo[Y]->initialize( vertices );
            pGridInfo[Z]->initialize( vertices );

        }
        ~gridTestData(){
            delete pGridInfo[X];
            delete pGridInfo[Y];
            delete pGridInfo[Z];
        }

        MonteRay_SpatialGrid::pArrayOfpGridInfo_t pGridInfo;
    };

    typedef singleDimRayTraceMap_t distances_t;
    typedef singleDimRayTraceMap_t rayTraceMap_t;
    TEST( CrossingDistance_in_1D_PosXDir ) {
        //CHECK(false);
        gridTestData data;
        MonteRay_CartesianGrid cart(3,data.pGridInfo);

        Position_t position ( -9.5, 0.5,  0.5 );
        Position_t direction(    1,   0,    0 );
        gpuRayFloat_t distance = 1.0;

        const unsigned dim = 0; const unsigned threadID=0;
        RayWorkInfo rayInfo(1,true);
        cart.crossingDistance(dim, threadID, rayInfo, position[dim], direction[dim], distance);
        singleDimRayTraceMap_t distances( rayInfo, 0, dim );

        CHECK_EQUAL( 2, distances.size() );
        CHECK_EQUAL( 0, distances.id(0) );
        CHECK_CLOSE( 0.5, distances.dist(0), 1e-6 );
        CHECK_EQUAL( 1, distances.id(1) );
        CHECK_CLOSE( 1.0, distances.dist(1), 1e-6 );
    }


    TEST( CrossingDistance_in_1D_NegXDir ) {
        gridTestData data;
        MonteRay_CartesianGrid cart(3,data.pGridInfo);

        Position_t position ( -8.5, 0.5,  0.5 );
        Position_t direction(    -1,   0,    0 );
        gpuRayFloat_t distance = 1.0;

        const unsigned dim = 0; const unsigned threadID=0;
        RayWorkInfo rayInfo(1,true);
        cart.crossingDistance(dim, threadID, rayInfo, position[dim], direction[dim], distance);
        singleDimRayTraceMap_t distances( rayInfo, 0, dim );

        CHECK_EQUAL( 2, distances.size() );
        CHECK_EQUAL( 1, distances.id(0) );
        CHECK_CLOSE( 0.5, distances.dist(0), 1e-6 );
        CHECK_EQUAL( 0, distances.id(1) );
        CHECK_CLOSE( 1.0, distances.dist(1), 1e-6 );
    }

    TEST( Outside_negSide_negDir ) {
        gridTestData data;
        MonteRay_CartesianGrid cart(3,data.pGridInfo);

        Position_t position ( -10.5, 0.5,  0.5 );
        Position_t direction(    -1,   0,    0 );
        gpuRayFloat_t distance = 2.0;

        const unsigned dim = 0; const unsigned threadID=0;
        RayWorkInfo rayInfo(1,true);
        cart.crossingDistance(dim, threadID, rayInfo, position[dim], direction[dim], distance);
        singleDimRayTraceMap_t distances( rayInfo, 0, dim );

        CHECK_EQUAL(  0, distances.size() );
    }

    TEST( Outside_posSide_posDir ) {
        gridTestData data;
        MonteRay_CartesianGrid cart(3,data.pGridInfo);

        Position_t position (  10.5, 0.5,  0.5 );
        Position_t direction(    1,   0,    0 );
        gpuRayFloat_t distance = 2.0;

        const unsigned dim = 0; const unsigned threadID=0;
        RayWorkInfo rayInfo(1,true);
        cart.crossingDistance(dim, threadID, rayInfo, position[dim], direction[dim], distance);
        singleDimRayTraceMap_t distances( rayInfo, 0, dim );

        CHECK_EQUAL(  0, distances.size() );
    }

    TEST( Outside_negSide_posDir ) {
        gridTestData data;
        MonteRay_CartesianGrid cart(3,data.pGridInfo);

        Position_t position ( -10.5, 0.5,  0.5 );
        Position_t direction(    1,   0,    0 );
        gpuRayFloat_t distance = 2.0;

        const unsigned dim = 0; const unsigned threadID=0;
        RayWorkInfo rayInfo(1,true);
        cart.crossingDistance(dim, threadID, rayInfo, position[dim], direction[dim], distance);
        singleDimRayTraceMap_t distances( rayInfo, 0, dim );

        CHECK_EQUAL( 3, distances.size() );
        CHECK_EQUAL( -1, distances.id(0) );
        CHECK_CLOSE( 0.5, distances.dist(0), 1e-6 );
        CHECK_EQUAL( 0, distances.id(1) );
        CHECK_CLOSE( 1.5, distances.dist(1), 1e-6 );
        CHECK_EQUAL( 1, distances.id(2) );
        CHECK_CLOSE( 2.0, distances.dist(2), 1e-6 );
    }

    TEST( Outside_posSide_negDir ) {
        gridTestData data;
        MonteRay_CartesianGrid cart(3,data.pGridInfo);

        Position_t position (  10.5, 0.5,  0.5 );
        Position_t direction(    -1,   0,    0 );
        gpuRayFloat_t distance = 2.0;

        const unsigned dim = 0; const unsigned threadID=0;
        RayWorkInfo rayInfo(1,true);
        cart.crossingDistance(dim, threadID, rayInfo, position[dim], direction[dim], distance);
        singleDimRayTraceMap_t distances( rayInfo, 0, dim );
        
        CHECK_EQUAL( 3, distances.size() );
        CHECK_EQUAL( 20, distances.id(0) );
        CHECK_CLOSE( 0.5, distances.dist(0), 1e-6 );
        CHECK_EQUAL( 19, distances.id(1) );
        CHECK_CLOSE( 1.5, distances.dist(1), 1e-6 );
        CHECK_EQUAL( 18, distances.id(2) );
        CHECK_CLOSE( 2.0, distances.dist(2), 1e-6 );
    }

    TEST( Crossing_entire_grid_starting_outside_finish_outside_pos_dir ) {
        gridTestData data;
        MonteRay_CartesianGrid cart(3,data.pGridInfo);

        Position_t position (  -10.5, 0.5,  0.5 );
        Position_t direction(    1,   0,    0 );
        gpuRayFloat_t distance = 21.0;

        const unsigned dim = 0; const unsigned threadID=0;
        RayWorkInfo rayInfo(1,true);
        cart.crossingDistance(dim, threadID, rayInfo, position[dim], direction[dim], distance);
        singleDimRayTraceMap_t distances( rayInfo, 0, dim );

        CHECK_EQUAL( 22, distances.size() );
        CHECK_EQUAL( -1, distances.id(0) );
        CHECK_CLOSE( 0.5, distances.dist(0), 1e-6 );
        CHECK_EQUAL( 0, distances.id(1) );
        CHECK_CLOSE( 1.5, distances.dist(1), 1e-6 );
        CHECK_EQUAL( 1, distances.id(2) );
        CHECK_CLOSE( 2.5, distances.dist(2), 1e-6 );
        CHECK_EQUAL( 17, distances.id(18) );
        CHECK_CLOSE( 18.5, distances.dist(18), 1e-6 );
        CHECK_EQUAL( 18, distances.id(19) );
        CHECK_CLOSE( 19.5, distances.dist(19), 1e-6 );
        CHECK_EQUAL( 19, distances.id(20) );
        CHECK_CLOSE( 20.5, distances.dist(20), 1e-6 );
        CHECK_EQUAL( 20, distances.id(21) );
        CHECK_CLOSE( 21.0, distances.dist(21), 1e-6 );

    }

    TEST( Crossing_entire_grid_starting_outside_finish_outside_neg_dir ) {
        gridTestData data;
        MonteRay_CartesianGrid cart(3,data.pGridInfo);

        Position_t position (  10.5, 0.5,  0.5 );
        Position_t direction(   -1,   0,    0 );
        gpuRayFloat_t distance = 21.0;

        const unsigned dim = 0; const unsigned threadID=0;
        RayWorkInfo rayInfo(1,true);
        cart.crossingDistance(dim, threadID, rayInfo, position[dim], direction[dim], distance);
        singleDimRayTraceMap_t distances( rayInfo, 0, dim );

        CHECK_EQUAL( 22, distances.size() );
        CHECK_EQUAL( 20, distances.id(0) );
        CHECK_CLOSE( 0.5, distances.dist(0), 1e-6 );
        CHECK_EQUAL( 19, distances.id(1) );
        CHECK_CLOSE( 1.5, distances.dist(1), 1e-6 );
        CHECK_EQUAL( 18, distances.id(2) );
        CHECK_CLOSE( 2.5, distances.dist(2), 1e-6 );
        CHECK_EQUAL( 2, distances.id(18) );
        CHECK_CLOSE( 18.5, distances.dist(18), 1e-6 );
        CHECK_EQUAL( 1, distances.id(19) );
        CHECK_CLOSE( 19.5, distances.dist(19), 1e-6 );
        CHECK_EQUAL( 0, distances.id(20) );
        CHECK_CLOSE( 20.5, distances.dist(20), 1e-6 );
        CHECK_EQUAL( -1, distances.id(21) );
        CHECK_CLOSE( 21.0, distances.dist(21), 1e-6 );
    }

    TEST( Inside_cross_out_negDir ) {
        gridTestData data;
        MonteRay_CartesianGrid cart(3,data.pGridInfo);

        Position_t position (  -8.5, 0.5,  0.5 );
        Position_t direction(    -1,   0,    0 );
        gpuRayFloat_t distance = 2.0;

        const unsigned dim = 0; const unsigned threadID=0;
        RayWorkInfo rayInfo(1,true);
        cart.crossingDistance(dim, threadID, rayInfo, position[dim], direction[dim], distance);
        singleDimRayTraceMap_t distances( rayInfo, 0, dim );

        CHECK_EQUAL( 3, distances.size() );
        CHECK_EQUAL( 1, distances.id(0) );
        CHECK_CLOSE( 0.5, distances.dist(0), 1e-6 );
        CHECK_EQUAL( 0, distances.id(1) );
        CHECK_CLOSE( 1.5, distances.dist(1), 1e-6 );
        CHECK_EQUAL( -1, distances.id(2) );
        CHECK_CLOSE( 2.0, distances.dist(2), 1e-6 );
    }

    TEST( Inside_cross_out_posDir ) {
        gridTestData data;
        MonteRay_CartesianGrid cart(3,data.pGridInfo);

        Position_t position (  8.5, 0.5,  0.5 );
        Position_t direction(    1,   0,    0 );
        gpuRayFloat_t distance = 2.0;

        const unsigned dim = 0; const unsigned threadID=0;
        RayWorkInfo rayInfo(1,true);
        cart.crossingDistance(dim, threadID, rayInfo, position[dim], direction[dim], distance);
        singleDimRayTraceMap_t distances( rayInfo, 0, dim );

        CHECK_EQUAL( 3, distances.size() );
        CHECK_EQUAL( 18, distances.id(0) );
        CHECK_CLOSE( 0.5, distances.dist(0), 1e-6 );
        CHECK_EQUAL( 19, distances.id(1) );
        CHECK_CLOSE( 1.5, distances.dist(1), 1e-6 );
        CHECK_EQUAL( 20, distances.id(2) );
        CHECK_CLOSE( 2.0, distances.dist(2), 1e-6 );
    }

    TEST( crossingDistance_2D_internal_hit_corner_posXDir_posYDir ) {
        GridBins_t* pGridInfo[3];

        pGridInfo[X] = new GridBins_t();
        pGridInfo[Y] = new GridBins_t();
        pGridInfo[Z] = new GridBins_t();

        pGridInfo[X]->initialize( -1, 1, 2);
        pGridInfo[Y]->initialize( -1, 1, 2);
        pGridInfo[Z]->initialize( -1, 1, 2);

        Position_t position (  -.5, -.5, -.5 );
        Position_t direction(  1.0,  1.0,  0.0 );
        direction.normalize();
        gpuRayFloat_t distance = 1.0*std::sqrt(2.0);

        MonteRay_CartesianGrid cart(3,pGridInfo);

        unsigned dim = 0;
        distances_t distances;
        CHECK_EQUAL( 3, cart.getDimension() );
        CHECK_EQUAL( 0, cart.getDimIndex(0, -0.5) );
        CHECK_EQUAL( 0, cart.getDimIndex(1, -0.5) );
        CHECK_EQUAL( 0, cart.getDimIndex(2, -0.5) );
        CHECK_EQUAL( 1, cart.getDimIndex(0, 0.5) );
        CHECK_EQUAL( 1, cart.getDimIndex(1, 0.5) );
        CHECK_EQUAL( 1, cart.getDimIndex(2, 0.5) );
        CHECK_EQUAL( 2, cart.getNumBins(0) );
        CHECK_EQUAL( 2, cart.getNumBins(1) );
        CHECK_EQUAL( 2, cart.getNumBins(2) );

        //cart.crossingDistance( distances, dim, position, direction, distance);

        unsigned threadID=0;
        RayWorkInfo rayInfo(1,true);
        cart.crossingDistance(dim, threadID, rayInfo, position[dim], direction[dim], distance);

        {
            singleDimRayTraceMap_t distances( rayInfo, 0, dim );

            CHECK_EQUAL( 2, distances.size() );
            CHECK_EQUAL( 0, distances.id(0) );
            CHECK_CLOSE( (0.5)*std::sqrt(2.0), distances.dist(0), 1e-6 );
            CHECK_EQUAL( 1, distances.id(1) );
            CHECK_CLOSE( (1.0)*std::sqrt(2.0), distances.dist(1), 1e-6 );
        }

        dim = 1;
        cart.crossingDistance(dim, threadID, rayInfo, position[dim], direction[dim], distance);

        {
            singleDimRayTraceMap_t distances( rayInfo, 0, dim );

            CHECK_EQUAL( 2, distances.size() );
            CHECK_EQUAL( 0, distances.id(0) );
            CHECK_CLOSE( (0.5)*std::sqrt(2.0), distances.dist(0), 1e-6 );
            CHECK_EQUAL( 1, distances.id(1) );
            CHECK_CLOSE( (1.0)*std::sqrt(2.0), distances.dist(1), 1e-6 );
        }

        delete pGridInfo[X];
        delete pGridInfo[Y];
        delete pGridInfo[Z];
    }


    TEST( crossingDistance_2D_start_on_an_external_corner_posX_posY ) {
        GridBins_t* pGridInfo[3];

        pGridInfo[X] = new GridBins_t();
        pGridInfo[Y] = new GridBins_t();
        pGridInfo[Z] = new GridBins_t();

        pGridInfo[X]->initialize( 0, 3, 3);
        pGridInfo[Y]->initialize( 0, 3, 3);
        pGridInfo[Z]->initialize( 0, 3, 3);

        Position_t position (  0.0, 0.0, 0.5 );
        Position_t direction(  1.0,  1.0,  0.0 );
        direction.normalize();
        gpuRayFloat_t distance = 10.0;

        MonteRay_CartesianGrid cart(3,pGridInfo);

        unsigned dim = 0;
        unsigned threadID=0;
        RayWorkInfo rayInfo(1,true);
        cart.crossingDistance(dim, threadID, rayInfo, position[dim], direction[dim], distance);

        {
            singleDimRayTraceMap_t distances( rayInfo, 0, dim );
            CHECK_EQUAL( 5, distances.size() );
            CHECK_EQUAL( -1, distances.id(0) );
            CHECK_CLOSE( (0.0)*std::sqrt(2.0), distances.dist(0), 1e-6 );
            CHECK_EQUAL( 0, distances.id(1) );
            CHECK_CLOSE( (1.0)*std::sqrt(2.0), distances.dist(1), 1e-6 );
            CHECK_EQUAL( 1, distances.id(2) );
            CHECK_CLOSE( (2.0)*std::sqrt(2.0), distances.dist(2), 1e-6 );
            CHECK_EQUAL( 2, distances.id(3) );
            CHECK_CLOSE( (3.0)*std::sqrt(2.0), distances.dist(3), 1e-6 );
            CHECK_EQUAL( 3, distances.id(4) );
            CHECK_CLOSE( 10.0, distances.dist(4), 1e-6 );
        }

        dim = 1;
        cart.crossingDistance(dim, threadID, rayInfo, position[dim], direction[dim], distance);

        {
            singleDimRayTraceMap_t distances( rayInfo, 0, dim );

            CHECK_EQUAL( 5, distances.size() );
            CHECK_EQUAL( -1, distances.id(0) );
            CHECK_CLOSE( (0.0)*std::sqrt(2.0), distances.dist(0), 1e-6 );
            CHECK_EQUAL( 0, distances.id(1) );
            CHECK_CLOSE( (1.0)*std::sqrt(2.0), distances.dist(1), 1e-6 );
            CHECK_EQUAL( 1, distances.id(2) );
            CHECK_CLOSE( (2.0)*std::sqrt(2.0), distances.dist(2), 1e-6 );
            CHECK_EQUAL( 2, distances.id(3) );
            CHECK_CLOSE( (3.0)*std::sqrt(2.0), distances.dist(3), 1e-6 );
            CHECK_EQUAL( 3, distances.id(4) );
            CHECK_CLOSE( 10.0, distances.dist(4), 1e-6 );
        }

        delete pGridInfo[X];
        delete pGridInfo[Y];
        delete pGridInfo[Z];
    }

    TEST( crossingDistance_2D_start_on_an_external_corner_negX_negY ) {
        GridBins_t* pGridInfo[3];

        pGridInfo[X] = new GridBins_t();
        pGridInfo[Y] = new GridBins_t();
        pGridInfo[Z] = new GridBins_t();

        pGridInfo[X]->initialize( 0, 3, 3);
        pGridInfo[Y]->initialize( 0, 3, 3);
        pGridInfo[Z]->initialize( 0, 3, 3);

        Position_t position (  3.0,  3.0, 0.5 );
        Position_t direction( -1.0, -1.0, 0.0 );
        direction.normalize();
        gpuRayFloat_t distance = 10.0;

        MonteRay_CartesianGrid cart(3,pGridInfo);

        unsigned dim = 0;

        unsigned threadID=0;
        RayWorkInfo rayInfo(1,true);
        cart.crossingDistance(dim, threadID, rayInfo, position[dim], direction[dim], distance);

        {
            singleDimRayTraceMap_t distances( rayInfo, 0, dim );

            CHECK_EQUAL( 5, distances.size() );
            CHECK_EQUAL( 3, distances.id(0) );
            CHECK_CLOSE( (0.0)*std::sqrt(2.0), distances.dist(0), 1e-6 );
            CHECK_EQUAL( 2, distances.id(1) );
            CHECK_CLOSE( (1.0)*std::sqrt(2.0), distances.dist(1), 1e-6 );
            CHECK_EQUAL( 1, distances.id(2) );
            CHECK_CLOSE( (2.0)*std::sqrt(2.0), distances.dist(2), 1e-6 );
            CHECK_EQUAL( 0, distances.id(3) );
            CHECK_CLOSE( (3.0)*std::sqrt(2.0), distances.dist(3), 1e-6 );
            CHECK_EQUAL( -1, distances.id(4) );
            CHECK_CLOSE( 10.0, distances.dist(4), 1e-6 );
        }

        dim = 1;
        cart.crossingDistance(dim, threadID, rayInfo, position[dim], direction[dim], distance);
        {
            singleDimRayTraceMap_t distances( rayInfo, 0, dim );

            CHECK_EQUAL( 5, distances.size() );
            CHECK_EQUAL( 3, distances.id(0) );
            CHECK_CLOSE( (0.0)*std::sqrt(2.0), distances.dist(0), 1e-6 );
            CHECK_EQUAL( 2, distances.id(1) );
            CHECK_CLOSE( (1.0)*std::sqrt(2.0), distances.dist(1), 1e-6 );
            CHECK_EQUAL( 1, distances.id(2) );
            CHECK_CLOSE( (2.0)*std::sqrt(2.0), distances.dist(2), 1e-6 );
            CHECK_EQUAL( 0, distances.id(3) );
            CHECK_CLOSE( (3.0)*std::sqrt(2.0), distances.dist(3), 1e-6 );
            CHECK_EQUAL( -1, distances.id(4) );
            CHECK_CLOSE( 10.0, distances.dist(4), 1e-6 );
        }

        delete pGridInfo[X];
        delete pGridInfo[Y];
        delete pGridInfo[Z];
    }

    TEST( crossingDistance_2D_start_outside_on_an_external_corner_posX_posY ) {
        GridBins_t* pGridInfo[3];

        pGridInfo[X] = new GridBins_t();
        pGridInfo[Y] = new GridBins_t();
        pGridInfo[Z] = new GridBins_t();

        pGridInfo[X]->initialize( 0, 3, 3);
        pGridInfo[Y]->initialize( 0, 3, 3);
        pGridInfo[Z]->initialize( 0, 3, 3);

        Position_t position ( -1.0, -1.0, 0.5 );
        Position_t direction(  1.0,  1.0, 0.0 );
        direction.normalize();
        gpuRayFloat_t distance = 10.0;

        MonteRay_CartesianGrid cart(3,pGridInfo);

        unsigned dim = 0;
        unsigned threadID=0;
        RayWorkInfo rayInfo(1,true);
        cart.crossingDistance(dim, threadID, rayInfo, position[dim], direction[dim], distance);

        {
            singleDimRayTraceMap_t distances( rayInfo, 0, dim );

            CHECK_EQUAL( 5, distances.size() );
            CHECK_EQUAL( -1, distances.id(0) );
            CHECK_CLOSE( (1.0)*std::sqrt(2.0), distances.dist(0), 1e-6 );
            CHECK_EQUAL( 0, distances.id(1) );
            CHECK_CLOSE( (2.0)*std::sqrt(2.0), distances.dist(1), 1e-6 );
            CHECK_EQUAL( 1, distances.id(2) );
            CHECK_CLOSE( (3.0)*std::sqrt(2.0), distances.dist(2), 1e-6 );
            CHECK_EQUAL( 2, distances.id(3) );
            CHECK_CLOSE( (4.0)*std::sqrt(2.0), distances.dist(3), 1e-6 );
            CHECK_EQUAL( 3, distances.id(4) );
            CHECK_CLOSE( 10.0, distances.dist(4), 1e-6 );
        }

        dim = 1;
        cart.crossingDistance(dim, threadID, rayInfo, position[dim], direction[dim], distance);

        {
            singleDimRayTraceMap_t distances( rayInfo, 0, dim );
            CHECK_EQUAL( 5, distances.size() );
            CHECK_EQUAL( -1, distances.id(0) );
            CHECK_CLOSE( (1.0)*std::sqrt(2.0), distances.dist(0), 1e-6 );
            CHECK_EQUAL( 0, distances.id(1) );
            CHECK_CLOSE( (2.0)*std::sqrt(2.0), distances.dist(1), 1e-6 );
            CHECK_EQUAL( 1, distances.id(2) );
            CHECK_CLOSE( (3.0)*std::sqrt(2.0), distances.dist(2), 1e-6 );
            CHECK_EQUAL( 2, distances.id(3) );
            CHECK_CLOSE( (4.0)*std::sqrt(2.0), distances.dist(3), 1e-6 );
            CHECK_EQUAL( 3, distances.id(4) );
            CHECK_CLOSE( 10.0, distances.dist(4), 1e-6 );
        }
        delete pGridInfo[X];
        delete pGridInfo[Y];
        delete pGridInfo[Z];
    }

    TEST( crossingDistance_2D_start_outside_an_external_corner_negX_negY ) {

        GridBins_t* pGridInfo[3];

        pGridInfo[X] = new GridBins_t();
        pGridInfo[Y] = new GridBins_t();
        pGridInfo[Z] = new GridBins_t();

        pGridInfo[X]->initialize( 0, 3, 3);
        pGridInfo[Y]->initialize( 0, 3, 3);
        pGridInfo[Z]->initialize( 0, 3, 3);

        Position_t position (  4.0,  4.0, 0.5 );
        Position_t direction( -1.0, -1.0, 0.0 );
        direction.normalize();
        gpuRayFloat_t distance = 10.0;

        MonteRay_CartesianGrid cart(3,pGridInfo);

        unsigned dim = 0;
        unsigned threadID=0;
        RayWorkInfo rayInfo(1,true);
        cart.crossingDistance(dim, threadID, rayInfo, position[dim], direction[dim], distance);

        {
            singleDimRayTraceMap_t distances( rayInfo, 0, dim );
            CHECK_EQUAL( 5, distances.size() );
            CHECK_EQUAL( 3, distances.id(0) );
            CHECK_CLOSE( (1.0)*std::sqrt(2.0), distances.dist(0), 1e-6 );
            CHECK_EQUAL( 2, distances.id(1) );
            CHECK_CLOSE( (2.0)*std::sqrt(2.0), distances.dist(1), 1e-6 );
            CHECK_EQUAL( 1, distances.id(2) );
            CHECK_CLOSE( (3.0)*std::sqrt(2.0), distances.dist(2), 1e-6 );
            CHECK_EQUAL( 0, distances.id(3) );
            CHECK_CLOSE( (4.0)*std::sqrt(2.0), distances.dist(3), 1e-6 );
            CHECK_EQUAL( -1, distances.id(4) );
            CHECK_CLOSE( 10.0, distances.dist(4), 1e-6 );
        }

        dim = 1;
        cart.crossingDistance(dim, threadID, rayInfo, position[dim], direction[dim], distance);

        {
            singleDimRayTraceMap_t distances( rayInfo, 0, dim );
            CHECK_EQUAL( 5, distances.size() );
            CHECK_EQUAL( 3, distances.id(0) );
            CHECK_CLOSE( (1.0)*std::sqrt(2.0), distances.dist(0), 1e-6 );
            CHECK_EQUAL( 2, distances.id(1) );
            CHECK_CLOSE( (2.0)*std::sqrt(2.0), distances.dist(1), 1e-6 );
            CHECK_EQUAL( 1, distances.id(2) );
            CHECK_CLOSE( (3.0)*std::sqrt(2.0), distances.dist(2), 1e-6 );
            CHECK_EQUAL( 0, distances.id(3) );
            CHECK_CLOSE( (4.0)*std::sqrt(2.0), distances.dist(3), 1e-6 );
            CHECK_EQUAL( -1, distances.id(4) );
            CHECK_CLOSE( 10.0, distances.dist(4), 1e-6 );
        }

        delete pGridInfo[X];
        delete pGridInfo[Y];
        delete pGridInfo[Z];
    }

}

}

