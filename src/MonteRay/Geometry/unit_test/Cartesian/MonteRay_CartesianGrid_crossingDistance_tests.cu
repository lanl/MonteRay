#include "hip/hip_runtime.h"
#include <UnitTest++.h>

#include "MonteRay_CartesianGrid.t.hh"
#include "MonteRay_SpatialGrid.hh"
#include "MonteRayVector3D.hh"
#include "MonteRay_GridBins.hh"
#include "MonteRayDefinitions.hh"
#include "ThirdParty/Array.hh"

namespace MonteRay_CartesianGrid_crossingDistance_tests{

using namespace MonteRay;

SUITE( MonteRay_CartesianGrid_crossingDistance_Tests) {
    typedef Vector3D<gpuRayFloat_t> Position_t;
    using GridBins_t = MonteRay_GridBins;
    enum coord {X,Y,Z,DIM};

    class CartesianGridTester {
    public:

        CartesianGridTester(){
            std::vector<gpuRayFloat_t> vertices{
                -10, -9, -8, -7, -6, -5, -4, -3, -2, -1,
                  0,  1,  2,  3,  4,  5,  6,  7,  8,  9,  10};

            pCart = std::make_unique<MonteRay_CartesianGrid>(3, 
              std::array<MonteRay_GridBins, 3>{
                MonteRay_GridBins{vertices},
                MonteRay_GridBins{vertices},
                MonteRay_GridBins{vertices}
              }
            );

        }

        std::unique_ptr<MonteRay_CartesianGrid> pCart;
    };

    typedef singleDimRayTraceMap_t distances_t;
    typedef singleDimRayTraceMap_t rayTraceMap_t;
    TEST_FIXTURE(CartesianGridTester, CrossingDistance_in_1D_PosXDir ) {
        auto& cart = *pCart;

        Position_t position ( -9.5, 0.5,  0.5 );
        Position_t direction(    1,   0,    0 );
        gpuRayFloat_t distance = 1.0;

        const unsigned dim = 0; const unsigned threadID=0;
        RayWorkInfo rayInfo(1,true);
        cart.crossingDistance(dim, threadID, rayInfo, position[dim], direction[dim], distance);
        singleDimRayTraceMap_t distances( rayInfo, 0, dim );

        CHECK_EQUAL( 2, distances.size() );
        CHECK_EQUAL( 0, distances.id(0) );
        CHECK_CLOSE( 0.5, distances.dist(0), 1e-6 );
        CHECK_EQUAL( 1, distances.id(1) );
        CHECK_CLOSE( 1.0, distances.dist(1), 1e-6 );
    }


    TEST_FIXTURE(CartesianGridTester,  CrossingDistance_in_1D_NegXDir ) {
        auto& cart = *pCart;

        Position_t position ( -8.5, 0.5,  0.5 );
        Position_t direction(    -1,   0,    0 );
        gpuRayFloat_t distance = 1.0;

        const unsigned dim = 0; const unsigned threadID=0;
        RayWorkInfo rayInfo(1,true);
        cart.crossingDistance(dim, threadID, rayInfo, position[dim], direction[dim], distance);
        singleDimRayTraceMap_t distances( rayInfo, 0, dim );

        CHECK_EQUAL( 2, distances.size() );
        CHECK_EQUAL( 1, distances.id(0) );
        CHECK_CLOSE( 0.5, distances.dist(0), 1e-6 );
        CHECK_EQUAL( 0, distances.id(1) );
        CHECK_CLOSE( 1.0, distances.dist(1), 1e-6 );
    }

    TEST_FIXTURE(CartesianGridTester,  Outside_negSide_negDir ) {
        auto& cart = *pCart;

        Position_t position ( -10.5, 0.5,  0.5 );
        Position_t direction(    -1,   0,    0 );
        gpuRayFloat_t distance = 2.0;

        const unsigned dim = 0; const unsigned threadID=0;
        RayWorkInfo rayInfo(1,true);
        cart.crossingDistance(dim, threadID, rayInfo, position[dim], direction[dim], distance);
        singleDimRayTraceMap_t distances( rayInfo, 0, dim );

        CHECK_EQUAL(  0, distances.size() );
    }

    TEST_FIXTURE(CartesianGridTester,  Outside_posSide_posDir ) {
        auto& cart = *pCart;

        Position_t position (  10.5, 0.5,  0.5 );
        Position_t direction(    1,   0,    0 );
        gpuRayFloat_t distance = 2.0;

        const unsigned dim = 0; const unsigned threadID=0;
        RayWorkInfo rayInfo(1,true);
        cart.crossingDistance(dim, threadID, rayInfo, position[dim], direction[dim], distance);
        singleDimRayTraceMap_t distances( rayInfo, 0, dim );

        CHECK_EQUAL(  0, distances.size() );
    }

    TEST_FIXTURE(CartesianGridTester,  Outside_negSide_posDir ) {
        auto& cart = *pCart;

        Position_t position ( -10.5, 0.5,  0.5 );
        Position_t direction(    1,   0,    0 );
        gpuRayFloat_t distance = 2.0;

        const unsigned dim = 0; const unsigned threadID=0;
        RayWorkInfo rayInfo(1,true);
        cart.crossingDistance(dim, threadID, rayInfo, position[dim], direction[dim], distance);
        singleDimRayTraceMap_t distances( rayInfo, 0, dim );

        CHECK_EQUAL( 3, distances.size() );
        CHECK_EQUAL( -1, distances.id(0) );
        CHECK_CLOSE( 0.5, distances.dist(0), 1e-6 );
        CHECK_EQUAL( 0, distances.id(1) );
        CHECK_CLOSE( 1.5, distances.dist(1), 1e-6 );
        CHECK_EQUAL( 1, distances.id(2) );
        CHECK_CLOSE( 2.0, distances.dist(2), 1e-6 );
    }

    TEST_FIXTURE(CartesianGridTester,  Outside_posSide_negDir ) {
        auto& cart = *pCart;

        Position_t position (  10.5, 0.5,  0.5 );
        Position_t direction(    -1,   0,    0 );
        gpuRayFloat_t distance = 2.0;

        const unsigned dim = 0; const unsigned threadID=0;
        RayWorkInfo rayInfo(1,true);
        cart.crossingDistance(dim, threadID, rayInfo, position[dim], direction[dim], distance);
        singleDimRayTraceMap_t distances( rayInfo, 0, dim );
        
        CHECK_EQUAL( 3, distances.size() );
        CHECK_EQUAL( 20, distances.id(0) );
        CHECK_CLOSE( 0.5, distances.dist(0), 1e-6 );
        CHECK_EQUAL( 19, distances.id(1) );
        CHECK_CLOSE( 1.5, distances.dist(1), 1e-6 );
        CHECK_EQUAL( 18, distances.id(2) );
        CHECK_CLOSE( 2.0, distances.dist(2), 1e-6 );
    }

    TEST_FIXTURE(CartesianGridTester,  Crossing_entire_grid_starting_outside_finish_outside_pos_dir ) {
        auto& cart = *pCart;

        Position_t position (  -10.5, 0.5,  0.5 );
        Position_t direction(    1,   0,    0 );
        gpuRayFloat_t distance = 21.0;

        const unsigned dim = 0; const unsigned threadID=0;
        RayWorkInfo rayInfo(1,true);
        cart.crossingDistance(dim, threadID, rayInfo, position[dim], direction[dim], distance);
        singleDimRayTraceMap_t distances( rayInfo, 0, dim );

        CHECK_EQUAL( 22, distances.size() );
        CHECK_EQUAL( -1, distances.id(0) );
        CHECK_CLOSE( 0.5, distances.dist(0), 1e-6 );
        CHECK_EQUAL( 0, distances.id(1) );
        CHECK_CLOSE( 1.5, distances.dist(1), 1e-6 );
        CHECK_EQUAL( 1, distances.id(2) );
        CHECK_CLOSE( 2.5, distances.dist(2), 1e-6 );
        CHECK_EQUAL( 17, distances.id(18) );
        CHECK_CLOSE( 18.5, distances.dist(18), 1e-6 );
        CHECK_EQUAL( 18, distances.id(19) );
        CHECK_CLOSE( 19.5, distances.dist(19), 1e-6 );
        CHECK_EQUAL( 19, distances.id(20) );
        CHECK_CLOSE( 20.5, distances.dist(20), 1e-6 );
        CHECK_EQUAL( 20, distances.id(21) );
        CHECK_CLOSE( 21.0, distances.dist(21), 1e-6 );

    }

    TEST_FIXTURE(CartesianGridTester,  Crossing_entire_grid_starting_outside_finish_outside_neg_dir ) {
        auto& cart = *pCart;

        Position_t position (  10.5, 0.5,  0.5 );
        Position_t direction(   -1,   0,    0 );
        gpuRayFloat_t distance = 21.0;

        const unsigned dim = 0; const unsigned threadID=0;
        RayWorkInfo rayInfo(1,true);
        cart.crossingDistance(dim, threadID, rayInfo, position[dim], direction[dim], distance);
        singleDimRayTraceMap_t distances( rayInfo, 0, dim );

        CHECK_EQUAL( 22, distances.size() );
        CHECK_EQUAL( 20, distances.id(0) );
        CHECK_CLOSE( 0.5, distances.dist(0), 1e-6 );
        CHECK_EQUAL( 19, distances.id(1) );
        CHECK_CLOSE( 1.5, distances.dist(1), 1e-6 );
        CHECK_EQUAL( 18, distances.id(2) );
        CHECK_CLOSE( 2.5, distances.dist(2), 1e-6 );
        CHECK_EQUAL( 2, distances.id(18) );
        CHECK_CLOSE( 18.5, distances.dist(18), 1e-6 );
        CHECK_EQUAL( 1, distances.id(19) );
        CHECK_CLOSE( 19.5, distances.dist(19), 1e-6 );
        CHECK_EQUAL( 0, distances.id(20) );
        CHECK_CLOSE( 20.5, distances.dist(20), 1e-6 );
        CHECK_EQUAL( -1, distances.id(21) );
        CHECK_CLOSE( 21.0, distances.dist(21), 1e-6 );
    }

    TEST_FIXTURE(CartesianGridTester,  Inside_cross_out_negDir ) {
        auto& cart = *pCart;

        Position_t position (  -8.5, 0.5,  0.5 );
        Position_t direction(    -1,   0,    0 );
        gpuRayFloat_t distance = 2.0;

        const unsigned dim = 0; const unsigned threadID=0;
        RayWorkInfo rayInfo(1,true);
        cart.crossingDistance(dim, threadID, rayInfo, position[dim], direction[dim], distance);
        singleDimRayTraceMap_t distances( rayInfo, 0, dim );

        CHECK_EQUAL( 3, distances.size() );
        CHECK_EQUAL( 1, distances.id(0) );
        CHECK_CLOSE( 0.5, distances.dist(0), 1e-6 );
        CHECK_EQUAL( 0, distances.id(1) );
        CHECK_CLOSE( 1.5, distances.dist(1), 1e-6 );
        CHECK_EQUAL( -1, distances.id(2) );
        CHECK_CLOSE( 2.0, distances.dist(2), 1e-6 );
    }

    TEST_FIXTURE(CartesianGridTester,  Inside_cross_out_posDir ) {
        auto& cart = *pCart;

        Position_t position (  8.5, 0.5,  0.5 );
        Position_t direction(    1,   0,    0 );
        gpuRayFloat_t distance = 2.0;

        const unsigned dim = 0; const unsigned threadID=0;
        RayWorkInfo rayInfo(1,true);
        cart.crossingDistance(dim, threadID, rayInfo, position[dim], direction[dim], distance);
        singleDimRayTraceMap_t distances( rayInfo, 0, dim );

        CHECK_EQUAL( 3, distances.size() );
        CHECK_EQUAL( 18, distances.id(0) );
        CHECK_CLOSE( 0.5, distances.dist(0), 1e-6 );
        CHECK_EQUAL( 19, distances.id(1) );
        CHECK_CLOSE( 1.5, distances.dist(1), 1e-6 );
        CHECK_EQUAL( 20, distances.id(2) );
        CHECK_CLOSE( 2.0, distances.dist(2), 1e-6 );
    }

    TEST( crossingDistance_2D_internal_hit_corner_posXDir_posYDir ) {
        Position_t position (  -.5, -.5, -.5 );
        Position_t direction(  1.0,  1.0,  0.0 );
        direction.normalize();
        gpuRayFloat_t distance = 1.0*std::sqrt(2.0);

        MonteRay_CartesianGrid cart(3, 
            MonteRay_GridBins{-1, 1, 2},
            MonteRay_GridBins{-1, 1, 2},
            MonteRay_GridBins{-1, 1, 2});

        unsigned dim = 0;
        distances_t distances;
        CHECK_EQUAL( 3, cart.getDimension() );
        CHECK_EQUAL( 0, cart.getDimIndex(0, -0.5) );
        CHECK_EQUAL( 0, cart.getDimIndex(1, -0.5) );
        CHECK_EQUAL( 0, cart.getDimIndex(2, -0.5) );
        CHECK_EQUAL( 1, cart.getDimIndex(0, 0.5) );
        CHECK_EQUAL( 1, cart.getDimIndex(1, 0.5) );
        CHECK_EQUAL( 1, cart.getDimIndex(2, 0.5) );
        CHECK_EQUAL( 2, cart.getNumBins(0) );
        CHECK_EQUAL( 2, cart.getNumBins(1) );
        CHECK_EQUAL( 2, cart.getNumBins(2) );

        //cart.crossingDistance( distances, dim, position, direction, distance);

        unsigned threadID=0;
        RayWorkInfo rayInfo(1,true);
        cart.crossingDistance(dim, threadID, rayInfo, position[dim], direction[dim], distance);

        {
            singleDimRayTraceMap_t distances( rayInfo, 0, dim );

            CHECK_EQUAL( 2, distances.size() );
            CHECK_EQUAL( 0, distances.id(0) );
            CHECK_CLOSE( (0.5)*std::sqrt(2.0), distances.dist(0), 1e-6 );
            CHECK_EQUAL( 1, distances.id(1) );
            CHECK_CLOSE( (1.0)*std::sqrt(2.0), distances.dist(1), 1e-6 );
        }

        dim = 1;
        cart.crossingDistance(dim, threadID, rayInfo, position[dim], direction[dim], distance);

        {
            singleDimRayTraceMap_t distances( rayInfo, 0, dim );

            CHECK_EQUAL( 2, distances.size() );
            CHECK_EQUAL( 0, distances.id(0) );
            CHECK_CLOSE( (0.5)*std::sqrt(2.0), distances.dist(0), 1e-6 );
            CHECK_EQUAL( 1, distances.id(1) );
            CHECK_CLOSE( (1.0)*std::sqrt(2.0), distances.dist(1), 1e-6 );
        }
    }


    TEST( crossingDistance_2D_start_on_an_external_corner_posX_posY ) {
        Position_t position (  0.0, 0.0, 0.5 );
        Position_t direction(  1.0,  1.0,  0.0 );
        direction.normalize();
        gpuRayFloat_t distance = 10.0;

        MonteRay_CartesianGrid cart(3, 
            MonteRay_GridBins{0, 3, 3},
            MonteRay_GridBins{0, 3, 3},
            MonteRay_GridBins{0, 3, 3});

        unsigned dim = 0;
        unsigned threadID=0;
        RayWorkInfo rayInfo(1,true);
        cart.crossingDistance(dim, threadID, rayInfo, position[dim], direction[dim], distance);

        {
            singleDimRayTraceMap_t distances( rayInfo, 0, dim );
            CHECK_EQUAL( 5, distances.size() );
            CHECK_EQUAL( -1, distances.id(0) );
            CHECK_CLOSE( (0.0)*std::sqrt(2.0), distances.dist(0), 1e-6 );
            CHECK_EQUAL( 0, distances.id(1) );
            CHECK_CLOSE( (1.0)*std::sqrt(2.0), distances.dist(1), 1e-6 );
            CHECK_EQUAL( 1, distances.id(2) );
            CHECK_CLOSE( (2.0)*std::sqrt(2.0), distances.dist(2), 1e-6 );
            CHECK_EQUAL( 2, distances.id(3) );
            CHECK_CLOSE( (3.0)*std::sqrt(2.0), distances.dist(3), 1e-6 );
            CHECK_EQUAL( 3, distances.id(4) );
            CHECK_CLOSE( 10.0, distances.dist(4), 1e-6 );
        }

        dim = 1;
        cart.crossingDistance(dim, threadID, rayInfo, position[dim], direction[dim], distance);

        {
            singleDimRayTraceMap_t distances( rayInfo, 0, dim );

            CHECK_EQUAL( 5, distances.size() );
            CHECK_EQUAL( -1, distances.id(0) );
            CHECK_CLOSE( (0.0)*std::sqrt(2.0), distances.dist(0), 1e-6 );
            CHECK_EQUAL( 0, distances.id(1) );
            CHECK_CLOSE( (1.0)*std::sqrt(2.0), distances.dist(1), 1e-6 );
            CHECK_EQUAL( 1, distances.id(2) );
            CHECK_CLOSE( (2.0)*std::sqrt(2.0), distances.dist(2), 1e-6 );
            CHECK_EQUAL( 2, distances.id(3) );
            CHECK_CLOSE( (3.0)*std::sqrt(2.0), distances.dist(3), 1e-6 );
            CHECK_EQUAL( 3, distances.id(4) );
            CHECK_CLOSE( 10.0, distances.dist(4), 1e-6 );
        }

    }

    TEST( crossingDistance_2D_start_on_an_external_corner_negX_negY ) {
        Position_t position (  3.0,  3.0, 0.5 );
        Position_t direction( -1.0, -1.0, 0.0 );
        direction.normalize();
        gpuRayFloat_t distance = 10.0;

        MonteRay_CartesianGrid cart(3, 
            MonteRay_GridBins{0, 3, 3},
            MonteRay_GridBins{0, 3, 3},
            MonteRay_GridBins{0, 3, 3});

        unsigned dim = 0;

        unsigned threadID=0;
        RayWorkInfo rayInfo(1,true);
        cart.crossingDistance(dim, threadID, rayInfo, position[dim], direction[dim], distance);

        {
            singleDimRayTraceMap_t distances( rayInfo, 0, dim );

            CHECK_EQUAL( 5, distances.size() );
            CHECK_EQUAL( 3, distances.id(0) );
            CHECK_CLOSE( (0.0)*std::sqrt(2.0), distances.dist(0), 1e-6 );
            CHECK_EQUAL( 2, distances.id(1) );
            CHECK_CLOSE( (1.0)*std::sqrt(2.0), distances.dist(1), 1e-6 );
            CHECK_EQUAL( 1, distances.id(2) );
            CHECK_CLOSE( (2.0)*std::sqrt(2.0), distances.dist(2), 1e-6 );
            CHECK_EQUAL( 0, distances.id(3) );
            CHECK_CLOSE( (3.0)*std::sqrt(2.0), distances.dist(3), 1e-6 );
            CHECK_EQUAL( -1, distances.id(4) );
            CHECK_CLOSE( 10.0, distances.dist(4), 1e-6 );
        }

        dim = 1;
        cart.crossingDistance(dim, threadID, rayInfo, position[dim], direction[dim], distance);
        {
            singleDimRayTraceMap_t distances( rayInfo, 0, dim );

            CHECK_EQUAL( 5, distances.size() );
            CHECK_EQUAL( 3, distances.id(0) );
            CHECK_CLOSE( (0.0)*std::sqrt(2.0), distances.dist(0), 1e-6 );
            CHECK_EQUAL( 2, distances.id(1) );
            CHECK_CLOSE( (1.0)*std::sqrt(2.0), distances.dist(1), 1e-6 );
            CHECK_EQUAL( 1, distances.id(2) );
            CHECK_CLOSE( (2.0)*std::sqrt(2.0), distances.dist(2), 1e-6 );
            CHECK_EQUAL( 0, distances.id(3) );
            CHECK_CLOSE( (3.0)*std::sqrt(2.0), distances.dist(3), 1e-6 );
            CHECK_EQUAL( -1, distances.id(4) );
            CHECK_CLOSE( 10.0, distances.dist(4), 1e-6 );
        }

    }

    TEST( crossingDistance_2D_start_outside_on_an_external_corner_posX_posY ) {
        Position_t position ( -1.0, -1.0, 0.5 );
        Position_t direction(  1.0,  1.0, 0.0 );
        direction.normalize();
        gpuRayFloat_t distance = 10.0;

        MonteRay_CartesianGrid cart(3, 
            MonteRay_GridBins{0, 3, 3},
            MonteRay_GridBins{0, 3, 3},
            MonteRay_GridBins{0, 3, 3});


        unsigned dim = 0;
        unsigned threadID=0;
        RayWorkInfo rayInfo(1,true);
        cart.crossingDistance(dim, threadID, rayInfo, position[dim], direction[dim], distance);

        {
            singleDimRayTraceMap_t distances( rayInfo, 0, dim );

            CHECK_EQUAL( 5, distances.size() );
            CHECK_EQUAL( -1, distances.id(0) );
            CHECK_CLOSE( (1.0)*std::sqrt(2.0), distances.dist(0), 1e-6 );
            CHECK_EQUAL( 0, distances.id(1) );
            CHECK_CLOSE( (2.0)*std::sqrt(2.0), distances.dist(1), 1e-6 );
            CHECK_EQUAL( 1, distances.id(2) );
            CHECK_CLOSE( (3.0)*std::sqrt(2.0), distances.dist(2), 1e-6 );
            CHECK_EQUAL( 2, distances.id(3) );
            CHECK_CLOSE( (4.0)*std::sqrt(2.0), distances.dist(3), 1e-6 );
            CHECK_EQUAL( 3, distances.id(4) );
            CHECK_CLOSE( 10.0, distances.dist(4), 1e-6 );
        }

        dim = 1;
        cart.crossingDistance(dim, threadID, rayInfo, position[dim], direction[dim], distance);

        {
            singleDimRayTraceMap_t distances( rayInfo, 0, dim );
            CHECK_EQUAL( 5, distances.size() );
            CHECK_EQUAL( -1, distances.id(0) );
            CHECK_CLOSE( (1.0)*std::sqrt(2.0), distances.dist(0), 1e-6 );
            CHECK_EQUAL( 0, distances.id(1) );
            CHECK_CLOSE( (2.0)*std::sqrt(2.0), distances.dist(1), 1e-6 );
            CHECK_EQUAL( 1, distances.id(2) );
            CHECK_CLOSE( (3.0)*std::sqrt(2.0), distances.dist(2), 1e-6 );
            CHECK_EQUAL( 2, distances.id(3) );
            CHECK_CLOSE( (4.0)*std::sqrt(2.0), distances.dist(3), 1e-6 );
            CHECK_EQUAL( 3, distances.id(4) );
            CHECK_CLOSE( 10.0, distances.dist(4), 1e-6 );
        }
    }

    TEST( crossingDistance_2D_start_outside_an_external_corner_negX_negY ) {
        Position_t position (  4.0,  4.0, 0.5 );
        Position_t direction( -1.0, -1.0, 0.0 );
        direction.normalize();
        gpuRayFloat_t distance = 10.0;


        MonteRay_CartesianGrid cart(3, 
            MonteRay_GridBins{0, 3, 3},
            MonteRay_GridBins{0, 3, 3},
            MonteRay_GridBins{0, 3, 3});

        unsigned dim = 0;
        unsigned threadID=0;
        RayWorkInfo rayInfo(1,true);
        cart.crossingDistance(dim, threadID, rayInfo, position[dim], direction[dim], distance);

        {
            singleDimRayTraceMap_t distances( rayInfo, 0, dim );
            CHECK_EQUAL( 5, distances.size() );
            CHECK_EQUAL( 3, distances.id(0) );
            CHECK_CLOSE( (1.0)*std::sqrt(2.0), distances.dist(0), 1e-6 );
            CHECK_EQUAL( 2, distances.id(1) );
            CHECK_CLOSE( (2.0)*std::sqrt(2.0), distances.dist(1), 1e-6 );
            CHECK_EQUAL( 1, distances.id(2) );
            CHECK_CLOSE( (3.0)*std::sqrt(2.0), distances.dist(2), 1e-6 );
            CHECK_EQUAL( 0, distances.id(3) );
            CHECK_CLOSE( (4.0)*std::sqrt(2.0), distances.dist(3), 1e-6 );
            CHECK_EQUAL( -1, distances.id(4) );
            CHECK_CLOSE( 10.0, distances.dist(4), 1e-6 );
        }

        dim = 1;
        cart.crossingDistance(dim, threadID, rayInfo, position[dim], direction[dim], distance);

        {
            singleDimRayTraceMap_t distances( rayInfo, 0, dim );
            CHECK_EQUAL( 5, distances.size() );
            CHECK_EQUAL( 3, distances.id(0) );
            CHECK_CLOSE( (1.0)*std::sqrt(2.0), distances.dist(0), 1e-6 );
            CHECK_EQUAL( 2, distances.id(1) );
            CHECK_CLOSE( (2.0)*std::sqrt(2.0), distances.dist(1), 1e-6 );
            CHECK_EQUAL( 1, distances.id(2) );
            CHECK_CLOSE( (3.0)*std::sqrt(2.0), distances.dist(2), 1e-6 );
            CHECK_EQUAL( 0, distances.id(3) );
            CHECK_CLOSE( (4.0)*std::sqrt(2.0), distances.dist(3), 1e-6 );
            CHECK_EQUAL( -1, distances.id(4) );
            CHECK_CLOSE( 10.0, distances.dist(4), 1e-6 );
        }

    }

}

}

