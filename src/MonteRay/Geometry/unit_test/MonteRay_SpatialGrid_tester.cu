#include "hip/hip_runtime.h"
#include <UnitTest++.h>

#include "MonteRay_SpatialGrid.hh"
#include <type_traits>
#include "UnitTestHelper.hh"

using namespace MonteRay;

namespace MonteRaySpatialGridTester{


SUITE( MonteRay_SpatialGrid_Tester ) {
  using Grid_t = MonteRay_SpatialGrid;

  class SpatialGridTester{
    public:
      std::array<MonteRay_GridBins, 3> cartGridBins{
        MonteRay_GridBins{-10.0, 10.0, 1},
        MonteRay_GridBins{-10.0, 10.0, 2},
        MonteRay_GridBins{-10.0, 10.0, 3}
      };

      std::array<MonteRay_GridBins, 3> cylGridBins{
        MonteRay_GridBins{0.0, 10.0, 10, MonteRay_GridBins::RADIAL},
        MonteRay_GridBins{-10.0, 10.0, 20},
        MonteRay_GridBins{-10.0, 10.0, 30}
      };
  };

  TEST_FIXTURE(SpatialGridTester, ConstructorsAndGetters){
    auto grid = MonteRay_SpatialGrid(TransportMeshType::Cartesian, cartGridBins);
    CHECK(TransportMeshType::Cartesian == grid.getCoordinateSystem());
    CHECK_EQUAL(3, grid.getDimension());
    CHECK_EQUAL(1, grid.getNumGridBins(0));
    CHECK_EQUAL(2, grid.getNumGridBins(1));
    CHECK_EQUAL(3, grid.getNumGridBins(2));

    grid = MonteRay_SpatialGrid(TransportMeshType::Cylindrical, cylGridBins);
    CHECK(TransportMeshType::Cylindrical == grid.getCoordinateSystem());
    CHECK_EQUAL(2, grid.getDimension());
    CHECK_EQUAL(10, grid.getNumGridBins(0));
    CHECK_EQUAL(20, grid.getNumGridBins(1));

    CHECK_EQUAL(1, grid.getMinVertex(0));
    CHECK_EQUAL(-10, grid.getMinVertex(1));
    CHECK_EQUAL(10, grid.getMaxVertex(0));
    CHECK_EQUAL(10, grid.getMaxVertex(1));

    CHECK_EQUAL(-4.0, grid.getVertex(1, 6));

    MonteRay_SpatialGrid::Position_t pos{9.5, 0, 9.5};
    CHECK_EQUAL(199, grid.getIndex(pos));

    CHECK_CLOSE(M_PI*(10*10 - 9*9), grid.getVolume(199), 1E-5);
    CHECK_EQUAL(10, grid.getNumVertices(0));
    CHECK_EQUAL(10, grid.getNumVerticesSq(0));
    CHECK_EQUAL(0, grid.getNumVerticesSq(1));
  }

#ifdef __HIPCC__
  __global__ void testSpatialGrid(bool* testVal, MonteRay_SpatialGrid* pGrid) {
    *testVal = true;

    GPU_CHECK(2 == pGrid->getDimension());
    GPU_CHECK(10 == pGrid->getNumGridBins(0));
    GPU_CHECK(20 == pGrid->getNumGridBins(1));

    GPU_CHECK(1 == pGrid->getMinVertex(0));
    GPU_CHECK(-10 == pGrid->getMinVertex(1));
    GPU_CHECK(10 == pGrid->getMaxVertex(0));
    GPU_CHECK(10 == pGrid->getMaxVertex(1));

    GPU_CHECK(-4.0 == pGrid->getVertex(1, 6));

    MonteRay_SpatialGrid::Position_t pos{9.5, 0, 9.5};
    GPU_CHECK(199 == pGrid->getIndex(pos));

    GPU_CHECK(10 == pGrid->getNumVertices(0));
    GPU_CHECK(10 == pGrid->getNumVerticesSq(0));
    GPU_CHECK(0 == pGrid->getNumVerticesSq(1));
  }

  TEST_FIXTURE(SpatialGridTester, GettersOnGPU){
    auto upGrid = std::make_unique<MonteRay_SpatialGrid>(TransportMeshType::Cylindrical, cylGridBins);
    auto pGrid = upGrid.get();
    bool* pTestVal;
    hipMallocManaged(&pTestVal, sizeof(bool));
    *pTestVal = false;
    testSpatialGrid<<<1, 1>>>(pTestVal, pGrid);
    hipDeviceSynchronize();
    CHECK(*pTestVal);
    hipFree(pTestVal);
  }

#endif

}

}// end namespace 
