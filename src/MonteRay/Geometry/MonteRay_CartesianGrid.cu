#include "hip/hip_runtime.h"
/*
 * MonteRayCartesianGrid.cc
 *
 *  Created on: Feb 2, 2018
 *      Author: jsweezy
 */

#include "MonteRay_CartesianGrid.hh"
#include "MonteRayDefinitions.hh"
#include "MonteRay_SingleValueCopyMemory.t.hh"
#include "MonteRayCopyMemory.t.hh"
#include "RayWorkInfo.hh"
#include "MonteRayParallelAssistant.hh"

#include <float.h>

namespace MonteRay {

using ptrCartesianGrid_result_t = MonteRay_SingleValueCopyMemory<MonteRay_CartesianGrid*>;

CUDA_CALLABLE_KERNEL  createDeviceInstance(MonteRay_CartesianGrid** pPtrInstance, ptrCartesianGrid_result_t* pResult, MonteRay_GridBins* pGridX, MonteRay_GridBins* pGridY, MonteRay_GridBins* pGridZ ) {
    *pPtrInstance = new MonteRay_CartesianGrid( 3, pGridX, pGridY, pGridZ );
    pResult->v = *pPtrInstance;
#ifndef NDEBUG
    if( debug ) printf( "Debug: createDeviceInstance -- pPtrInstance = %d\n", pPtrInstance );
#endif
}

CUDA_CALLABLE_KERNEL  deleteDeviceInstance(MonteRay_CartesianGrid** pPtrInstance) {
    delete *pPtrInstance;
}

CUDAHOST_CALLABLE_MEMBER
MonteRay_CartesianGrid*
MonteRay_CartesianGrid::getDeviceInstancePtr() {
    return devicePtr;
}


CUDA_CALLABLE_MEMBER
MonteRay_CartesianGrid::MonteRay_CartesianGrid(unsigned dim, pArrayOfpGridInfo_t pBins) :
MonteRay_GridSystemInterface(dim)
{
    MONTERAY_VERIFY( dim == DimMax, "MonteRay_CartesianGrid::ctor -- only 3-D is allowed" ); // No greater than 3-D.

    DIM = 3;
    for(auto i = 0; i< dim; ++i) {
        pGridBins[i] = pBins[i];
    }
}

CUDA_CALLABLE_MEMBER
MonteRay_CartesianGrid::MonteRay_CartesianGrid(unsigned dim, GridBins_t* pGridX, GridBins_t* pGridY, GridBins_t* pGridZ ) :
MonteRay_GridSystemInterface(dim)
{
    MONTERAY_VERIFY( dim == DimMax, "MonteRay_CartesianGrid::ctor -- only 3-D is allowed" ); // No greater than 3-D.

    DIM = 3;
    pGridBins[0] = pGridX;
    pGridBins[1] = pGridY;
    pGridBins[2] = pGridZ;
}

CUDA_CALLABLE_MEMBER
MonteRay_CartesianGrid::~MonteRay_CartesianGrid(void){
#ifdef __HIPCC__
#ifndef __CUDA_ARCH__
    if( ptrDevicePtr ) {
        deleteDeviceInstance<<<1,1>>>( ptrDevicePtr );
        hipDeviceSynchronize();
    }
    MonteRayDeviceFree( ptrDevicePtr );
#endif
#endif
}

CUDAHOST_CALLABLE_MEMBER
void
MonteRay_CartesianGrid::copyToGPU(void) {
#ifndef NDEBUG
    if( debug ) std::cout << "Debug: MonteRay_CartesianGrid::copyToGPU \n";
#endif
#ifdef __HIPCC__
    if( ! MonteRay::isWorkGroupMaster() ) return;

    ptrDevicePtr = (MonteRay_CartesianGrid**) MONTERAYDEVICEALLOC(sizeof(MonteRay_CartesianGrid*), std::string("device - MonteRay_CartesianGrid::ptrDevicePtr") );

    pGridBins[0]->copyToGPU();
    pGridBins[1]->copyToGPU();
    pGridBins[2]->copyToGPU();

    std::unique_ptr<ptrCartesianGrid_result_t> ptrResult = std::unique_ptr<ptrCartesianGrid_result_t>( new ptrCartesianGrid_result_t() );
    ptrResult->copyToGPU();

    createDeviceInstance<<<1,1>>>( ptrDevicePtr, ptrResult->devicePtr, pGridBins[0]->devicePtr, pGridBins[1]->devicePtr, pGridBins[2]->devicePtr );
    hipDeviceSynchronize();
    ptrResult->copyToCPU();
    devicePtr = ptrResult->v;

#endif
}

CUDA_CALLABLE_MEMBER
unsigned
MonteRay_CartesianGrid::getIndex( const GridBins_t::Position_t& particle_pos) const{
#ifndef NDEBUG
    if( debug ) printf("Debug: MonteRay_CartesianGrid::getIndex -- starting\n");
#endif

    int indices[3]= {0,0,0};
    for( unsigned d = 0; d < DIM; ++d ) {
#ifndef NDEBUG
        if( debug ) printf("Debug: MonteRay_CartesianGrid::getIndex -- d = %d\n",d);
#endif
        indices[d] = getDimIndex(d, particle_pos[d] );

        // outside the grid
        if( isIndexOutside(d, indices[d] ) ) {
            return OUTSIDE_INDEX;
        }
    }


#ifndef NDEBUG
    if( debug ) printf("Debug: MonteRay_CartesianGrid::getIndex -- calling calcIndex\n");
#endif
    return calcIndex( indices );
}

CUDA_CALLABLE_MEMBER
gpuRayFloat_t
MonteRay_CartesianGrid::getVolume(unsigned index ) const {

    gpuRayFloat_t volume=1.0;
    //    if( regular ) {
    //        for( unsigned d=0; d < DIM; ++d ) {
    //            volume *= pGridBins[d]->delta;
    //        }
    //    } else {
    uint3 indices = calcIJK( index );
    volume *= pGridBins[0]->vertices[ indices.x + 1 ] - pGridBins[0]->vertices[ indices.x ];
    volume *= pGridBins[1]->vertices[ indices.y + 1 ] - pGridBins[1]->vertices[ indices.y ];
    volume *= pGridBins[2]->vertices[ indices.z + 1 ] - pGridBins[2]->vertices[ indices.z ];
    //    }
    return volume;
}

CUDA_CALLABLE_MEMBER
unsigned
MonteRay_CartesianGrid::getNumBins( unsigned d) const {
#ifndef NDEBUG
    if( debug ) printf("Debug: MonteRay_CartesianGrid::getNumBins -- d= %d\n", d);
    if( debug ) printf("Debug: MonteRay_CartesianGrid::getNumBins --calling pGridBins[d]->getNumBins()\n");
#endif
    return pGridBins[d]->getNumBins();
}

CUDA_CALLABLE_MEMBER
unsigned
MonteRay_CartesianGrid::calcIndex( const int indices[] ) const{
    unsigned index = indices[0];
    if( DIM > 1 ) {
        index += indices[1]* getNumBins(0);
    }
    if( DIM > 2 ) {
        index += indices[2] * getNumBins(0)*getNumBins(1);
    }
    return index;
}

CUDA_CALLABLE_MEMBER
uint3
MonteRay_CartesianGrid::calcIJK( unsigned index ) const {
    uint3 indices;

    uint3 offsets;
    offsets.x = 1;

    offsets.y = getNumBins(0);
    offsets.z = getNumBins(0)*getNumBins(1);

    MONTERAY_ASSERT(offsets.z > 0 );
    MONTERAY_ASSERT(offsets.y > 0 );
    MONTERAY_ASSERT(offsets.x > 0 );

    indices.z = index / offsets.z;
    index -= indices.z * offsets.z;

    indices.y = index / offsets.y;
    index -= indices.y * offsets.y;

    indices.x = index / offsets.x;

    return indices;
}

CUDA_CALLABLE_MEMBER
bool
MonteRay_CartesianGrid::isOutside( const int i[] ) const {
    for( unsigned d=0; d<DIM; ++d){
        if( isIndexOutside(d, i[d]) ) return true;
    }
    return false;
}

CUDA_CALLABLE_MEMBER
void
MonteRay_CartesianGrid::rayTrace(
        const unsigned threadID,
        RayWorkInfo& rayInfo,
        const GridBins_t::Position_t& particle_pos,
        const GridBins_t::Position_t& particle_dir,
        const gpuRayFloat_t distance,
        const bool outsideDistances ) const{

#ifndef NDEBUG
    if( debug ) printf( "Debug: MonteRay_CartesianGrid::rayTrace -- \n");
#endif

    int indices[3] = {0, 0, 0}; // current position indices in the grid, must be int because can be outside

    for( unsigned d=0; d<DIM; ++d){

        indices[d] = getDimIndex(d, particle_pos[d] );

#ifndef NDEBUG
        if( debug ) printf( "Debug: MonteRay_CartesianGrid::rayTrace -- dimension=%d, index=%d\n", d, indices[d]);
#endif

        planarCrossingDistance( d, threadID, rayInfo, *(pGridBins[d]), particle_pos[d], particle_dir[d], distance,indices[d]);

#ifndef NDEBUG
        if( debug ) printf( "Debug: MonteRay_CartesianGrid::rayTrace -- dimension=%d, number of planar crossings = %d\n", d, rayInfo.getCrossingSize(d,threadID) );
#endif

        // if outside and ray doesn't move inside then ray never enters the grid
        if( isIndexOutside(d,indices[d]) && rayInfo.getCrossingSize(d,threadID) == 0  ) {
            return;
        }
    }

    orderCrossings<3>( threadID, rayInfo, indices, distance, outsideDistances );

#ifndef NDEBUG
    if( debug ) printf( "Debug: MonteRay_CartesianGrid::rayTrace -- number of total crossings = %d\n", rayInfo.getRayCastSize(threadID) );
#endif
    return;
}


CUDA_CALLABLE_MEMBER
DirectionAndSpeed MonteRay_CartesianGrid::convertToCellReferenceFrame(
    const Vector3D<gpuRayFloat_t>& cellVelocity,
    const GridBins_t::Position_t&, // here to maintain same API as other grid types
    GridBins_t::Direction_t dir,
    gpuRayFloat_t speed) const
{
  dir = dir*speed - cellVelocity;
  speed = dir.magnitude();
  dir /= speed;
  return {dir, speed};
}

CUDA_CALLABLE_MEMBER DistAndDir
MonteRay_CartesianGrid::getMinDistToSurface(
       const GridBins_t::Position_t& pos,
       const GridBins_t::Direction_t& dir,
       const int indices[] 
       ) const {
 int d = 0;
 gpuRayFloat_t minDistToSurf = (pGridBins[d]->vertices[indices[d] + Math::signbit(-dir[d])] - pos[d])/dir[d];
 unsigned minDistIndex = 0;
 for (d=1; d<DIM; ++d){
   auto distToSurface = (pGridBins[d]->vertices[indices[d] + Math::signbit(-dir[d])] - pos[d])/dir[d];
   if (distToSurface < minDistToSurf){
     minDistIndex = d;
     minDistToSurf = distToSurface;
   }
  }
 if (minDistToSurf < 0) {
   minDistToSurf = 0;
 }
 return {minDistToSurf, minDistIndex, std::signbit(-dir[minDistIndex])};
}

CUDA_CALLABLE_MEMBER
gpuRayFloat_t MonteRay_CartesianGrid::getDistanceToInsideOfMesh(const GridBins_t::Position_t& pos, const GridBins_t::Direction_t& dir) const {
  gpuRayFloat_t dist = 0.0;
  for (int d = 0; d < static_cast<int>(DIM); d++){
    dist = Math::max(dist, pGridBins[d]->distanceToGetInsideLinearMesh(pos, dir, d));
  }
  return dist;
}

CUDA_CALLABLE_MEMBER
void
MonteRay_CartesianGrid::crossingDistance(
        const unsigned dim,
        const unsigned threadID,
        RayWorkInfo& rayInfo,
        const GridBins_t::Position_t& pos,
        const GridBins_t::Direction_t& dir,
        const gpuRayFloat_t distance ) const {

    crossingDistance( dim, threadID, rayInfo, pos[dim], dir[dim], distance);
}

CUDA_CALLABLE_MEMBER
void
MonteRay_CartesianGrid::crossingDistance(
        const unsigned dim,
        const unsigned threadID,
        RayWorkInfo& rayInfo,
        const gpuRayFloat_t pos,
        const gpuRayFloat_t dir,
        const gpuRayFloat_t distance ) const {

#ifndef NDEBUG
    if( debug ) printf( "Debug: MonteRay_CartesianGrid::crossingDistance( dim, threadID, rayInfo, float_t pos, float_t dir, float_t distance ) const \n");
#endif
    crossingDistance(dim, threadID, rayInfo, *(pGridBins[dim]), pos, dir, distance, false);
    return;
}

CUDA_CALLABLE_MEMBER
void
MonteRay_CartesianGrid::crossingDistance(
        const unsigned dim,
        const unsigned threadID,
        RayWorkInfo& rayInfo,
        const GridBins_t& Bins,
        const gpuRayFloat_t pos,
        const gpuRayFloat_t dir,
        const gpuRayFloat_t distance,
        const bool equal_spacing) const {

#ifndef NDEBUG
    if( debug ) printf( "Debug: MonteRay_CartesianGrid::crossingDistance( dim, threadID, rayInfo, GridBins_t& Bins, float_t pos, float_t dir, float_t distance, bool equal_spacing) const \n");
#endif
    int index = Bins.getLinearIndex(pos);
#ifndef NDEBUG
    if( debug ) printf( "Debug: MonteRay_CartesianGrid::crossingDistance -- calling MonteRay_GridSystemInterface::planarCrossingDistance.\n");
#endif
    planarCrossingDistance( dim, threadID, rayInfo, Bins, pos, dir, distance, index);
    return;
}


} /* namespace MonteRay */
