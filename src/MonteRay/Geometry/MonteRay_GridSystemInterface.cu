#include "hip/hip_runtime.h"
/*
 * MonteRayGridSystemInterface.cc
 *
 *  Created on: Feb 2, 2018
 *      Author: jsweezy
 */

#include "MonteRay_GridSystemInterface.t.hh"
#include <float.h>

#include "RayWorkInfo.hh"

namespace MonteRay {

CUDA_CALLABLE_MEMBER
void
singleDimRayTraceMap_t::add( const int cell, const gpuRayFloat_t dist) {
    MONTERAY_ASSERT( N < MAXNUMVERTICES-1);
    CellId[N] = cell;
    distance[N] = dist;
    ++N;
}

// for conversion of old tests
CUDA_CALLABLE_MEMBER
singleDimRayTraceMap_t::singleDimRayTraceMap_t(RayWorkInfo& rayInfo, const unsigned threadID, int dim){

    if( dim < 0 ) {
        unsigned raySize = rayInfo.getRayCastSize(threadID);
        for( unsigned i=0; i<raySize; ++i ) {
            add( rayInfo.getRayCastCell(threadID,i),  rayInfo.getRayCastDist(threadID,i));
        }
        return;
    }

    for( unsigned i=0; i<rayInfo.getCrossingSize(dim,threadID); ++i ) {
        add( rayInfo.getCrossingCell(dim,threadID,i),  rayInfo.getCrossingDist(dim,threadID,i));
    }
}

CUDA_CALLABLE_MEMBER
void
rayTraceList_t::add( const unsigned cell, const gpuRayFloat_t dist) {
    MONTERAY_ASSERT( N < 2*MAXNUMVERTICES-1);
    CellId[N] = cell;
    distance[N] = dist;
    ++N;
}

// for conversion of old tests
CUDA_CALLABLE_MEMBER
rayTraceList_t::rayTraceList_t(RayWorkInfo& rayInfo, const unsigned threadID, int dim){

    if( dim < 0 ) {
        for( unsigned i=0; i<rayInfo.getRayCastSize(threadID); ++i ) {
            add( rayInfo.getRayCastCell(threadID,i),  rayInfo.getRayCastDist(threadID,i));
        }
        return;
    }

    for( unsigned i=0; i<rayInfo.getCrossingSize(dim,threadID); ++i ) {
        add( rayInfo.getCrossingCell(dim,threadID,i),  rayInfo.getCrossingDist(dim,threadID,i));
    }
}

template<unsigned NUMDIM>
CUDA_CALLABLE_MEMBER
void
MonteRay_GridSystemInterface::orderCrossings(
        const unsigned threadID,
        RayWorkInfo& rayInfo,
        int indices[],
        const gpuRayFloat_t distance,
        const bool outsideDistances ) const {

    // Order the distance crossings to provide a rayTrace

#ifndef NDEBUG
    const bool debug = false;

    if( debug ) {
        printf( "Debug: *************************************************************\n");
        printf( "Debug: Starting GridSystemInterface::orderCrossings\n");
    }

    if( debug )  {
        for( unsigned d = 0; d<NUMDIM; ++d) {
            printf( "Debug: GridSystemInterface::orderCrossings -- dim=%d\n",d);
            for( unsigned i = 0; i< rayInfo.getCrossingSize(d,threadID); ++i) {
                printf( "Debug: ----------------------------------- -- distances[%d].id[%d]=%d, distances[%d].dist[%d]=%f\n", d,i, rayInfo.getCrossingCell(d,threadID,i), d,i, rayInfo.getCrossingDist(d,threadID,i));
            }
        }
    }
#endif

    unsigned start[NUMDIM]; // current location in the distance[i] vector
    unsigned   end[NUMDIM]; //    last location in the distance[i] vector

    unsigned maxNumCrossings = 0;
    for( unsigned i=0; i<NUMDIM; ++i){
        start[i] = 0;
        end[i] = rayInfo.getCrossingSize(i,threadID);
        maxNumCrossings += end[i];
    }

#ifndef NDEBUG
    if( debug ) printf( "Debug: GridSystemInterface::orderCrossings -- maxNumCrossings=%d\n",maxNumCrossings);
#endif

    gpuRayFloat_t minDistances[NUMDIM];

    bool outside;
    gpuRayFloat_t priorDistance = 0.0;

    for( unsigned i=0; i<maxNumCrossings; ++i){

        for( unsigned d = 0; d<NUMDIM; ++d) {
            if( start[d] < end[d] ) {
                minDistances[d] = rayInfo.getCrossingDist(d, threadID, start[d] );
            } else {
                minDistances[d] = inf;
            }
        }

#ifndef NDEBUG
        if( debug )  {
            for( unsigned d = 0; d<NUMDIM; ++d) {
                printf( "Debug: GridSystemInterface::orderCrossings -- dim=%u, minDistance[%u]=%f\n",d, d, minDistances[d]);
            }
        }
#endif

        unsigned minDim = 0;
        gpuRayFloat_t minDist = minDistances[0];
        for( unsigned i = 1; i<NUMDIM; ++i){
            if( minDistances[i] < minDist ) {
                minDim = i;
                minDist = minDistances[i];
            }
        }

#ifndef NDEBUG
        if( debug ) printf( "Debug: GridSystemInterface::orderCrossings -- minDim=%d\n",minDim);
        if( debug ) printf( "Debug: GridSystemInterface::orderCrossings -- minDist=%f\n",minDist);
#endif

        indices[minDim] = rayInfo.getCrossingCell(minDim, threadID, start[minDim] );

        // test for outside of the grid
        outside = isOutside( indices );

#ifndef NDEBUG
        if( debug ) {
            if( outside )  printf( "Debug: ray is outside \n" );
            if( !outside ) printf( "Debug: ray is inside \n" );
        }
#endif

        gpuRayFloat_t currentDistance = rayInfo.getCrossingDist(minDim, threadID, start[minDim] );

        if( !outside || outsideDistances ) {
            gpuRayFloat_t deltaDistance = currentDistance - priorDistance;

            MONTERAY_ASSERT_MSG( ( deltaDistance >= 0.0 ),
                    "ERROR:  MONTERAY -- MonteRay_GridSystemInterface::orderCrossings, delta distance is negative");

            unsigned global_index;
            if( !outside ) {
                global_index = calcIndex( indices );
            } else {
                global_index = MonteRay_GridSystemInterface::OUTSIDE_GRID;
            }
            rayInfo.addRayCastCell( threadID, global_index, deltaDistance );

#ifndef NDEBUG
            if( debug ) {
                printf( "Debug: ****************** \n" );
                printf( "Debug:  Entry Num    = %d\n", rayInfo.getRayCastSize(threadID) );
                printf( "Debug:     index[0]  = %d\n", indices[0] );
                printf( "Debug:     index[1]  = %d\n", indices[1] );
                printf( "Debug:     index[2]  = %d\n", indices[2] );
                printf( "Debug:     distance  = %f\n", deltaDistance );
            }
#endif

        }

        if( currentDistance >= distance ) {
            break;
        }

#ifndef NDEBUG
        if( debug ) {
            if( start[minDim]+1 >= rayInfo.getCrossingSize(minDim,threadID) ) {
                printf( "Debug: Error - start[minDim]+1 >= distances[minDim].size() \n");
                printf( "Debug:                   minDim = %d\n", minDim );
                printf( "Debug:          start[minDim]+1 = %d\n", start[minDim]+1 );
                printf( "Debug: distances[minDim].size() = %d\n", rayInfo.getCrossingSize(minDim,threadID) );
            }
        }
#endif

        MONTERAY_ASSERT( minDim < NUMDIM );
        MONTERAY_ASSERT( start[minDim]+1 < rayInfo.getCrossingSize(minDim,threadID) );

        indices[minDim] = rayInfo.getCrossingCell(minDim, threadID, start[minDim]+1 );

        if( ! outside ) {
            if( isIndexOutside(minDim, indices[minDim] ) ) {
                // ray has moved outside of grid
                break;
            }
        }

        ++start[minDim];
        priorDistance = currentDistance;
    }

    return;
}

template
CUDA_CALLABLE_MEMBER
void
MonteRay_GridSystemInterface::orderCrossings<1U>(
        const unsigned threadID,
        RayWorkInfo& rayInfo,
        int indices[],
        const gpuRayFloat_t distance,
        const bool outsideDistances ) const;

template
CUDA_CALLABLE_MEMBER
void
MonteRay_GridSystemInterface::orderCrossings<2U>(
        const unsigned threadID,
        RayWorkInfo& rayInfo,
        int indices[],
        const gpuRayFloat_t distance,
        const bool outsideDistances ) const;

template
CUDA_CALLABLE_MEMBER
void
MonteRay_GridSystemInterface::orderCrossings<3U>(
        const unsigned threadID,
        RayWorkInfo& rayInfo,
        int indices[],
        const gpuRayFloat_t distance,
        const bool outsideDistances ) const;

CUDA_CALLABLE_MEMBER
void MonteRay_GridSystemInterface::planarCrossingDistance(
        const unsigned dim,
        const unsigned threadID,
        RayWorkInfo& rayInfo,
        const GridBins_t& Bins,
        const gpuRayFloat_t pos,
        const gpuRayFloat_t dir,
        const gpuRayFloat_t distance,
        const int start_index) const {

#ifndef NDEBUG
    const bool debug = false;

    if( debug ) printf( "Debug: MonteRay_GridSystemInterface::planarCrossingDistance --- \n" );
#endif

    //  constexpr gpuRayFloat_t epsilon = std::numeric_limits<gpuRayFloat_t>::epsilon();
#ifdef __HIPCC__
    if( abs(dir) <= FLT_EPSILON ) { return; }
#else
    if( std::abs(dir) <= FLT_EPSILON ) { return; }
#endif

#ifndef NDEBUG
    if( debug ) printf( "Debug: MonteRay_GridSystemInterface::planarCrossingDistance  -- Bins=%p \n", &Bins );
#endif

    if( start_index < 0 ) {
        if( dir < 0.0 ) {
            return;
        }
    }

    int nBins = Bins.getNumBins();

#ifndef NDEBUG
    if( debug ) printf( "Debug: MonteRay_GridSystemInterface::planarCrossingDistance - nBins=%d\n", nBins );
#endif

    if( start_index >= nBins ) {
        if( dir > 0.0 ) {
            return;
        }
    }

#ifdef __CUDA_ARCH__
    unsigned offset = int(signbit(-dir));
#else
    unsigned offset = int(std::signbit(-dir));
#endif

#ifndef NDEBUG
    if( debug ) printf( "Debug: MonteRay_GridSystemInterface::planarCrossingDistance - offset=%d\n", offset );
#endif

    int end_index = offset*(nBins-1);;

#ifdef __CUDA_ARCH__
    int dirIncrement = copysignf( 1, dir );
#else
    int dirIncrement = std::copysign( 1, dir );
#endif

#ifdef __HIPCC__
    unsigned num_indices = abs(end_index - start_index ) + 1;
#else
    unsigned num_indices = std::abs(end_index - start_index ) + 1;
#endif

    int current_index = start_index;

    // Calculate boundary crossing distances
    gpuRayFloat_t invDir = 1/dir;
    bool rayTerminated = false;
    for( int i = 0; i < num_indices ; ++i ) {

        //MONTERAY_ASSERT( (current_index + offset) >= 0 );
        MONTERAY_ASSERT( (current_index + offset) < Bins.getNumBins()+1 );

        gpuRayFloat_t minDistance = ( Bins.vertices[current_index + offset] - pos) * invDir;

        //if( rayDistance == inf ) {
        //    // ray doesn't cross plane
        //    break;
        //}

        if( minDistance >= distance ) {
            rayInfo.addCrossingCell(dim, threadID, current_index, distance );
            rayTerminated = true;
            break;
        }

        if( minDistance < 0.0 ) {
            minDistance = 0.0;
        }
        rayInfo.addCrossingCell(dim, threadID, current_index, minDistance );

        current_index += dirIncrement;
    }

    if( !rayTerminated ) {
        // finish with distance into area outside
        rayInfo.addCrossingCell(dim, threadID, current_index, distance );
        rayTerminated = true;
    }

    return;
}

} /* namespace MonteRay */
