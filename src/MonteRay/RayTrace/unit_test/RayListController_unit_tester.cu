#include "hip/hip_runtime.h"
#include <UnitTest++.h>

#include <iostream>
#include <functional>
#include <cmath>

#include "GPUUtilityFunctions.hh"
#include "BasicTally.hh"
#include "ExpectedPathLength.hh"
#include "MonteRay_timer.hh"
#include "RayListInterface.hh"
#include "RayListController.hh"
#include "GPUErrorCheck.hh"
#include "MaterialProperties.hh"

#include "UnitControllerBase.hh"
#include "MonteRay_SpatialGrid.hh"

namespace RayListController_unit_tester{

using namespace MonteRay;

SUITE( RayListController_unit_tester_basic_tests ) {

    class UnitControllerSetup: public UnitControllerBase {
    public:
        UnitControllerSetup(){

            pGrid = std::make_unique<MonteRay_SpatialGrid>(TransportMeshType::Cartesian, 
              std::array<MonteRay_GridBins, 3>{
              MonteRay_GridBins{-5, 5, 10},
              MonteRay_GridBins{-5, 5, 10},
              MonteRay_GridBins{-5, 5, 10} }
            );

            pTally = std::make_unique<BasicTally>(pGrid->getNumCells());

            // Density of 1.0 for mat number 0
            MaterialProperties::Builder matPropBuilder{};
            matPropBuilder.disableMemoryReduction();
            matPropBuilder.initializeMaterialDescription( std::vector<int>( pGrid->getNumCells(), 0), std::vector<float>( pGrid->getNumCells(), 1.0), pGrid->getNumCells());
            pMatProps = std::make_unique<MaterialProperties>(matPropBuilder.build());

        }

        void setup(){
#ifdef __HIPCC__
            gpuErrchk( hipPeekAtLastError() );
#endif

#ifdef __HIPCC__
            gpuErrchk( hipPeekAtLastError() );
#endif
        }

        std::unique_ptr<MaterialProperties> pMatProps;
        std::unique_ptr<MonteRay_SpatialGrid> pGrid;
        std::unique_ptr<BasicTally> pTally;
    };

    TEST_FIXTURE(UnitControllerSetup, ctor_set_capacity ){
        std::cout << "Debug: CollisionPointController_unit_tester -- ctor\n";
        CollisionPointController<MonteRay_SpatialGrid> controller( 1,
                1024,
                pGrid.get(),
                pMatList.get(),
                pMatProps.get(),
                pTally.get() );

        CHECK_EQUAL(100000, controller.capacity());
        CHECK_EQUAL(0, controller.size());
        controller.setCapacity(10);
        CHECK_EQUAL(10, controller.capacity());
    }

    TEST_FIXTURE(UnitControllerSetup, add_a_particle ){
        std::cout << "Debug: CollisionPointController_unit_tester -- add_a_particle\n";
        CollisionPointController<MonteRay_SpatialGrid> controller( 1,
                32,
                pGrid.get(),
                pMatList.get(),
                pMatProps.get(),
                pTally.get() );

        unsigned i = pGrid->getIndex( Position_t( 0.0, 0.0, 0.0 ) );
        ParticleRay_t particle;

        particle.pos[0] = 0.0;
        particle.pos[1] = 0.0;
        particle.pos[2] = 0.0;

        particle.dir[0] = 1.0;
        particle.dir[1] = 0.0;
        particle.dir[2] = 0.0;

        particle.energy[0] = 1.0;
        particle.weight[0] = 1.0;
        particle.index = i;
        particle.detectorIndex = 1;
        particle.particleType = 0;

        controller.add( particle );
        CHECK_EQUAL(1, controller.size());
    }

    TEST_FIXTURE(UnitControllerSetup, add_a_particle_via_ptr ){
        std::cout << "Debug: CollisionPointController_unit_tester -- add_a_particle_via_ptr1\n";
        CollisionPointController<MonteRay_SpatialGrid> controller( 1,
                32,
                pGrid.get(),
                pMatList.get(),
                pMatProps.get(),
                pTally.get() );

        unsigned i = pGrid->getIndex( Position_t(0.0, 0.0, 0.0) );

        ParticleRay_t particle;
        particle.pos[0] = 0.0;
        particle.pos[1] = 0.0;
        particle.pos[2] = 0.0;
        particle.dir[0] = 1.0;
        particle.dir[1] = 0.0;
        particle.dir[2] = 0.0;
        particle.energy[0] = 1.0;
        particle.weight[0] = 1.0;
        particle.index = i;
        particle.detectorIndex = 99;
        particle.particleType = 0;

        controller.add( &particle );
        CHECK_EQUAL(1, controller.size());
    }

    TEST_FIXTURE(UnitControllerSetup, add_two_particles_via_ptr ){
        std::cout << "Debug: CollisionPointController_unit_tester -- add_a_particle_via_ptr2\n";
        CollisionPointController<MonteRay_SpatialGrid> controller( 1,
                1024,
                pGrid.get(),
                pMatList.get(),
                pMatProps.get(),
                pTally.get() );

        unsigned i = pGrid->getIndex( Position_t(0.0, 0.0, 0.0) );

        ParticleRay_t particle[2];
        particle[0].pos[0] = 1.0;
        particle[0].pos[1] = 2.0;
        particle[0].pos[2] = 3.0;
        particle[0].dir[0] = 4.0;
        particle[0].dir[1] = 5.0;
        particle[0].dir[2] = 6.0;
        particle[0].energy[0] = 7.0;
        particle[0].weight[0] = 8.0;
        particle[0].index = 9;
        particle[0].detectorIndex = 99;
        particle[0].particleType = 0;

        particle[1].pos[0] = 11.0;
        particle[1].pos[1] = 12.0;
        particle[1].pos[2] = 13.0;
        particle[1].dir[0] = 14.0;
        particle[1].dir[1] = 15.0;
        particle[1].dir[2] = 16.0;
        particle[1].energy[0] = 17.0;
        particle[1].weight[0] = 18.0;
        particle[1].index = 19;
        particle[1].detectorIndex = 99;
        particle[1].particleType = 0;

        controller.add( particle, 2 );
        CHECK_EQUAL(2, controller.size());
    }

    TEST_FIXTURE(UnitControllerSetup, add_ten_particles_via_ptr ){
        std::cout << "Debug: CollisionPointController_unit_tester -- add_a_particle_via_ptr3\n";
        setup();
        CollisionPointController<MonteRay_SpatialGrid> controller( 1,
                32,
                pGrid.get(),
                pMatList.get(),
                pMatProps.get(),
                pTally.get() );

        ParticleRay_t particle[10];
        for( auto i = 0; i < 10; ++i ){
            particle[i].pos[0] = 1.0;
            particle[i].pos[1] = 2.0;
            particle[i].pos[2] = 3.0;
            particle[i].dir[0] = 4.0;
            particle[i].dir[1] = 5.0;
            particle[i].dir[2] = 6.0;
            particle[i].energy[0] = 7.0;
            particle[i].weight[0] = 8.0;
            particle[i].index = i;
            particle[i].detectorIndex = 99;
            particle[i].particleType = 0;
        }
        controller.setCapacity(3);
        controller.add( particle, 10 );
        CHECK_EQUAL(1, controller.size());
        CHECK_EQUAL(3, controller.getNFlushes());
    }

    TEST_FIXTURE(UnitControllerSetup, single_ray ){
        std::cout << "Debug: CollisionPointController_unit_tester -- single_ray\n";
        CollisionPointController<MonteRay_SpatialGrid> controller( 1,
                1,
                pGrid.get(),
                pMatList.get(),
                pMatProps.get(),
                pTally.get() );

        setup();

        unsigned int matID=0;
        gpuFloatType_t energy = 1.0;
        gpuFloatType_t density = 1.0;
        double testXS = pMatList->material(matID).getTotalXS(energy, density);
        CHECK_CLOSE(.602214179f/1.00866491597f, testXS, 1e-6);

        gpuFloatType_t x = 0.5;
        gpuFloatType_t y = 0.5;
        gpuFloatType_t z = 0.5;

        unsigned i = pGrid->getIndex( Position_t(x, y, z) );
        CHECK_EQUAL( 555, i);

        ParticleRay_t particle;

        particle.pos[0] = x;
        particle.pos[1] = y;
        particle.pos[2] = z;

        particle.dir[0] = 1.0;
        particle.dir[1] = 0.0;
        particle.dir[2] = 0.0;

        particle.energy[0] = 1.0;
        particle.weight[0] = 1.0;
        particle.index = i;
        particle.detectorIndex = 1;
        particle.particleType = 0;

        controller.add(  particle );

        std::cout << "Debug: CollisionPointController_unit_tester -- single_ray - flushing controller \n";
        controller.flush(true);

        std::cout << "Debug: CollisionPointController_unit_tester -- single_ray - copyToCPU \n";

        float distance = 0.5f;
        CHECK_CLOSE( (1.0f-std::exp(-testXS*distance))/testXS, pTally->getTally(i), 1e-5 );
        std::cout << "Debug: CollisionPointController_unit_tester -- finished- single_ray\n";
    }

    TEST_FIXTURE(UnitControllerSetup, write_single_ray_to_file ){
        std::cout << "Debug: CollisionPointController_unit_tester -- write_single_ray_to_file\n";
        CollisionPointController<MonteRay_SpatialGrid> controller( 1,
                1,
                pGrid.get(),
                pMatList.get(),
                pMatProps.get(),
                pTally.get() );

        setup();

        controller.setOutputFileName( "single_ray_collision.bin" );
        CHECK_EQUAL( true, controller.isSendingToFile() );

        unsigned int matID=0;
        gpuFloatType_t energy = 1.0;
        gpuFloatType_t density = 1.0;
        double testXS = pMatList->material(matID).getTotalXS(energy, density);
        CHECK_CLOSE(.602214179f/1.00866491597f, testXS, 1e-6);

        gpuFloatType_t x = 0.5;
        gpuFloatType_t y = 0.5;
        gpuFloatType_t z = 0.5;

        unsigned i = pGrid->getIndex( Position_t(x, y, z) );
        CHECK_EQUAL( 555, i);

        ParticleRay_t particle;

        particle.pos[0] = x;
        particle.pos[1] = y;
        particle.pos[2] = z;

        particle.dir[0] = 1.0;
        particle.dir[1] = 0.0;
        particle.dir[2] = 0.0;

        particle.energy[0] = 1.0;
        particle.weight[0] = 1.0;
        particle.index = i;
        particle.detectorIndex = 1;
        particle.particleType = 0;

        controller.add(  particle );

        controller.flush(true);
    }


    TEST_FIXTURE(UnitControllerSetup, read_single_ray_to_file ){
        std::cout << "Debug: CollisionPointController_unit_tester -- read_single_ray_to_file\n";
        CollisionPointController<MonteRay_SpatialGrid> controller( 1,
                1,
                pGrid.get(),
                pMatList.get(),
                pMatProps.get(),
                pTally.get() );

        setup();


        unsigned int matID=0;
        gpuFloatType_t energy = 1.0;
        gpuFloatType_t density = 1.0;
        double testXS = pMatList->material(matID).getTotalXS(energy, density);
        CHECK_CLOSE(.602214179f/1.00866491597f, testXS, 1e-6);

        gpuFloatType_t x = 0.5;
        gpuFloatType_t y = 0.5;
        gpuFloatType_t z = 0.5;

        unsigned i = pGrid->getIndex( Position_t(x, y, z) );
        CHECK_EQUAL( 555, i);

        controller.readCollisionsFromFile( "single_ray_collision.bin" );

        float distance = 0.5f;
        CHECK_CLOSE( (1.0f-std::exp(-testXS*distance))/testXS, pTally->getTally(i), 1e-5 );
    }

    TEST_FIXTURE(UnitControllerSetup, set_write_to_file_only_via_ctor ){
        std::cout << "Debug: CollisionPointController_unit_tester -- add_a_particle\n";
        CollisionPointController<MonteRay_SpatialGrid> controller( 2, std::string("collisionPoints_via_ctor_test_file.bin") );
        CHECK_EQUAL( true, controller.isSendingToFile() );

        gpuFloatType_t x = 0.5;
        gpuFloatType_t y = 0.5;
        gpuFloatType_t z = 0.5;

        unsigned i = pGrid->getIndex( Position_t(x, y, z) );
        CHECK_EQUAL( 555, i);

        ParticleRay_t particle;

        particle.pos[0] = x;
        particle.pos[1] = y;
        particle.pos[2] = z;

        particle.dir[0] = 1.0;
        particle.dir[1] = 0.0;
        particle.dir[2] = 0.0;

        particle.energy[0] = 1.0;
        particle.weight[0] = 1.0;
        particle.index = i;
        particle.detectorIndex = 1;
        particle.particleType = 0;

        controller.add(  particle );

        controller.flush(true);
    }

    TEST_FIXTURE(UnitControllerSetup, read_single_ray_to_file_from_writeonly_ctor ){
        std::cout << "Debug: CollisionPointController_unit_tester -- read_single_ray_to_file_from_writeonly_ctor\n";
        CollisionPointController<MonteRay_SpatialGrid> controller( 1,
                1,
                pGrid.get(),
                pMatList.get(),
                pMatProps.get(),
                pTally.get() );

        setup();


        unsigned int matID=0;
        gpuFloatType_t energy = 1.0;
        gpuFloatType_t density = 1.0;
        double testXS = pMatList->material(matID).getTotalXS(energy, density);
        CHECK_CLOSE(.602214179f/1.00866491597f, testXS, 1e-6);

        gpuFloatType_t x = 0.5;
        gpuFloatType_t y = 0.5;
        gpuFloatType_t z = 0.5;

        unsigned i = pGrid->getIndex( Position_t(x, y, z) );
        CHECK_EQUAL( 555, i);

        controller.readCollisionsFromFile( "collisionPoints_via_ctor_test_file.bin" );

        float distance = 0.5f;
        CHECK_CLOSE( (1.0f-std::exp(-testXS*distance))/testXS, pTally->getTally(i), 1e-5 );
    }
}

}
