#include <UnitTest++.h>

#include <iostream>
#include <cstdio>

#include "Material.hh"
#include "GPUUtilityFunctions.hh"
#include "SimpleVector.hh"

using namespace MonteRay;

struct CrossSection{
  int zaid;
  constexpr auto ZAID() const {return zaid;}
  constexpr gpuFloatType_t AWR() const {return 1.0;}
  constexpr gpuFloatType_t getTotalXS(gpuFloatType_t) const {return 1.0;}
};

struct CrossSectionList{
  SimpleVector<CrossSection> xs_vec;

  const CrossSection& getXSByZAID(int ZAID) const {
    auto loc = std::find_if(xs_vec.begin(), xs_vec.end(), 
        [ZAID](auto&& xs){ return xs.ZAID() == ZAID; } );
    const CrossSection* retval = (loc != xs_vec.end()) ?  &(*loc) : nullptr;
    return retval;
  }
};


class MaterialFixture{
  public:
  using Material_t = Material<CrossSection>;
  CrossSectionList xsList;
  Material_t mat;

  MaterialFixture(){
    using XS = CrossSection;
    xsList = CrossSectionList{{XS{1001}, XS{2004}, XS{6012}}};
    auto mb = Material_t::make_builder(xsList);
    mb.addIsotope(2.0, 1001);
    mb.addIsotope(3.0, 2004);
    mb.addIsotope(5.0, 6012);
    mat = mb.build();
  }
};

SUITE( Material_tester ) {

  constexpr double close = 1.0E-6;

  TEST_FIXTURE( MaterialFixture, builder ) {

    CHECK_CLOSE(mat.fraction(0), 0.2, close);
    CHECK_CLOSE(mat.fraction(1), 0.3, close);
    CHECK_CLOSE(mat.fraction(2), 0.5, close);

    CHECK_EQUAL(1001, mat.xs(0).ZAID());
    CHECK_EQUAL(2004, mat.xs(1).ZAID());
    CHECK_EQUAL(6012, mat.xs(2).ZAID());

    CHECK_CLOSE(mat.atomicWeight(), neutron_molar_mass, close);

    CHECK_EQUAL(mat.numIsotopes(), 3);

  }

  TEST_FIXTURE( MaterialFixture, testing) {

    CHECK_CLOSE(mat.fraction(0), 0.2, close);
    CHECK_CLOSE(mat.fraction(1), 0.3, close);
    CHECK_CLOSE(mat.fraction(2), 0.5, close);

    CHECK_EQUAL(1001, mat.xs(0).ZAID());
    CHECK_EQUAL(2004, mat.xs(1).ZAID());
    CHECK_EQUAL(6012, mat.xs(2).ZAID());

    CHECK_CLOSE(mat.atomicWeight(), neutron_molar_mass, close);

    CHECK_EQUAL(mat.numIsotopes(), 3);

  }

  TEST_FIXTURE( MaterialFixture, TotalXS ) {
    gpuFloatType_t E = 1.0;
    gpuFloatType_t density = 2.0;
    CHECK_CLOSE(mat.getMicroTotalXS(E), 1.0, close);
    CHECK_CLOSE(mat.getTotalXS(E, density), mat.getMicroTotalXS(E) * density * AvogadroBarn / mat.atomicWeight(), close);

#ifdef __HIPCC__
    int* zaid;
    hipMallocManaged(&zaid, sizeof(int));
    gpuFloatType_t* micro;
    hipMallocManaged(&micro, sizeof(gpuFloatType_t));
    gpuFloatType_t* macro;
    hipMallocManaged(&macro, sizeof(gpuFloatType_t));

    auto matl = mat;
    auto func = [=] __device__ () {
      *zaid = matl.xs(1).ZAID();
      *micro = matl.getMicroTotalXS(E);
      *macro = matl.getTotalXS(E, density);
    };

    d_invoker<<<1, 1>>>(func);
    hipDeviceSynchronize();
    CHECK_EQUAL(*zaid, 2004);
    CHECK_CLOSE(*micro,  1.0, close);
    CHECK_CLOSE(*macro, mat.getMicroTotalXS(E) * density * AvogadroBarn / mat.atomicWeight(), close);
    hipFree(zaid);
    hipFree(micro);
    hipFree(macro);
#endif

  }

  TEST_FIXTURE ( MaterialFixture, write_and_read ){

    std::stringstream stream;
    mat.write(stream);
    Material_t::Builder<CrossSectionList> mat_builder(xsList);
    mat_builder.read(stream);
    auto newMat = mat_builder.build();
    CHECK_EQUAL(newMat.atomicWeight(), mat.atomicWeight());
    CHECK_EQUAL(newMat.numIsotopes(), mat.numIsotopes());
    for (size_t i = 0; i < newMat.numIsotopes(); i++){
      CHECK_EQUAL(newMat.fraction(i), mat.fraction(i));
      CHECK_EQUAL(newMat.xs(i).AWR(), mat.xs(i).AWR());
    }
  }

}
