#include <hip/hip_runtime.h>

#include <iostream>

#include "MonteRayDefinitions.hh"

#include "Material_test_helper.hh"


MaterialTestHelper::MaterialTestHelper(){}

MaterialTestHelper::~MaterialTestHelper(){}

void MaterialTestHelper::setupTimers(){
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
}

void MaterialTestHelper::stopTimers(){
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	float elapsedTime;

	hipEventElapsedTime(&elapsedTime, start, stop );

	std::cout << "MaterialTestHelper: Elapsed time in CUDA kernel=" << elapsedTime << " msec" << std::endl;
}


