#include <iostream>

#include "MonteRayDefinitions.hh"

#include "Material_test_helper.hh"


MaterialTestHelper::MaterialTestHelper(){}

MaterialTestHelper::~MaterialTestHelper(){}

void MaterialTestHelper::setupTimers(){
#ifdef __HIPCC__
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
#endif
}

void MaterialTestHelper::stopTimers(){
	float elapsedTime;

#ifdef __HIPCC__
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop );
#endif

	std::cout << "MaterialTestHelper: Elapsed time in CUDA kernel=" << elapsedTime << " msec" << std::endl;
}


